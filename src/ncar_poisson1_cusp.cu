#include "hip/hip_runtime.h"
#include <math.h>

#include <stdlib.h>
#include "grid.h"
#include "ncar_poisson1_cusp.h"
#include "exact_solution.h"

#include <cusp/hyb_matrix.h>
#include <cusp/coo_matrix.h>
#include <cusp/blas.h>
#include <cusp/print.h>
#include <cusp/krylov/bicgstab.h>
#include <cusp/array1d.h>
#include <cusp/io/matrix_market.h>
#include <cusp/transpose.h>
using namespace std;

#define FLOAT float

NCARPoisson1_CUSP::NCARPoisson1_CUSP(ExactSolution* _solution, GPU* subdomain_, Derivative* der_, int rank, int dim_num_) :
        NCARPoisson1(_solution, subdomain_, der_, rank, dim_num_)
{}

//----------------------------------------------------------------------

NCARPoisson1_CUSP::~NCARPoisson1_CUSP() {
}
//----------------------------------------------------------------------
// Solve the poisson system.
// NOTE: this routine is old and uses a possibly incorrect method for solving with the
// neumann boundary conditions. I am starting an alternate routine to solve the system
// in the same fashion that Joe solved the system.
void NCARPoisson1_CUSP::solve(Communicator* comm_unit) {

    if (subdomain == NULL) {
        cerr
                << "In " << __FILE__
                << " No GPU class passed to Constructor. Cannot perform intermediate communication/updates in solver."
                << endl;
        exit(EXIT_FAILURE);
    } else {

        int nb = subdomain->global_boundary_nodes.size();
        // All interior and boundary nodes are included in the stencils.
        // The first nb Q_stencils should be the global boundary nodes
        int ni = subdomain->Q_stencils.size() - nb;

        int nn = (nb + ni) ;
        double err_norm = 100;
        double prev_err_norm = 100;

        double left_eps = 0.1;
        double right_eps = 30.;
        double new_eps = left_eps;

        double prev_eps = left_eps;

        bool goodDirection, wentRight;
        cout << "Allocating GPU arrays " <<endl;

        //cusp::array1d<FLOAT, cusp::device_memory> exact(nn, 0);
        //cusp::array1d<FLOAT, cusp::device_memory> approx_sol(nn, 0);
       // cusp::array1d<FLOAT, cusp::device_memory> error(nn, 0);
       // cusp::array1d<FLOAT, cusp::device_memory> expected(nn, 0);
       // cusp::array1d<FLOAT, cusp::device_memory> diff_lapl(nn, 0);

        int iter = 0;
        cout << "ENTERING EPSILON SEARCH LOOP " <<endl;
        while (err_norm > 1e-4 && iter < 1)
        {
            // TODO: solve this in parallel
            //     comm_unit->broadcastObjectUpdates(subdomain);

            // Do NOT use GPU as buffer for computation
            // Only go up to the number of stencils since we solve for a subset of the values in U_G
            // Since U_G in R is at end of U_G vector we can ignore those.
            //for (int i = 0; i < s.size(); i++) {
            //    s[i] = subdomain->U_G[i];
            //}

            //for (int i = 0; i < lapl_deriv.size(); i++) {
            //  Vec3& v = (*rbf_centers)[i];
            //    printf("(local: %d), lapl(%f,%f,%f)= %f\t%f\n", i, v.x(), v.y(), v.z(),
            //            lapl_deriv[i], s[i]);
            //}

            // Evan TODO:
            //
            // 1) Build a sparse matrix representation for all the interior derivative weights LA
            // 2) Build a full vector F = laplacian(u)
            // 3)

            // We are forming:
            //
            //  [ w_ddr -1/r*I  ]  [ A_boundary ]     [ 0  ]
            //  [ w_lapl    ]  [ A_interior ]  =  [ f_interior ]
            //
            // where w_lapl are the laplacian RBFFD weights for interior nodes (ni x nb + ni)
            // w_dr are the d/dr RBFFD weights (that is, the operator xd/dx +
            //

            // The w_lapl, weights for the laplacian, require (d^2 Phi / dx^2 + d^2 Phi / dy^2 + d^2 Phi / dz^2).
            // For w_ddr, weights for dA/dr, require (x*dPhi/dx + y*dPhi/dy + z*dPhi/dz). That means we will need to get
            // the stencil centers (Vec3) into the

            new_eps = left_eps + abs(right_eps - left_eps)/2.;
new_eps = 1.;
            der->setEpsilon(new_eps);
            cout << "USING EPSILON: " << new_eps << endl;

            int numNonZeros = 0;
#if 1
            for (int i = 0; i < nb + ni; i++) {
                //subdomain->printStencil(subdomain->Q_stencils[i], "Q[i]");
                // Compute all derivatives for our centers and return the number of
                // weights that will be available
                numNonZeros += der->computeWeights(subdomain->G_centers, subdomain->Q_stencils[i], i, dim_num);
            }
#else
            for (int i = 0; i < nb; i++) {
                // 1 nonzero per row for boundary (dirichlet has I since we know values and dont need weights)
                //der->computeWeights(subdomain->G_centers, subdomain->Q_stencils[i], subdomain->Q_stencils[i][0], dim_num);
                numNonZeros += 1;
            }
            for (int i = nb; i < nb + ni; i++) {
                //subdomain->printStencil(subdomain->Q_stencils[i], "Q[i]");
                // Compute all derivatives for our centers and return the number of
                // weights that will be available
                numNonZeros += der->computeWeights(subdomain->G_centers, subdomain->Q_stencils[i], subdomain->Q_stencils[i][0], dim_num);
            }
#endif
            cusp::coo_matrix<int, FLOAT, cusp::host_memory> L_host(nn, nn, numNonZeros);
            cusp::array1d<FLOAT, cusp::host_memory> F_host(nn, 0); // Initializes all elements to 0

            int indx = 0;

            // Fill Boundary weights
            for (int i = 0; i < nb; i++) {
                double* x_weights = der->getXWeights(subdomain->Q_stencils[i][0]);
                double* y_weights = der->getYWeights(subdomain->Q_stencils[i][0]);
                double* z_weights = der->getZWeights(subdomain->Q_stencils[i][0]);

                // DONT FORGET TO ADD IN THE -1/r on the stencil center weights
                Vec3& center = subdomain->G_centers[subdomain->Q_stencils[i][0]];
                double r = center.magnitude();

                if (r < 1e-8) {
                    cerr << "WARNING! VANISHING SPHERE RADIUS! CANNOT FILL -1/r in " << __FILE__ << endl;
                    exit(EXIT_FAILURE);
                }

#if 0
                // NEUMANN CONDITION:
                for (int j = 0; j < subdomain->Q_stencils[i].size(); j++) {
                        L_host.row_indices[indx] = i;
                        L_host.column_indices[indx] = subdomain->Q_stencils[i][j];
                        L_host.values[indx] = (center.x() / r) * x_weights[j] + (center.y()/r) * y_weights[j] + (center.z()/r) * z_weights[j];

                        // Remember to remove 1/r for the boundary condition: r d/dr(a/r) = 0
                        // When j == 0 we should have i = Q_stencil[i][j] (i.e., its the center element
                        if (j == 0) {
                            L_host.values[indx] -= 1./r;
                        }
                        indx++;
                 }
#else
#if 1
                // DIRICHLET CONDITION WITH 0s:
                for (int j = 0; j < subdomain->Q_stencils[i].size(); j++) {
                        L_host.row_indices[indx] = i;
                        L_host.column_indices[indx] = subdomain->Q_stencils[i][j];
                        L_host.values[indx] = 0;

                        if (j == 0) {
                            L_host.values[indx] = 1.f;
                        }
                        indx++;
                 }

#else
                // DIRICHLET CONDITION WITHOUT 0s
                        L_host.row_indices[indx] = i;
                        L_host.column_indices[indx] = i;
                        if (subdomain->Q_stencils[i][0] != i) {
                            cout << "WARNING!! i <> j" <<endl;
                            exit(EXIT_FAILURE);
                        }

                            L_host.values[indx] = 1.f;
                        indx++;
#endif
#endif
            }
//            cout << "INDX at end of boundary fill: " << indx << " NUM ROWS: " << nb+ni << endl;

            // Fill Interior weights
            for (int i = nb; i < nb+ni; i++) {
                double* lapl_weights = der->getLaplWeights(subdomain->Q_stencils[i][0]);
                for (int j = 0; j < subdomain->Q_stencils[i].size(); j++) {
                        L_host.row_indices[indx] = i;
                        L_host.column_indices[indx] = subdomain->Q_stencils[i][j];
                        L_host.values[indx] = (FLOAT)lapl_weights[j];
                        //cout << "lapl_weights[" << j << "] = " << lapl_weights[j] << endl;
                        indx++;
                 }
                Vec3& v = subdomain->G_centers[subdomain->Q_stencils[i][0]];
                F_host[i] = (FLOAT)exactSolution->laplacian(v.x(), v.y(), v.z(), 0.);
            }

            if ((indx - numNonZeros) != 0) {
                cerr << "WARNING! HOST MATRIX WAS NOT FILLED CORRECTLY. DISCREPANCY OF " << (indx - numNonZeros) << " NONZERO ELEMENTS!" << endl;
                exit(EXIT_FAILURE);
            }

           // cusp::print_matrix(F_host);

            // The way we fill the matrix is sorted by row so calling this has no effect
            //L_host.sort_by_row();
            cusp::io::write_matrix_market_file(L_host, "L.mtx");

            cusp::csr_matrix<int, FLOAT, cusp::device_memory> L_device;
            cusp::io::read_matrix_market_file(L_device, "L.mtx");

            cout << "READY TO SOLVE: " << endl;

            //cusp::csr_matrix<int, FLOAT, cusp::device_memory> L_device  = L_host;
            //cusp::transpose(L_host, L_device);

            cusp::array1d<FLOAT, cusp::device_memory> F_device = F_host;
            //cusp::print_matrix(F_host);

            cusp::array1d<FLOAT, cusp::device_memory> x_device(L_device.num_rows, 0.f);

            // set stopping criteria:
            //  iteration_limit    = 100
            //  relative_tolerance = 1e-6
            cusp::verbose_monitor<FLOAT> monitor(F_device, 100, 1e-6);

            // set preconditioner (identity)
            cusp::identity_operator<float, cusp::device_memory> M(L_device.num_rows, L_device.num_rows);

            // solve the linear system A * x = b with the BiConjugate Gradient Stabilized method
            cusp::krylov::bicgstab(L_device, x_device, F_device, monitor, M);

            // check residual norm
            cusp::array1d<float, cusp::device_memory> residual(L_device.num_rows, 0.0f);
            //L_device(x_device, residual);
            cusp::blas::axpby(x_device, F_device, residual, -1.0f, 1.0f);

            cout << "AXPBY RESIDUAL 2-NORM: " << cusp::blas::nrm2(residual) << endl;

            cusp::array1d<FLOAT, cusp::host_memory> x_host = x_device;
            cusp::array1d<FLOAT, cusp::host_memory> exact_H(F_device.size(), 0.f);

            //cout << "F = [";
            for (int i = 0; i < nb + ni; i++) {
                exact_H[subdomain->Q_stencils[i][0]] = (FLOAT)exactSolution->at(subdomain->G_centers[subdomain->Q_stencils[i][0]], 0.);
            //    cout << F_host[i] <<"; ";
            }
            //cout << "];" << endl;
            //cusp::array1d<FLOAT, cusp::device_memory> exact_D = exact_H;

            //cusp::array1d<FLOAT, cusp::device_memory> error_D(L_device.num_rows);
            // Compute our errors

            cout << "Error = [";
            cusp::array1d<FLOAT, cusp::host_memory> error_H(L_device.num_rows,0.f);
            for (int i = 0; i < L_device.num_rows; i++) {
                error_H[i] = x_host[i] - exact_H[i];
                cout << error_H[i] << ";\n";
            }
            cout << "]; " << endl;
            //error = (approx_sol - exact);
            //cusp::blas::axpby(exact_D, x_device, error_D, 1., -1.);

            //cusp::array1d<FLOAT, cusp::host_memory> error_H = error_D;

         //   cout << "Exact: "; cusp::print_matrix(exact_H);
         //   cout << "Sol: "; cusp::print_matrix(x_device);
         //   cout << "Error: "; cusp::print_matrix(error_H);
            //        expected = L*exact;
            //        diff_lapl = expected - F;

            prev_err_norm = err_norm;
            err_norm = cusp::blas::nrm2(error_H);

            cout << "2 Norm: " << err_norm << endl;

            if (prev_err_norm > err_norm) {
                goodDirection = true;
            } else {
                goodDirection = false;
            }

            if (goodDirection) {
                if (wentRight) {
                    prev_eps = left_eps;
                    left_eps = new_eps;
                    wentRight = true;
                } else {
                    prev_eps = right_eps;
                    right_eps = new_eps;
                    wentRight = false;
                }
            } else {
                if (wentRight) {
                    left_eps = prev_eps;
                    right_eps = new_eps;
                    wentRight = false;
                } else {
                    right_eps = prev_eps;
                    left_eps = new_eps;
                    wentRight = true;
                }
            }

            iter ++;
        }

#if 0
        cusp::array1d<FLOAT, cusp::host_memory> results[5](nn);

        results[0] = approx_sol;
        results[1] = exact;
        results[2] = error;
        results.print("\n\nRESULTS\n (APPROX SOLUTION; \tEXACT SOLUTION; \tABS ERROR \tExpected Laplacian(Using L*ExactSolution)\t Diff EXPECTED & EXACT Laplacian\n");

        // results.col(3) = expected;
        // results.col(4) = diff_lapl;
        cusp::print_matrix(approx_sol);
#endif
        //A.print("Full Solution (A) = ");
        //exact.print("Exact = ");
        //error.print("Error = ");


        cout.flush();
    }
    return;
}

//----------------------------------------------------------------------
