// includes, GL
#include <GL/glew.h>
#include <GL/glut.h>
#include <cuda_gl_interop.h>
#include <cutil.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
//#include "array_cuda_t.h"

#include "map.h"

//----------------------------------------------------------------------
extern "C++" 
void unmapBufferObject(unsigned int vbo)
{
	CUDA_SAFE_CALL(cudaGLUnmapBufferObject(vbo));
}
//----------------------------------------------------------------------
extern "C++" 
void unregisterBufferObject(unsigned int vbo)
{
	CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(vbo));
}
//----------------------------------------------------------------------
extern "C++" 
void registerBufferObject(unsigned int vbo)
{
	//printf("register: vbo= %d\n", vbo);
	CUDA_SAFE_CALL(cudaGLRegisterBufferObject(vbo));
}
//----------------------------------------------------------------------
extern "C++" 
void cudaInit()
{
	CUT_DEVICE_INIT();
}
//----------------------------------------------------------------------
extern "C" 
void copyFromDeviceToDevice(void* dst, const void* src, size_t count)
{
	hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice);
}
//----------------------------------------------------------------------
extern "C" 
void copyFromDeviceToHost(void* dst, const void* src, size_t count)
{
	hipMemcpy(dst, src, count, hipMemcpyDeviceToHost);
}
//----------------------------------------------------------------------
