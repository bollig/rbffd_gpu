// THIS IS adapted from verbose_monitor.cu
// PROVIDED BY THE CUSP v0.1 EXAMPLES

#include "grids/grid_reader.h"
#include "rbffd/rbffd.h"
#include "timer_eb.h" 

#include <cusp/hyb_matrix.h>
#include <cusp/ell_matrix.h>
#include <cusp/csr_matrix.h>
#include <cusp/coo_matrix.h>
#include <cusp/monitor.h>
#include <cusp/krylov/cg.h>
#include <cusp/krylov/gmres.h>
#include <cusp/gallery/poisson.h>
#include <cusp/print.h>
#include <cusp/array2d.h>
#include <cusp/multiply.h>
#include <cusp/blas.h>


#include <thrust/host_vector.h> 
#include <thrust/device_vector.h>
#include <thrust/generate.h>

#include <iostream>
#include <sstream> 
using namespace std;

// TODO: 
//  benchmark assembly
//  benchmark spmv
//  benchmark CPU vs GPU spmv

EB::TimerList tm;

template <typename MatT>
void benchmarkMultiplyHost(MatT& A) {
    // If we multiply a vector of 1s we should see our result equal 0 (if our
    // RBF-FD weights are good)
    cusp::array1d<double, cusp::host_memory> x(A.num_rows, 1);
#if 0
    // generate random data on the host to make sure when we multiply we
    // actually replace b
    thrust::host_vector<double> h_vec(A.num_rows);
    thrust::generate(h_vec.begin(), h_vec.end(), rand);

    // transfer to device and compute sum
    cusp::array1d<double, cusp::host_memory> b = h_vec;
    x = h_vec;
#else 
    cusp::array1d<double, cusp::host_memory> b(A.num_rows, 1);
#endif 
    cusp::multiply(A, x, b); 


    std::cout << "l1   Norm: " << cusp::blas::nrm1(b) << std::endl;  
    std::cout << "l2   Norm: " << cusp::blas::nrm2(b) << std::endl;  
    std::cout << "linf Norm: " << cusp::blas::nrmmax(b) << std::endl;  

#if 0
    cusp::array1d<double, cusp::host_memory> b_host = b;
    std::cout << "l1   Norm: " << cusp::blas::nrm1(b_host) << std::endl;  
    std::cout << "l2   Norm: " << cusp::blas::nrm2(b_host) << std::endl;  
    std::cout << "linf Norm: " << cusp::blas::nrmmax(b_host) << std::endl;  
#endif 
#if 0
    cusp::print(b); 
#endif 
}

template <typename MatT>
void benchmarkMultiplyDevice(MatT& A) {
    // If we multiply a vector of 1s we should see our result equal 0 (if our
    // RBF-FD weights are good)
    cusp::array1d<double, cusp::device_memory> x(A.num_rows, 1);
#if 0
    // generate random data on the host to make sure when we multiply we
    // actually replace b
    thrust::host_vector<double> h_vec(A.num_rows);
    thrust::generate(h_vec.begin(), h_vec.end(), rand);

    // transfer to device and compute sum
    cusp::array1d<double, cusp::device_memory> b = h_vec;
    x = h_vec;
#else 
    cusp::array1d<double, cusp::device_memory> b(A.num_rows, 1);
#endif 
    cusp::multiply(A, x, b); 

#if 1
    std::cout << "l1   Norm: " << cusp::blas::nrm1(b) << std::endl;  
    std::cout << "l2   Norm: " << cusp::blas::nrm2(b) << std::endl;  
    std::cout << "linf Norm: " << cusp::blas::nrmmax(b) << std::endl;  
#endif 
    hipDeviceSynchronize();
#if 0
    cusp::print(b); 
#endif 
}

void test_COO ( RBFFD& der, Grid& grid, int platform) {
    typedef cusp::coo_matrix<int, double, cusp::host_memory> MatType; 
    typedef cusp::coo_matrix<int, double, cusp::device_memory> MatTypeGPU; 

    unsigned int N = grid.getNodeListSize(); 
    unsigned int n = grid.getMaxStencilSize(); 

    std::ostringstream timer_basic_name; 
    timer_basic_name << N; 
    timer_basic_name << "_COO"; 
    if (platform) {
        timer_basic_name << "_GPU"; 
    } else {
        timer_basic_name << "_CPU"; 
    }

    std::ostringstream assemble_timer_name(timer_basic_name.str()); 
    std::ostringstream multiply_timer_name(timer_basic_name.str()); 
    assemble_timer_name << "_assemble"; 
    std::string assemble_label = timer_basic_name.str() + " Assemble"; 
    multiply_timer_name << "_multiply"; 
    std::string multiply_label = timer_basic_name.str() + " Multiply"; 

    if (!tm.contains(assemble_timer_name.str())) {
        tm[assemble_timer_name.str()] = new EB::Timer(assemble_label.c_str());  
        tm[multiply_timer_name.str()] = new EB::Timer(multiply_label.c_str());
    }
    std::cout << "WORKING ON: " << assemble_label.c_str() << std::endl;
    tm[assemble_timer_name.str()]->start();

    MatType A( N , N , N*n ); 

    unsigned int ind = 0; 
    for (int i = 0; i < A.num_rows; i++) {
        StencilType& sten = grid.getStencil(i); 
        double* lapl = der.getStencilWeights(RBFFD::LSFC, i); 

        // Off diagonals
        for (unsigned int j = 0; j < n; j++) {
            A.row_indices[ind] =  i; 
            A.column_indices[ind] =  sten[j]; 
            A.values[ind] =  -lapl[j]; 
            ind++; 
        }
    }
    tm[assemble_timer_name.str()]->stop();
#if 0
    std::cout << "N = " << N << "\t n = " << n << std::endl;
    cusp::array2d<double, cusp::host_memory> A_full(A); 
    cusp::print(A_full); 
    cusp::print(A); 
#endif
    std::cout << "\t\t\tMultiply\n";
    tm[multiply_timer_name.str()]->start();
    if (platform) {
        MatTypeGPU A_gpu(A); 
        benchmarkMultiplyDevice<MatTypeGPU>(A_gpu); 
    } else { 
        benchmarkMultiplyHost<MatType>(A); 
    }
    tm[multiply_timer_name.str()]->stop();
}

void test_CSR ( RBFFD& der, Grid& grid, int platform) {
    typedef cusp::csr_matrix<int, double, cusp::host_memory> MatType; 
    typedef cusp::csr_matrix<int, double, cusp::device_memory> MatTypeGPU; 

    unsigned int N = grid.getNodeListSize(); 
    unsigned int n = grid.getMaxStencilSize(); 

    std::ostringstream timer_basic_name; 
    timer_basic_name << N; 
    timer_basic_name << "_CSR"; 
    if (platform) {
        timer_basic_name << "_GPU"; 
    } else {
        timer_basic_name << "_CPU"; 
    }

    std::ostringstream assemble_timer_name(timer_basic_name.str()); 
    std::ostringstream multiply_timer_name(timer_basic_name.str()); 
    assemble_timer_name << "_assemble"; 
    std::string assemble_label = timer_basic_name.str() + " Assemble"; 
    multiply_timer_name << "_multiply"; 
    std::string multiply_label = timer_basic_name.str() + " Multiply"; 


    if (!tm.contains(assemble_timer_name.str())) {
        tm[assemble_timer_name.str()] = new EB::Timer(assemble_label.c_str());  
        tm[multiply_timer_name.str()] = new EB::Timer(multiply_label.c_str());
    }

    std::cout << "WORKING ON: " << assemble_label.c_str() << std::endl;
    tm[assemble_timer_name.str()]->start();

    MatType A( N , N , N*n ); 

    unsigned int ind = 0; 
    for (int i = 0; i < A.num_rows; i++) {
        StencilType& sten = grid.getStencil(i); 
        double* lapl = der.getStencilWeights(RBFFD::LSFC, i); 

        A.row_offsets[i] = ind;

        // Off diagonals
        for (unsigned int j = 0; j < n; j++) {
            A.column_indices[ind] =  sten[j]; 
            A.values[ind] =  -lapl[j]; 
            ind++; 
        }
    }
    tm[assemble_timer_name.str()]->stop();
    std::cout << "\t\t\tMultiply\n"; 
#if 0
    std::cout << "N = " << N << "\t n = " << n << std::endl;
    cusp::array2d<double, cusp::host_memory> A_full(A); 
    cusp::print(A_full); 
    cusp::print(A); 
#endif 
    tm[multiply_timer_name.str()]->start();
    if (platform) {
        MatTypeGPU A_gpu(A); 
        benchmarkMultiplyDevice<MatTypeGPU>(A_gpu); 
    } else { 
        benchmarkMultiplyHost<MatType>(A); 
    }
    tm[multiply_timer_name.str()]->stop();
}


void test_ELL ( RBFFD& der, Grid& grid, int platform) {
    typedef cusp::ell_matrix<int, double, cusp::host_memory> MatType; 
    typedef cusp::ell_matrix<int, double, cusp::device_memory> MatTypeGPU; 

    unsigned int N = grid.getNodeListSize(); 
    unsigned int n = grid.getMaxStencilSize(); 

    std::ostringstream timer_basic_name; 
    timer_basic_name << N; 
    timer_basic_name << "_ELL"; 
    if (platform) {
        timer_basic_name << "_GPU"; 
    } else {
        timer_basic_name << "_CPU"; 
    }

    std::ostringstream assemble_timer_name(timer_basic_name.str()); 
    std::ostringstream multiply_timer_name(timer_basic_name.str()); 
    assemble_timer_name << "_assemble"; 
    std::string assemble_label = timer_basic_name.str() + " Assemble"; 
    multiply_timer_name << "_multiply"; 
    std::string multiply_label = timer_basic_name.str() + " Multiply"; 

    if (!tm.contains(assemble_timer_name.str())) {
        tm[assemble_timer_name.str()] = new EB::Timer(assemble_label.c_str());  
        tm[multiply_timer_name.str()] = new EB::Timer(multiply_label.c_str());
    }

    std::cout << "WORKING ON: " << assemble_label.c_str() << std::endl;
    tm[assemble_timer_name.str()]->start();

    // Allocate a (N,N) matrix with (N*n) total nonzeros and at most (n) nonzero per row
    MatType A( N , N , N*n , n ); 

    for (int i = 0; i < A.num_rows; i++) {
        StencilType& sten = grid.getStencil(i); 
        double* lapl = der.getStencilWeights(RBFFD::LSFC, i); 

        // Off diagonals
        for (unsigned int j = 0; j < n; j++) {
            A.column_indices(i, j) =  sten[j]; 
            A.values(i, j) =  -lapl[j]; 
        }
    }
    tm[assemble_timer_name.str()]->stop();
#if 0
    std::cout << "N = " << N << "\t n = " << n << std::endl;
    cusp::array2d<double, cusp::host_memory> A_full(A); 
    cusp::print(A_full); 
    cusp::print(A); 
#endif 
    std::cout << "\t\t\tMultiply\n";
    tm[multiply_timer_name.str()]->start();
    if (platform) {
        MatTypeGPU A_gpu(A); 
        benchmarkMultiplyDevice<MatTypeGPU>(A_gpu); 
    } else { 
        benchmarkMultiplyHost<MatType>(A); 
    }
    tm[multiply_timer_name.str()]->stop();
}

void test_HYB ( RBFFD& der, Grid& grid, int platform) {

    // The HYB format has both an ELL (where ALL rows have n nonzeros) and a
    // COO (surplus nonzeros per row. In our case we know we will ALWAYS have n
    // nonzeros for stencil weights per row, unless a weight computes to 0.
    // This means HYB is equivalent to ELL for us. If we convert from ELL to
    // HYB we *might* see a performance boost if their constructor is smart
    // enough to check for 0's, but I doubt it. We will still fill a HYB matrix
    // and test performance. perhaps there are other efficiency differences
    // between the two formats. 

    typedef cusp::hyb_matrix<int, double, cusp::host_memory> MatType; 
    typedef cusp::hyb_matrix<int, double, cusp::device_memory> MatTypeGPU; 

    unsigned int N = grid.getNodeListSize(); 
    unsigned int n = grid.getMaxStencilSize(); 

    std::ostringstream timer_basic_name; 
    timer_basic_name << N; 
    timer_basic_name << "_HYB"; 
    if (platform) {
        timer_basic_name << "_GPU"; 
    } else {
        timer_basic_name << "_CPU"; 
    }

    std::ostringstream assemble_timer_name(timer_basic_name.str()); 
    std::ostringstream multiply_timer_name(timer_basic_name.str()); 
    assemble_timer_name << "_assemble"; 
    std::string assemble_label = timer_basic_name.str() + " Assemble"; 
    multiply_timer_name << "_multiply"; 
    std::string multiply_label = timer_basic_name.str() + " Multiply"; 

    if (!tm.contains(assemble_timer_name.str())) {
        tm[assemble_timer_name.str()] = new EB::Timer(assemble_label.c_str());  
        tm[multiply_timer_name.str()] = new EB::Timer(multiply_label.c_str());
    }

    std::cout << "WORKING ON: " << assemble_label.c_str() << std::endl;
    tm[assemble_timer_name.str()]->start();
    // Allocate a (N,N) matrix with (N*n) total nonzeros and at most (n) nonzero per row
    // and 0 extra non-zeros per row
    MatType A( N , N , N*n , 0 , n ); 

    for (int i = 0; i < A.num_rows; i++) {
        StencilType& sten = grid.getStencil(i); 
        double* lapl = der.getStencilWeights(RBFFD::LSFC, i); 

        // Off diagonals
        for (unsigned int j = 0; j < n; j++) {
            A.ell.column_indices(i, j) =  sten[j]; 
            A.ell.values(i, j) =  -lapl[j]; 
            // A.coo.row_indices[ind] = 0; ...
        }
    }
    tm[assemble_timer_name.str()]->stop();
#if 0
    std::cout << "N = " << N << "\t n = " << n << std::endl;
    cusp::array2d<double, cusp::host_memory> A_full(A); 
    cusp::print(A_full); 
    cusp::print(A); 
#endif 
    std::cout << "\t\t\tMultiply\n";
    tm[multiply_timer_name.str()]->start();
    if (platform) {
        MatTypeGPU A_gpu(A); 
        benchmarkMultiplyDevice<MatTypeGPU>(A_gpu); 
    } else { 
        benchmarkMultiplyHost<MatType>(A); 
    } 
    tm[multiply_timer_name.str()]->stop();
}



void testSPMV(int MAT_TYPE, int PLATFORM, RBFFD& der, Grid& grid) { 

    switch (MAT_TYPE) {
        case 0:  
            test_COO(der, grid, PLATFORM); 
            break; 
        case 1: 
            test_CSR(der, grid, PLATFORM); 
            break; 
        case 2: 
            test_ELL(der, grid, PLATFORM); 
            break; 
        case 3: 
            test_HYB(der, grid, PLATFORM); 
            break; 
        default: 
            std::cout << "INVALID SPMV TYPE\n"; 
            break;  
    }
}


int main(void)
{
    bool writeIntermediate = true; 

    std::vector<std::string> grids; 
#if 0
    grids.push_back("~/GRIDS/md/md003.00016"); 
    grids.push_back("~/GRIDS/md/md031.01024"); 
    grids.push_back("~/GRIDS/md/md050.02601"); 
    grids.push_back("~/GRIDS/md/md063.04096"); 
    grids.push_back("~/GRIDS/md/md089.08100"); 
    grids.push_back("~/GRIDS/md/md127.16384"); 
    grids.push_back("~/GRIDS/md/md165.27556"); 
#endif 

    grids.push_back("~/GRIDS/geoff/scvtmesh_100k_nodes.ascii"); 


    for (size_t i = 0; i < grids.size(); i++) {
        std::string& grid_name = grids[i]; 

        std::string weight_timer_name = grid_name + " Calc Weights";  

        tm[weight_timer_name] = new EB::Timer(weight_timer_name.c_str()); 


        // Get contours from rbfzone.blogspot.com to choose eps_c1 and eps_c2 based on stencil_size (n)
        unsigned int stencil_size = 40;
        double eps_c1 = 0.027;
        double eps_c2 = 0.274;


        GridReader grid(grid_name, 4); 
        grid.setMaxStencilSize(stencil_size); 
        // We do not read until generate is called: 

        Grid::GridLoadErrType err = grid.loadFromFile(); 
        if (err == Grid::NO_GRID_FILES) 
        {
            grid.generate();
            if (writeIntermediate) {
                grid.writeToFile(); 
            }
        } 
        std::cout << "Generate Stencils\n";
        Grid::GridLoadErrType st_err = grid.loadStencilsFromFile(); 
        if (st_err == Grid::NO_STENCIL_FILES) {
            //            grid.generateStencils(Grid::ST_BRUTE_FORCE);   
            //grid.generateStencils(Grid::ST_KDTREE);   
            grid.setNSHashDims(50, 50,50);  
            grid.generateStencils(Grid::ST_HASH);   
            if (writeIntermediate) {
                grid.writeToFile(); 
            }
        }


        std::cout << "Generate RBFFD Weights\n"; 
        tm[weight_timer_name]->start(); 
        RBFFD der(RBFFD::LSFC | RBFFD::XSFC | RBFFD::YSFC | RBFFD::ZSFC, &grid, 3, 0); 
        der.setEpsilonByParameters(eps_c1, eps_c2);
        int der_err = der.loadAllWeightsFromFile(); 
        if (der_err) {
            der.computeAllWeightsForAllStencils(); 

            tm[weight_timer_name]->start(); 
            if (writeIntermediate) {
                der.writeAllWeightsToFile(); 
            }
        }

        cout << "Allocating device memory\n" << std::endl;

        // enum MAT_TYPES {COO, CSR, ELL, HYB};
        // enum PLATFORMS {CPU, GPU}; 
        // j indexes MAT_TYPES. 
        for (int k = 0; k < 5; k++) {
#if 1
            for (int j = 0; j < 4; j++) 
#else 
                int j = 3;
#endif 
            {
                // CPU: 
                testSPMV(j, 1, der, grid); 
                // GPU: 
                testSPMV(j, 0, der, grid); 
            }
        }

#if 0
        cusp::array1d<double, cusp::host_memory> x(A.num_rows, 1); 
        cusp::array1d<double, cusp::host_memory> b = x; 

        std::cout << "Multiplying matrices\n";
        cusp::multiply(A,x,b); 
#endif 
    }
#if 0
    exit(-1);
    // Convert the 2D memory block to a sparse representation
    cusp::coo_matrix<int, double, cusp::device_memory> A_dev(A);

    // allocate storage for solution (x) and right hand side (b)
    cusp::array1d<double, cusp::device_memory> x_dev(A.num_rows, 0);
    cusp::array1d<double, cusp::device_memory> b_dev(A.num_rows, 1);

    // set stopping criteria:
    //  iteration_limit    = 100
    //  relative_tolerance = 1e-8
    cusp::verbose_monitor<double> monitor(b, 100, 1e-8);

    cout << "Starting GMRES\n" << std::endl;

    // solve the linear system A x = b
    cusp::krylov::gmres(A_dev, x_dev, b_dev, 30, monitor);

    cout << "GMRES complete\n" << std::endl;

    // monitor will report solver progress and results
#endif 
    tm.printAll();
    tm.writeToFile();
    return EXIT_SUCCESS;
}

