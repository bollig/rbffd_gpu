#include "hip/hip_runtime.h"
// TODO : test this: 
//#define CUSP_USE_TEXTURE_MEMORY

// THIS IS adapted from verbose_monitor.cu
// PROVIDED BY THE CUSP v0.1 EXAMPLES

#include "grids/grid_reader.h"
#include "rbffd/rbffd.h"
#include "timer_eb.h" 

#include "stokes_steady_cusp.h"

#include <cusp/hyb_matrix.h>
#include <cusp/ell_matrix.h>
#include <cusp/csr_matrix.h>
#include <cusp/coo_matrix.h>
#include <cusp/monitor.h>
#include <cusp/krylov/cg.h>
#include <cusp/krylov/gmres.h>
#include <cusp/gallery/poisson.h>
#include <cusp/print.h>
#include <cusp/array2d.h>
#include <cusp/multiply.h>
#include <cusp/blas.h>
#include <cusp/io/matrix_market.h>
#include <cusp/precond/diagonal.h>
#include <cusp/precond/ainv.h>
#include <cusp/precond/smoothed_aggregation.h>
#include <cusp/precond/aggregate.h>
#include <cusp/precond/smooth.h>
#include <cusp/precond/strength.h>

#include <thrust/host_vector.h> 
#include <thrust/device_vector.h>
#include <thrust/generate.h>


#include "utils/spherical_harmonics.h"

#include <iomanip>
#include <iostream>
#include <sstream> 
#include <map>
#include <fstream> 
#include <typeinfo> 
using namespace std;

namespace cusp
{
    StokesSteady::StokesSteady(RBFFD& der_ref, Grid& grid_ref, int PrimeGPU) 
        : der(der_ref), 
        grid(grid_ref),
        primeGPU(PrimeGPU) 
    {
    // N should be number of stencils in domain
    // n should be number of nodes per stencil
    // nb_bnd should be number of boundary nodes in domain
        N = grid.getNodeListSize(); 
        n = grid.getMaxStencilSize(); 
        nb_bnd = grid.getBoundaryIndicesSize();
        nrows = 4 * N + 4; 
        ncols = 4 * N + 4; 
        NNZ = 9*n*N+2*(4*N)+2*(3*N);  

        setupTimers();
    }


    void StokesSteady::setupTimers() {

        if (primeGPU) {
            sprintf(test_name, "%u PRIMING THE GPU", N);  
            sprintf(assemble_timer_name, "%u Primer Assemble", N);
            sprintf(copy_timer_name,     "%u Primer Copy To CUSP_DEVICE_CSR", N); 
            sprintf(test_timer_name, "%u Primer GMRES test", N); 
        } else { 
            sprintf(test_name, "%u GMRES GPU (CUSP_DEVICE_CSR)", N);  
            sprintf(assemble_timer_name, "%u CUSP_HOST_CSR Assemble", N);
            sprintf(copy_timer_name,     "%u CUSP_HOST_CSR Copy To CUSP_DEVICE_CSR", N); 
            sprintf(test_timer_name, "%u GPU GMRES test", N); 
        }

        if (!tm.contains(assemble_timer_name)) { tm[assemble_timer_name] = new EB::Timer(assemble_timer_name); } 
        if (!tm.contains(copy_timer_name)) { tm[copy_timer_name] = new EB::Timer(copy_timer_name); } 
        if (!tm.contains(test_timer_name)) { tm[test_timer_name] = new EB::Timer(test_timer_name); } 

    }


    void StokesSteady::SpMV_Device(DEVICE_MAT_t& A, DEVICE_VEC_t& F, DEVICE_VEC_t& U_exact, DEVICE_VEC_t& U_approx_gpu) {

    }


    // Perform GMRES on GPU
    void StokesSteady::GMRES_Device(DEVICE_MAT_t& A, DEVICE_VEC_t& F, DEVICE_VEC_t& U_exact, DEVICE_VEC_t& U_approx_gpu) {
#if 1
        size_t restart = 300; 
        int max_iters = 1000; 
        double rel_tol = 1e-6; 
#else 
        // Maximum number of iterations (total) 
        size_t max_iters = 500; 
        // restart the process every "restart" iterations
        size_t restart = 200; 
        double rel_tol = 1e-8; 
#endif 

        try {

            //    cusp::convergence_monitor<double> monitor( F, max_iters, 0, 1e-3); 
            cusp::default_monitor<double> monitor( F, max_iters, rel_tol ); //, max_iters, rel_tol);// , 1e-3); 
            //cusp::default_monitor<double> monitor( F, -1, rel_tol ); //, max_iters, rel_tol);// , 1e-3); 

            std::cout << "GMRES Starting Residual Norm: " << monitor.residual_norm() << std::endl;

            // 1e-8, 10000, 300); 
            int precondType = -1; 
            switch (precondType) {
                case 0: 
                    {
                        // Jacobi Preconditioning (DIAGONAL)
                        // Probably wont work well for RBF-FD since we're not diagonally dominant
                        cusp::precond::diagonal<double, cusp::device_memory> M(A);
                        cusp::krylov::gmres(A, U_approx_gpu, F, restart, monitor, M); 
                    }
                    break;
                case 1:
                    {
                        // Smoothed Aggregation (Algebraic MultiGrid. Works for Nonsym?)
                        cusp::precond::smoothed_aggregation<int, double, cusp::device_memory> M(A);
                        cusp::krylov::gmres(A, U_approx_gpu, F, restart, monitor, M); 
                    }
                    break; 
#if 0 
                    // ONLY SPD MATRICES
                case 0: 
                    // AINV using static dropping
                    cusp::precond::scaled_bridson_ainv<double, cusp::device_memory> M(A, 0, 10);
                    cusp::krylov::gmres(A, U_approx_gpu, F, restart, monitor, M); 
                    break; 
#endif 
#if 0 
                    // ONLY SPD MATRICES
                case 1: 
                    // AINV using standard drop tolerance
                    cusp::precond::scaled_bridson_ainv<double, cusp::device_memory> M(A, .1);
                    cusp::krylov::gmres(A, U_approx_gpu, F, restart, monitor, M); 
                    break; 
#endif 
#if 0
                    // ONLY FOR SPD MATRICES
                case 2: 
                    // AINV using novel cusp dropping strategy (TODO: lookup) 
                    cusp::precond::bridson_ainv<double, cusp::device_memory> M(A, 0, -1, true, 2);
                    cusp::krylov::gmres(A, U_approx_gpu, F, restart, monitor, M); 
#endif 
                case 2: 
                    {
                        // AINV using novel cusp dropping strategy 
                        // assumes that sparsity pattern of precond is same as A, plus
                        // 2 extra nonzeros per row 
                        // VERY SLOW TO BUILD; DOES NOT CONVERGE
                        cusp::precond::nonsym_bridson_ainv<double, cusp::device_memory> M(A, 0, -1, true, 2);
                        cusp::krylov::gmres(A, U_approx_gpu, F, restart, monitor, M); 
                    }
                case 3: 
                    {
                        // AINV using novel cusp dropping strategy 
                        // Assume 40 nonzeros per row, drop everthing else. 
                        // VERY SLOW TO BUILD; DOES NOT CONVERGE
                        cusp::precond::nonsym_bridson_ainv<double, cusp::device_memory> M(A, 0.1, 10, false, 0);
                        cusp::krylov::gmres(A, U_approx_gpu, F, restart, monitor, M); 
                    }
                default: 
                    // Solve unpreconditioned Au = F
                    cusp::krylov::gmres(A, U_approx_gpu, F, restart, monitor); 
            }
            hipDeviceSynchronize(); 

            //    monitor.print();

            if (monitor.converged())
            {
                std::cout << "\n[+++] Solver converged to " << monitor.relative_tolerance() << " relative tolerance";       
                std::cout << " after " << monitor.iteration_count() << " iterations" << std::endl << std::endl;
            }
            else
            {
                std::cout << "\n[XXX] Solver reached iteration limit " << monitor.iteration_limit() << " before converging";
                std::cout << " to " << monitor.relative_tolerance() << " relative tolerance " << std::endl << std::endl;
            }

            std::cout << "GMRES Iterations: " << monitor.iteration_count() << std::endl;
            std::cout << "GMRES Iteration Limit: " << monitor.iteration_limit() << std::endl;
            std::cout << "GMRES Residual Norm: " << monitor.residual_norm() << std::endl;
            std::cout << "GMRES Relative Tol: " << monitor.relative_tolerance() << std::endl;
            std::cout << "GMRES Absolute Tol: " << monitor.absolute_tolerance() << std::endl;
            std::cout << "GMRES Target Residual (Abs + Rel*norm(F)): " << monitor.tolerance() << std::endl;
        }
        catch(std::bad_alloc &e)
        {
            std::cerr << "Ran out of memory trying to compute GMRES: " << e.what() << std::endl;
            exit(-1);
        }
        catch(thrust::system_error &e)
        {
            std::cerr << "Some other error happened during GMRES: " << e.what() << std::endl;
            exit(-1);
        }


        try {

            typedef cusp::array1d<double, DEVICE_VEC_t>::view DEVICE_VEC_VIEW_t; 

            DEVICE_VEC_VIEW_t U_approx_view(U_exact.begin()+(U_exact.size() - F.size()), U_exact.end()); 

            DEVICE_VEC_t diff(U_approx_gpu); 

            //cusp::blas::axpy(U_exact.begin()+(U_exact.size() - F.size()), U_exact.end(), diff.begin(),  -1); 
            cusp::blas::axpy(U_approx_view, diff, -1); 

            std::cout << "Rel l1   Norm: " << cusp::blas::nrm1(diff) / cusp::blas::nrm1(U_exact) << std::endl;  
            std::cout << "Rel l2   Norm: " << cusp::blas::nrm2(diff) / cusp::blas::nrm2(U_exact) << std::endl;  
            std::cout << "Rel linf Norm: " << cusp::blas::nrmmax(diff) / cusp::blas::nrmmax(U_exact) << std::endl;  
        }
        catch(std::bad_alloc &e)
        {
            std::cerr << "Ran out of memory trying to compute Error Norms: " << e.what() << std::endl;
            exit(-1);
        }
        catch(thrust::system_error &e)
        {
            std::cerr << "Some other error happened during Error Norms: " << e.what() << std::endl;
            exit(-1);
        }
    }

    //---------------------------------

    void StokesSteady::assemble_System_Stokes( RBFFD& der, Grid& grid, HOST_MAT_t& A, HOST_VEC_t& F, HOST_VEC_t& U_exact){
        double eta = 1.;
        //double Ra = 1.e6;

        // We have different nb_stencils and nb_nodes when we parallelize. The subblocks might not be full
        unsigned int nb_stencils = grid.getStencilsSize();
        unsigned int nb_nodes = grid.getNodeListSize(); 
        unsigned int max_stencil_size = grid.getMaxStencilSize();
        unsigned int N = nb_nodes;
        // ---------------------------------------------------

        //------------- Fill the RHS of the System -------------
        // This is our manufactured solution:
        SphericalHarmonic::Sph32 UU; 
        SphericalHarmonic::Sph32105 VV; 
        SphericalHarmonic::Sph32 WW; 
        SphericalHarmonic::Sph32 PP; 

        std::vector<NodeType>& nodes = grid.getNodeList(); 

        //------------- Fill F -------------

        // U
        for (unsigned int j = 0; j < N; j++) {
            unsigned int row_ind = j + 0*N;
            NodeType& node = nodes[j]; 
            double Xx = node.x(); 
            double Yy = node.y(); 
            double Zz = node.z(); 

            U_exact[row_ind] = UU.eval(Xx,Yy,Zz); 
            F[row_ind] = -UU.lapl(Xx,Yy,Zz) + PP.d_dx(Xx,Yy,Zz);  
        }
#if 1

        // V
        for (unsigned int j = 0; j < N; j++) {
            unsigned int row_ind = j + 1*N;
            NodeType& node = nodes[j]; 
            double Xx = node.x(); 
            double Yy = node.y(); 
            double Zz = node.z(); 
            //double rr = sqrt(node.x()*node.x() + node.y()*node.y() + node.z()*node.z());
            //double dir = node.y();

            // F[row_ind] = (Ra * Temperature(j) * dir) / rr;  
            U_exact[row_ind] = VV.eval(Xx,Yy,Zz); 
            F[row_ind] = -VV.lapl(Xx,Yy,Zz) + PP.d_dy(Xx,Yy,Zz);  
        }

        // W
        for (unsigned int j = 0; j < N; j++) {
            unsigned int row_ind = j + 2*N;
            NodeType& node = nodes[j];
            double Xx = node.x(); 
            double Yy = node.y(); 
            double Zz = node.z(); 

            U_exact[row_ind] = WW.eval(Xx,Yy,Zz); 
            F[row_ind] = -WW.lapl(Xx,Yy,Zz) + PP.d_dz(Xx,Yy,Zz);  
        }

        // P
        for (unsigned int j = 0; j < N; j++) {
            unsigned int row_ind = j + 3*N;
            NodeType& node = nodes[j]; 
            double Xx = node.x(); 
            double Yy = node.y(); 
            double Zz = node.z(); 

            U_exact[row_ind] = PP.eval(Xx,Yy,Zz); 
            F[row_ind] = UU.d_dx(Xx,Yy,Zz) + VV.d_dy(Xx,Yy,Zz) + WW.d_dz(Xx,Yy,Zz);  
        }
#endif
        // Sum of U
        F[4*N+0] = 0.;

        // Sum of V
        F[4*N+1] = 0.;

        // Sum of W
        F[4*N+2] = 0.;

        // Sum of P
        F[4*N+3] = 0.;




        unsigned int ind = 0; 

        // -----------------  Fill LHS --------------------
        //
        // U (block)  row
        for (unsigned int i = 0; i < nb_stencils; i++) {
            StencilType& st = grid.getStencil(i);

            // TODO: change these to *SFC weights (when computed)
            double* ddx = der.getStencilWeights(RBFFD::XSFC, i);
            double* lapl = der.getStencilWeights(RBFFD::LSFC, i); 

            unsigned int diag_row_ind = i + 0*N;

            A.row_offsets[diag_row_ind] = ind; 


            for (unsigned int j = 0; j < st.size(); j++) {
                unsigned int diag_col_ind = st[j] + 0*N;

                A.column_indices[ind] = diag_col_ind; 
                A.values[ind]  = -eta * lapl[j];  
                ind++; 
            }
            for (unsigned int j = 0; j < st.size(); j++) {
                unsigned int diag_col_ind = st[j] + 3*N;

                A.column_indices[ind] = diag_col_ind; 
                A.values[ind]  = ddx[j];  
                ind++; 
            }

            // Added constraint to square mat and close nullspace
            A.column_indices[ind] = 4*N+0; 
            A.values[ind]  = 1;  
            ind++; 
        }

        // V (block)  row
        for (unsigned int i = 0; i < nb_stencils; i++) {
            StencilType& st = grid.getStencil(i);

            // TODO: change these to *SFC weights (when computed)
            double* ddy = der.getStencilWeights(RBFFD::YSFC, i);
            double* lapl = der.getStencilWeights(RBFFD::LSFC, i); 

            unsigned int diag_row_ind = i + 1*N;
            A.row_offsets[diag_row_ind] = ind; 

            for (unsigned int j = 0; j < st.size(); j++) {
                unsigned int diag_col_ind = st[j] + 1*N;

                A.column_indices[ind] = diag_col_ind; 
                A.values[ind]  = -eta * lapl[j];  
                ind++; 
            }
            for (unsigned int j = 0; j < st.size(); j++) {
                unsigned int diag_col_ind = st[j] + 3*N;

                A.column_indices[ind] = diag_col_ind; 
                A.values[ind]  = ddy[j];  
                ind++; 
            }

            // Added constraint to square mat and close nullspace
            A.column_indices[ind] = 4*N+1; 
            A.values[ind]  = 1;  
            ind++; 
        }

        // W (block)  row
        for (unsigned int i = 0; i < nb_stencils; i++) {
            StencilType& st = grid.getStencil(i);

            // TODO: change these to *SFC weights (when computed)
            double* ddz = der.getStencilWeights(RBFFD::ZSFC, i);
            double* lapl = der.getStencilWeights(RBFFD::LSFC, i); 

            unsigned int diag_row_ind = i + 2*N;
            A.row_offsets[diag_row_ind] = ind; 

            for (unsigned int j = 0; j < st.size(); j++) {
                unsigned int diag_col_ind = st[j] + 2*N;

                A.column_indices[ind] = diag_col_ind; 
                A.values[ind]  = -eta * lapl[j];  
                ind++; 
            }
            for (unsigned int j = 0; j < st.size(); j++) {
                unsigned int diag_col_ind = st[j] + 3*N;

                A.column_indices[ind] = diag_col_ind; 
                A.values[ind]  = ddz[j];
                ind++; 
            }

            // Added constraint to square mat and close nullspace
            A.column_indices[ind] = 4*N+2; 
            A.values[ind]  = 1;  
            ind++; 
        }


        // P (block)  row
        for (unsigned int i = 0; i < nb_stencils; i++) {
            StencilType& st = grid.getStencil(i);

            // TODO: change these to *SFC weights (when computed)
            double* ddx = der.getStencilWeights(RBFFD::XSFC, i);
            double* ddy = der.getStencilWeights(RBFFD::YSFC, i);
            double* ddz = der.getStencilWeights(RBFFD::ZSFC, i);

            unsigned int diag_row_ind = i + 3*N;
            A.row_offsets[diag_row_ind] = ind; 

            for (unsigned int j = 0; j < st.size(); j++) {
                unsigned int diag_col_ind = st[j] + 0*N;

                A.column_indices[ind] = diag_col_ind; 
                A.values[ind]  = ddx[j]; 
                ind++; 
            }
            for (unsigned int j = 0; j < st.size(); j++) {
                unsigned int diag_col_ind = st[j] + 1*N;

                A.column_indices[ind] = diag_col_ind; 
                A.values[ind]  = ddy[j]; 
                ind++; 
            }
            for (unsigned int j = 0; j < st.size(); j++) {
                unsigned int diag_col_ind = st[j] + 2*N;

                A.column_indices[ind] = diag_col_ind; 
                A.values[ind]  = ddz[j]; 
                ind++; 
            }

            // Added constraint to square mat and close nullspace
            A.column_indices[ind] = 4*N+3; 
            A.values[ind]  = 1;  
            ind++; 
        }

        // ------ EXTRA CONSTRAINT ROWS -----
        unsigned int diag_row_ind = 4*N;
        A.row_offsets[diag_row_ind] = ind;
        // U
        for (unsigned int j = 0; j < N; j++) {
            unsigned int diag_col_ind = j + 0*N;

            A.column_indices[ind] = diag_col_ind; 
            A.values[ind]  = 1;  
            ind++; 
        }

        diag_row_ind++; 
        A.row_offsets[diag_row_ind] = ind; 
        // V
        for (unsigned int j = 0; j < N; j++) {
            unsigned int diag_col_ind = j + 1*N;

            A.column_indices[ind] = diag_col_ind; 
            A.values[ind]  = 1;  
            ind++; 
        }

        diag_row_ind++; 
        A.row_offsets[diag_row_ind] = ind; 
        // W
        for (unsigned int j = 0; j < N; j++) {
            unsigned int diag_col_ind = j + 2*N;

            A.column_indices[ind] = diag_col_ind; 
            A.values[ind]  = 1;  
            ind++; 
        }

        diag_row_ind++; 
        A.row_offsets[diag_row_ind] = ind; 
        // P
        for (unsigned int j = 0; j < N; j++) {
            unsigned int diag_col_ind = j + 3*N;

            A.column_indices[ind] = diag_col_ind; 
            A.values[ind]  = 1;  
            ind++; 
        }

        // VERY IMPORTANT. UNSPECIFIED LAUNCH FAILURES ARE CAUSED BY FORGETTING THIS!
        A.row_offsets[4*N+4] = ind; 
    }


    void StokesSteady::write_System ( HOST_MAT_t& A, HOST_VEC_t& F, HOST_VEC_t& U_exact )
    {
        write_to_file(F, "output/F.mtx"); 
        write_to_file(U_exact, "output/U_exact.mtx"); 
        cusp::io::write_matrix_market_file(A,"output/LHS.mtx"); 
    }

    void StokesSteady::write_Solution( Grid& grid, HOST_VEC_t& U_exact, DEVICE_VEC_t& U_approx_gpu ) 
    {
        // IF we want to write details we need to copy back to host. 
        HOST_VEC_t U_approx(U_exact.size());

        if (U_approx_gpu.size() == U_exact.size()) {
            thrust::copy(U_approx_gpu.begin(), U_approx_gpu.end(), U_approx.begin());
        } else {
            thrust::copy(U_exact.begin(), U_exact.begin()+nb_bnd, U_approx.begin());
            thrust::copy(U_approx_gpu.begin(), U_approx_gpu.end(), U_approx.begin()+nb_bnd);
        }

        write_to_file(U_approx, "output/U_gpu.mtx"); 
    }

    void StokesSteady::assemble() {
        std::cout << "Assembling: " << test_name << std::endl;

        // ----- ASSEMBLE -----
        tm[assemble_timer_name]->start(); 
        A = new HOST_MAT_t(nrows, ncols, NNZ); 
        F = new HOST_VEC_t(nrows, 0);
        U_exact = new HOST_VEC_t(nrows, 0);
        assemble_System_Stokes(der, grid, *A, *F, *U_exact); 

        tm[assemble_timer_name]->stop(); 

        if (!primeGPU) {
            //write_System(*A, *F, *U_exact); 
        }
        // ----- SOLVE -----

        tm[copy_timer_name]->start();

        A_gpu = new DEVICE_MAT_t(*A); 
        F_gpu = new DEVICE_VEC_t(*F); 
        U_exact_gpu = new DEVICE_VEC_t(*U_exact); 
        U_approx_gpu = new DEVICE_VEC_t(F->size(), 0);

        tm[copy_timer_name]->stop();

    }

    void StokesSteady::solve() {
        std::cout << "Solving: " << test_name << std::endl;

        tm[test_timer_name]->start();
        // Use GMRES to solve A*u = F
        #if 1
        GMRES_Device(*A_gpu, *F_gpu, *U_exact_gpu, *U_approx_gpu);
        #else 
// Start by testing the parallel SpMV
        SpMV_Device(*A_gpu, *F_gpu, *U_exact_gpu, *U_approx_gpu);   
        #endif 
        tm[test_timer_name]->stop();

        if (!primeGPU) {
            write_Solution(grid, *U_exact, *U_approx_gpu); 
        }
    }

    StokesSteady::~StokesSteady() {
        std::cout << "Cleanup aisle three..."; 
        // Cleanup
        delete(A);
        delete(A_gpu);
        delete(F);
        delete(U_exact);
        delete(F_gpu);
        delete(U_exact_gpu);
        delete(U_approx_gpu);
        std::cout << "Done\n";
    }

};
