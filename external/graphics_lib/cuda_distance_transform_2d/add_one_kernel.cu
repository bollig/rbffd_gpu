#include "hip/hip_runtime.h"
#ifndef _ADD_ONE_KERNEL_H_
#define _ADD_ONE_KERNEL_H_
#
#include <stdio.h>
//#include "scan_efficient.cu"
#include <vector>
#include <cutil.h>

#include "local_macros.h"


///////////////////////////////////////////////////////////////////////////////
// Work-efficient compute implementation of scan, one thread per 2 elements

__global__ void scan_workefficient(float4 *g_idata, float4 *sum, int n);
__global__ void scan_workefficient_2(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width);
__global__ void scan_workefficient_3(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width);
__global__ void scan_workefficient_4(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width);
__global__ void scan_workefficient_5(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width);
__global__ void scan_workefficient_6(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width);
__global__ void scan_workefficient_7(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width);
__global__ void scan_workefficient_8(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width);
__global__ void scan_workefficient_8_larger(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width);
__global__ void scan_test_incoherent(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width);

#define ATOMICS
//Comment this line and add '-arch sm_11' to the command line to enable atomic operations.
#undef ATOMICS

//----------------------------------------------------------------------
__global__ void
//transformKernel(float* g_odata, int width, int height)
addKernel(float* g_odata, float* o_data, int width, int height)
{
#if 1
    // calculate normalized texture coordinates
    //unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    //unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	//unsigned int indx = y*width + x;

    unsigned int x = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    unsigned int y = __mul24(blockIdx.y,blockDim.y) + threadIdx.y;
	unsigned int indx = __mul24(y,width) + x;

	o_data[indx] = g_odata[indx] + 2.2f;
#endif
}
//----------------------------------------------------------------------
__global__ void
addArrayToArray(float* o_data, float* i1_data, float* i2_data, int width, int height)
{
    unsigned int x = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    unsigned int y = __mul24(blockIdx.y,blockDim.y) + threadIdx.y;
	unsigned int indx = __mul24(y,width) + x;

#if 0
   __shared__ float blocki1[16*8];
   __shared__ float blocki2[16*8];

   unsigned int index_block = __mul24(threadIdx.y, blockDim.x) + threadIdx.x;
   blocki1[index_block] = i1_data[indx];
   blocki2[index_block] = i2_data[indx];
   __syncthreads();

   o_data[indx]  =  blocki1[index_block] + blocki2[index_block];
#else 
   o_data[indx]  =  i1_data[indx] + i2_data[indx];
#endif
}
//----------------------------------------------------------------------
#define nb_seeds    8*16
__global__ void
binEfficientKernel(int* data, int* bins, int width, int h, int nb_bins)
{
    unsigned int x = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    unsigned int y = __mul24(blockIdx.y,blockDim.y) + threadIdx.y;
	unsigned int indx = __mul24(y,width) + x;

   // of course, in this test case, the integers are randomly distributed, 
   // but let us get an idea of efficiency anyway. 

   __shared__ int block[8][16];
   __shared__ int bins_s[nb_seeds]; // seeds 
   __shared__ int seed[nb_seeds]; // seeds 

	block[threadIdx.y][threadIdx.x] = data[indx];

	unsigned int loc = threadIdx.x + __mul24(threadIdx.y, blockDim.x);

	if (loc < nb_seeds) bins_s[loc] = 0;

	__syncthreads();

	unsigned int s = block[threadIdx.y][threadIdx.x];
    seed[s % nb_seeds] = s;

	// some of these threads are contending with each other. Is there a way
	// to lock shared memory?
    bins_s[s % nb_seeds] += 1;   // counter

	__syncthreads();

	 #if 0
	 if (loc < nb_seeds && bins_s[loc] > 0) {
	 	//bins[seed[loc]] += bins_s[loc];
	 	atomicAdd(bins+seed[loc], bins_s[loc]);
	 }
	 #endif
}
//----------------------------------------------------------------------
__global__ void
centroidVoronoiScanKernel(float4* data, float4* bins, int width, int h, int nb_bins, int4* newSeeds, int edge)
{
    unsigned int x = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    unsigned int y = __mul24(blockIdx.y,blockDim.y) + threadIdx.y;
	unsigned int indx = __mul24(y,width) + x;

    // Dynamically allocated shared memory for scan kernels
    //extern  __shared__  float temp[];
}
//----------------------------------------------------------------------
__global__ void
centroidVoronoiKernel(float4* data, float4* bins, int width, int h, int nb_bins)
{
// Create one histogram per block. Make blocks do as much work as possible. 
// Follow by a histogram merge across blocks

// brute force appraoch
    unsigned int x = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    unsigned int y = __mul24(blockIdx.y,blockDim.y) + threadIdx.y;
	unsigned int indx = __mul24(y,width) + x;

	// transform (x,y) into a seed
	float4 s = data[indx];
	//int sx = s.x * sz3; // hardcoded sz3d
	//int gx = 
	//int sy = s.y * sz3; // hardcoded sz3d
	//int sz = s.z * sz3; // hardcoded sz3d
	//int seed = sx + sz3*(sy + sz*sz3);
	int seed = s.w;

	//unsigned int ix = data[indx];
	int ix = seed;
	//atomicAdd(bins+ix, 1);
	//float4 yy = make_float4(1,1,1,1);
	float4 bb = bins[ix];
	bb.x = bb.x + s.x;
	bb.y = bb.y + s.y;
	bb.z = bb.z + s.z;
	bb.w = bb.w + 1.;
	bins[ix] = bb;   // Without atomic, I will have synchronization problems
	//bins[ix] = bins[ix] + yy;
}
//----------------------------------------------------------------------
__global__ void
binKernel(int* data, int* bins, int width, int h, int nb_bins)
{
    unsigned int x = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    unsigned int y = __mul24(blockIdx.y,blockDim.y) + threadIdx.y;
	unsigned int indx = __mul24(y,width) + x;

	#if 0
	unsigned int ix = data[indx];
	atomicAdd(bins+ix, 1);
	//bins[ix] = bins[ix] + 1;
	#endif
}
//----------------------------------------------------------------------
#if 1
extern "C++"
void add(float* h, float* ho, int w, int he)
{
		addKernel<<<dim3(32,32), dim3(32,8)>>>(h, ho, w, he);
}
#endif
//----------------------------------------------------------------------
extern "C++"
void add(dim3& grid, dim3& block, float* h, float* ho, int w, int he)
{
		addKernel<<<grid, block>>>(h, ho, w, he);
}
//----------------------------------------------------------------------
extern "C++" void bins_call(dim3& grid , dim3& block, int* h, int* bins, int w, int he, int nb_bins) 
{
	binKernel<<<grid, block>>>(h, bins, w, he, nb_bins);
}
//----------------------------------------------------------------------
extern "C++" void bins_efficient_call(dim3& grid , dim3& block, int* h, int* bins, int w, int he, int nb_bins) 
{
	binEfficientKernel<<<grid, block>>>(h, bins, w, he, nb_bins);
}
//----------------------------------------------------------------------
extern "C++" void centroid_voronoi(dim3& grid , dim3& block, float4* h, float4* bins, int w, int he, 
    int nb_bins, int sz3d, std::vector<int4>& newSeeds, int edge) 
{
	edge = 16;
    const unsigned int shared_mem_size = sizeof(float4) * edge*edge; // square side: 2*edge
	printf(" shared mem size: %d\n", shared_mem_size);

	dim3 g(4,4,1);
	//dim3 b(edge/2, edge, 1);
	dim3 b(edge*edge/2, 1, 1);
	int nb_threads  = b.x * b.y * b.z;

	for (int i=0; i < newSeeds.size(); i++) {
		int4& s = newSeeds[i];
		printf("x,y,z,w= %d, %d, %d, %d\n", s.x, s.y, s.z, s.w);
	}
	exit(0);

	scan_workefficient<<<g, b, 2*shared_mem_size>>>(h, bins, nb_threads*2);
	//scan_workefficient<<<g, block>>>(h, bins, block.x*block.y*block.z);

	float4* h_h = (float4*) malloc(sizeof(float4)*nb_threads);
	float4* bins_h = (float4*) malloc(sizeof(float4)*nb_threads);

	hipMemcpy(h_h, h, nb_threads*sizeof(float4), hipMemcpyDeviceToHost);
	for (int i=0; i < 10; i++) {
		printf("h_h= %f, %f, %f, %f\n", h_h[i].x, h_h[i].y, h_h[i].z, h_h[i].w);
	}

	float tot = 0;
	for (int i=0; i < nb_threads; i++) {
		tot += h_h[i].w;
	}
	printf("total = %f\n", tot);


	printf("nb_threads= %d\n", nb_threads);
	printf("b size: %d\n", b.x*b.y*b.z);
	hipMemcpy(bins_h, bins, nb_threads*sizeof(float4), hipMemcpyDeviceToHost);
	//for (int j=0; j < nb_threads; j++) {
	for (int j=0; j < 1; j++) {
		printf("sum= %f, %f, %f, %f\n", bins_h[j].x, bins_h[j].y, bins_h[j].z, bins_h[j].w);
	}
	exit(0);
}
//----------------------------------------------------------------------
extern "C++" void centroid_voronoi_2(dim3& grid , dim3& block, float4* h, float4* bins, int w, int he, 
    int nb_bins, int sz3d, int4* newSeeds, int edge, int nbSeeds) 
{
	edge = 16; // tile 8 x 8 around seed: 0,1,2,3,4=seed,5,6,7  (seed-edge/2, seed+edge/2-1)

    const unsigned int shared_mem_size = sizeof(float4) * edge*edge + 4; // square side: 2*edge
	//shared_mem_size += 4;  // for   int4 seed
	printf(" shared mem size: %d\n", shared_mem_size);

	printf("nbSeeds= %d\n", nbSeeds);
	printf("w,he= %d, %d\n", w, he);

	//dim3 b(edge/2, edge, 1);
	dim3 b(edge, edge/2, 1); // two data elements per thread
	dim3 g(nbSeeds,1,1);
	int nb_threads  = b.x * b.y * b.z;

	printf("sizeof(float4)= %d\n", sizeof(float4)); // 16


	if (nb_threads*2 != edge*edge) {
		printf("error in nb_threads or edge\n");
		exit(0);
	}

	int width = w;
	printf("width= %d\n", width);

	float4* h_h = (float4*) malloc(sizeof(float4)*edge*edge);

	printf("nbSeeds= %d\n", nbSeeds);
	int4* newSeeds_d;
	CUDA_SAFE_CALL(hipMalloc((void**) &newSeeds_d, sizeof(int4)*nbSeeds));
	CUDA_SAFE_CALL(hipMemcpy(newSeeds_d, newSeeds, sizeof(int4)*nbSeeds, hipMemcpyHostToDevice));

// Assume that width = height (SHOULD BE MADE MORE GENERAL)
	//scan_workefficient_2<<<g, b, 2*shared_mem_size>>>(h, bins, newSeeds_d, nb_threads*2, edge, width);

	// More efficient scan (hopefully)
	scan_workefficient_3<<<g, b, 2*shared_mem_size>>>(h, bins, newSeeds_d, nb_threads*2, edge, width);
	//scan_test_incoherent<<<g, b, 2*shared_mem_size>>>(h, bins, newSeeds_d, nb_threads*2, edge, width);

#if 1

	float4* bins_h = (float4*) malloc(sizeof(float4)*nbSeeds);

	CUDA_SAFE_CALL(hipMemcpy(h_h, h, edge*edge*sizeof(float4), hipMemcpyDeviceToHost));
	for (int i=0; i < edge*edge; i++) {
		printf("h_h= %f, %f, %f, %f\n", h_h[i].x, h_h[i].y, h_h[i].z, h_h[i].w);
	}
	//return;

	float tot = 0;
	for (int i=0; i < nb_threads; i++) {
		tot += h_h[i].w;
	}
	printf("total = %f\n", tot);


	printf("nbSeeds= %d\n", nbSeeds);
	printf("nb_threads= %d\n", nb_threads);
	printf("b size: %d\n", b.x*b.y*b.z);

	hipMemcpy(bins_h, bins, nbSeeds*sizeof(float4), hipMemcpyDeviceToHost);
	int count = 0;
	for (int j=0; j < nbSeeds; j++) { //}
		//printf("(%d) sum= %f, %f, %f, %f\n", j, bins_h[j].x, bins_h[j].y, bins_h[j].z, bins_h[j].w);
		count += (int) bins_h[j].w;
	}
	printf("total count= %d\n", count);
#endif
}
//----------------------------------------------------------------------
extern "C++" void centroid_voronoi_4(dim3& grid , dim3& block, float4* h, float4* bins, int w, int he, 
    int nb_bins, int sz3d, int4* newSeeds, int edge, int nbSeeds) 
{
	edge = 16; // tile 8 x 8 around seed: 0,1,2,3,4=seed,5,6,7  (seed-edge/2, seed+edge/2-1)

	unsigned int num_elements = edge*edge;
    unsigned int extra_space = num_elements / NUM_BANKS;

#ifdef ZERO_BANK_CONFLICTS
    extra_space += extra_space / NUM_BANKS;
#endif

    const unsigned int shared_mem_size = sizeof(float4)*num_elements + extra_space + 4; // in bytes


	printf(" shared mem size: %d\n", shared_mem_size);

	printf("nbSeeds= %d\n", nbSeeds);
	printf("w,he= %d, %d\n", w, he);

	//dim3 b(edge/2, edge, 1);
	dim3 b(edge, edge, 1); // two data elements per thread
	dim3 g(nbSeeds,1,1);
	int nb_threads  = b.x * b.y * b.z;

	printf("sizeof(float4)= %d\n", sizeof(float4)); // 16


	if (nb_threads != edge*edge) {
		printf("error in nb_threads or edge\n");
		exit(0);
	}

	int width = w;
	printf("width= %d\n", width);

	float4* h_h = (float4*) malloc(sizeof(float4)*edge*edge);

	printf("nbSeeds= %d\n", nbSeeds);
	int4* newSeeds_d;
	CUDA_SAFE_CALL(hipMalloc((void**) &newSeeds_d, sizeof(int4)*nbSeeds));
	CUDA_SAFE_CALL(hipMemcpy(newSeeds_d, newSeeds, sizeof(int4)*nbSeeds, hipMemcpyHostToDevice));

// Assume that width = height (SHOULD BE MADE MORE GENERAL)

	scan_workefficient_4<<<g, b, shared_mem_size>>>(h, bins, newSeeds_d, nb_threads*2, edge, width);

#if 1

	float4* bins_h = (float4*) malloc(sizeof(float4)*nbSeeds);

	CUDA_SAFE_CALL(hipMemcpy(h_h, h, edge*edge*sizeof(float4), hipMemcpyDeviceToHost));
	//for (int i=0; i < edge*edge; i++) {
		//printf("h_h= %f, %f, %f, %f\n", h_h[i].x, h_h[i].y, h_h[i].z, h_h[i].w);
	//}
	//return;

	float tot = 0;
	for (int i=0; i < nb_threads; i++) {
		tot += h_h[i].w;
	}
	printf("total = %f\n", tot);


	printf("nbSeeds= %d\n", nbSeeds);
	printf("nb_threads= %d\n", nb_threads);
	printf("b size: %d\n", b.x*b.y*b.z);

	hipMemcpy(bins_h, bins, nbSeeds*sizeof(float4), hipMemcpyDeviceToHost);
	int count = 0;
	for (int j=0; j < nbSeeds; j++) { 
		//printf("(%d) sum= %f, %f, %f, %f\n", j, bins_h[j].x, bins_h[j].y, bins_h[j].z, bins_h[j].w);
		count += (int) bins_h[j].w;
	}
	printf("total count= %d\n", count);
#endif
}
//----------------------------------------------------------------------
extern "C++" void centroid_voronoi_5(dim3& grid , dim3& block, float4* h, float4* bins, int w, int he, 
    int nb_bins, int sz3d, int4* newSeeds, int edge, int nbSeeds) 
{
// Objective: remove all coalescing and bank conflicts
	edge = 16; // tile 8 x 8 around seed: 0,1,2,3,4=seed,5,6,7  (seed-edge/2, seed+edge/2-1)

	unsigned int num_elements = edge*edge;
    unsigned int extra_space = num_elements / NUM_BANKS;

#ifdef ZERO_BANK_CONFLICTS
    extra_space += extra_space / NUM_BANKS;
#endif

    const unsigned int shared_mem_size = sizeof(float4)*(num_elements + extra_space + 1); // in bytes


	printf(" shared mem size: %d\n", shared_mem_size);

	printf("nbSeeds= %d\n", nbSeeds);
	printf("w,he= %d, %d\n", w, he);

	dim3 b(4*edge, 1, 1); // 64 threads per block: 2 warps. Later, change to 128 threads. 
	//dim3 b(edge, edge, 1); // two data elements per thread
	dim3 g(nbSeeds,1,1);
	int nb_threads  = b.x * b.y * b.z;
	printf("block: %d, %d, %d\n", b.x, b.y, b.z);

	printf("sizeof(float4)= %d\n", sizeof(float4)); // 16

	if (nb_threads != 4*edge) {
		printf("error in nb_threads or edge\n");
		exit(0);
	}

	int width = w;
	printf("width= %d\n", width);

	float4* h_h = (float4*) malloc(sizeof(float4)*edge*edge);

	printf("nbSeeds= %d\n", nbSeeds);
	int4* newSeeds_d;
	CUDA_SAFE_CALL(hipMalloc((void**) &newSeeds_d, sizeof(int4)*nbSeeds));
	CUDA_SAFE_CALL(hipMemcpy(newSeeds_d, newSeeds, sizeof(int4)*nbSeeds, hipMemcpyHostToDevice));

// Assume that width = height (SHOULD BE MADE MORE GENERAL)

	for (int i=0; i < 10; i++) {
		printf("newSeeds[%d]= %d, %d, %d, %d\n", i, newSeeds[i].x, newSeeds[i].y, newSeeds[i].z, newSeeds[i].w);
	}

	scan_workefficient_5<<<g, b, shared_mem_size>>>(h, bins, newSeeds_d, nb_threads, edge, width);
	//scan_workefficient_6<<<g, b, shared_mem_size>>>(h, bins, newSeeds_d, nb_threads, edge, width);
	//exit(0);

#if 1

	float4* bins_h = (float4*) malloc(sizeof(float4)*nbSeeds);

	CUDA_SAFE_CALL(hipMemcpy(h_h, h, edge*edge*sizeof(float4), hipMemcpyDeviceToHost));
	for (int i=0; i < 10; i++) {
		printf("h_h= %f, %f, %f, %f\n", h_h[i].x, h_h[i].y, h_h[i].z, h_h[i].w);
	}

	printf("nbSeeds= %d\n", nbSeeds);
	printf("nb_threads= %d\n", nb_threads);
	printf("b size: %d\n", b.x*b.y*b.z);

	hipMemcpy(bins_h, bins, nbSeeds*sizeof(float4), hipMemcpyDeviceToHost);
	int count = 0;
	for (int j=0; j < nbSeeds; j++) { 
		//if (j < 5) printf("(%d) sum= %f, %f, %f, %f\n", j, bins_h[j].x, bins_h[j].y, bins_h[j].z, bins_h[j].w);
		printf("(%d) sum= %f, %f, %f, %f\n", j, bins_h[j].x, bins_h[j].y, bins_h[j].z, bins_h[j].w);
		count += (int) bins_h[j].w;
	}
	printf("total count= %d\n", count);
#endif
}
//----------------------------------------------------------------------
extern "C++" float4* centroid_voronoi_7(float4* h, float4* bins, int w, int he, 
    int nb_bins, int sz3d, int4* newSeeds, int edge, int nbSeeds) 
// return 
{
// Objective: remove all coalescing and bank conflicts
	edge = 16; // tile 8 x 8 around seed: 0,1,2,3,4=seed,5,6,7  (seed-edge/2, seed+edge/2-1)

	unsigned int num_elements = edge*edge;
    unsigned int extra_space = num_elements / NUM_BANKS;

#ifdef ZERO_BANK_CONFLICTS
    extra_space += extra_space / NUM_BANKS;
#endif

    const unsigned int shared_mem_size = sizeof(float4)*(num_elements + extra_space + 1); // in bytes


#undef SCAN7

#ifdef SCAN7
	dim3 b(4*edge, 4, 1); // 256=4*64 threads per block: 2 warps. Later, change to 128 threads. 
#else
	dim3 b(edge, edge, 1); // 256=4*64 threads per block: 2 warps. Later, change to 128 threads. 
#endif

	//dim3 b(edge, edge, 1); // two data elements per thread
	dim3 g(nbSeeds,1,1);
	int nb_threads  = b.x * b.y * b.z;


	if (nb_threads != 4*4*edge) {
		printf("error in nb_threads or edge\n");
		exit(0);
	}

	int width = w;

	float4* h_h = (float4*) malloc(sizeof(float4)*edge*edge);

	int4* newSeeds_d;
	CUDA_SAFE_CALL(hipMalloc((void**) &newSeeds_d, sizeof(int4)*nbSeeds));
	CUDA_SAFE_CALL(hipMemcpy(newSeeds_d, newSeeds, sizeof(int4)*nbSeeds, hipMemcpyHostToDevice));

// Assume that width = height (SHOULD BE MADE MORE GENERAL)

	for (int i=0; i < 10; i++) {
		printf("newSeeds[%d]= %d, %d, %d, %d\n", i, newSeeds[i].x, newSeeds[i].y, newSeeds[i].z, newSeeds[i].w);
	}

	//scan_workefficient_7<<<g, b, shared_mem_size>>>(h, bins, newSeeds_d, nb_threads, edge, width);
	scan_workefficient_8<<<g, b, shared_mem_size>>>(h, bins, newSeeds_d, nb_threads, edge, width);
	//scan_workefficient_8_larger<<<g, b, shared_mem_size>>>(h, bins, newSeeds_d, nb_threads, edge, width);

	float4* bins_h = (float4*) malloc(sizeof(float4)*nbSeeds);
	CUDA_SAFE_CALL(hipMemcpy(bins_h, bins, nbSeeds*sizeof(float4), hipMemcpyDeviceToHost));


#if 1
	printf("nbSeeds= %d\n", nbSeeds);
	printf("nb_threads= %d\n", nb_threads);
	printf("b size: %d\n", b.x*b.y*b.z);

	printf("--- GPU centroid histogram inside add_one_kernel.cu ----\n");
	int count = 0;
	for (int j=0; j < nbSeeds; j++) { 
		if (j < 20) printf("(%d) sum= %f, %f, %f, %f\n", j, bins_h[j].x, bins_h[j].y, bins_h[j].z, bins_h[j].w);
		//printf("(%d) sum= %f, %f, %f, %f\n", j, bins_h[j].x, bins_h[j].y, bins_h[j].z, bins_h[j].w);
		count += (int) bins_h[j].w;
	}
	printf("add_one: total count= %d\n", count);
#endif

	return bins_h;
}
//----------------------------------------------------------------------

#endif
