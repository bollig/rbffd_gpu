// THIS IS adapted from verbose_monitor.cu
// PROVIDED BY THE CUSP v0.1 EXAMPLES

#include "grids/grid_reader.h"
#include "rbffd/rbffd.h"
#include "timer_eb.h" 

#include <cusp/hyb_matrix.h>
#include <cusp/ell_matrix.h>
#include <cusp/csr_matrix.h>
#include <cusp/coo_matrix.h>
#include <cusp/monitor.h>
#include <cusp/krylov/cg.h>
#include <cusp/krylov/gmres.h>
#include <cusp/gallery/poisson.h>
#include <cusp/print.h>
#include <cusp/array2d.h>
#include <cusp/multiply.h>
#include <cusp/blas.h>


#include <thrust/host_vector.h> 
#include <thrust/device_vector.h>
#include <thrust/generate.h>

#include <iostream>
#include <sstream> 
using namespace std;

// TODO: 
// Sort CSR, ELL, HYB by column. (use std::pair<unsigned int, unsigned int>
// (sten[j], j) and sort on sten[j]. Then use the sorted j's to index sten[]
// and lapl[]

EB::TimerList tm;

template <typename MatT>
void benchmarkMultiplyHost(MatT& A) {
    // If we multiply a vector of 1s we should see our result equal 0 (if our
    // RBF-FD weights are good)
    cusp::array1d<double, cusp::host_memory> x(A.num_rows, 1);
#if 0
    // generate random data on the host to make sure when we multiply we
    // actually replace b
    thrust::host_vector<double> h_vec(A.num_rows);
    thrust::generate(h_vec.begin(), h_vec.end(), rand);

    // transfer to device and compute sum
    cusp::array1d<double, cusp::host_memory> b = h_vec;
    x = h_vec;
#else 
    cusp::array1d<double, cusp::host_memory> b(A.num_rows, 1);
#endif 
    cusp::multiply(A, x, b); 


    std::cout << "l1   Norm: " << cusp::blas::nrm1(b) << std::endl;  
    std::cout << "l2   Norm: " << cusp::blas::nrm2(b) << std::endl;  
    std::cout << "linf Norm: " << cusp::blas::nrmmax(b) << std::endl;  

#if 0
    cusp::array1d<double, cusp::host_memory> b_host = b;
    std::cout << "l1   Norm: " << cusp::blas::nrm1(b_host) << std::endl;  
    std::cout << "l2   Norm: " << cusp::blas::nrm2(b_host) << std::endl;  
    std::cout << "linf Norm: " << cusp::blas::nrmmax(b_host) << std::endl;  
#endif 
#if 0
    cusp::print(b); 
#endif 
}

template <typename MatT>
void benchmarkMultiplyDevice(MatT& A) {
    // If we multiply a vector of 1s we should see our result equal 0 (if our
    // RBF-FD weights are good)
    cusp::array1d<double, cusp::device_memory> x(A.num_rows, 1);
#if 0
    // generate random data on the host to make sure when we multiply we
    // actually replace b
    thrust::host_vector<double> h_vec(A.num_rows);
    thrust::generate(h_vec.begin(), h_vec.end(), rand);

    // transfer to device and compute sum
    cusp::array1d<double, cusp::device_memory> b = h_vec;
    x = h_vec;
#else 
    cusp::array1d<double, cusp::device_memory> b(A.num_rows, 1);
#endif 
    cusp::multiply(A, x, b); 

    hipDeviceSynchronize();
#if 1
    std::cout << "l1   Norm: " << cusp::blas::nrm1(b) << std::endl;  
    std::cout << "l2   Norm: " << cusp::blas::nrm2(b) << std::endl;  
    std::cout << "linf Norm: " << cusp::blas::nrmmax(b) << std::endl;  
#endif 

#if 0
    cusp::print(b); 
#endif 
}

void test_COO ( RBFFD& der, Grid& grid, int platform) {
    typedef cusp::coo_matrix<int, double, cusp::host_memory> MatType; 
    typedef cusp::coo_matrix<int, double, cusp::device_memory> MatTypeGPU; 

    char *matString = "COO"; 
    char platformString[4]; 
    if (platform) {
        sprintf(platformString, "GPU"); 
    } else {
        sprintf(platformString, "CPU"); 
    }

    unsigned int N = grid.getNodeListSize(); 
    unsigned int n = grid.getMaxStencilSize(); 

    char assemble_timer_name[256]; 
    char copy_timer_name[256]; 
    char multiply_timer_name[256]; 

    sprintf(assemble_timer_name, "%u %s %s Assemble", N, matString, platformString); 
    sprintf(copy_timer_name,     "%u %s %s Send2Dev", N, matString, platformString); 
    sprintf(multiply_timer_name, "%u %s %s Multiply", N, matString, platformString); 

    if (!tm.contains(assemble_timer_name)) {
        tm[assemble_timer_name] = new EB::Timer(assemble_timer_name);  
        tm[copy_timer_name] = new EB::Timer(copy_timer_name);  
        tm[multiply_timer_name] = new EB::Timer(multiply_timer_name);
    }
    std::cout << "WORKING ON: " << assemble_timer_name << std::endl;
    tm[assemble_timer_name]->start();

    MatType A( N , N , N*n ); 

    unsigned int ind = 0; 
    for (int i = 0; i < A.num_rows; i++) {
        StencilType& sten = grid.getStencil(i); 
        double* lapl = der.getStencilWeights(RBFFD::LSFC, i); 

        // Off diagonals
        for (unsigned int j = 0; j < n; j++) {
            A.row_indices[ind] =  i; 
            A.column_indices[ind] =  sten[j]; 
            A.values[ind] = -lapl[j]; 
            ind++; 
        }
    }
    tm[assemble_timer_name]->stop();
#if 0
    std::cout << "N = " << N << "\t n = " << n << std::endl;
    cusp::array2d<double, cusp::host_memory> A_full(A); 
    cusp::print(A_full); 
    cusp::print(A); 
#endif
    std::cout << "\t\t\tMultiply\n";

    if (platform) {
        tm[copy_timer_name]->start();
        MatTypeGPU A_gpu(A); 
        tm[copy_timer_name]->stop();
        tm[multiply_timer_name]->start();
        benchmarkMultiplyDevice<MatTypeGPU>(A_gpu); 
    } else { 
        tm[multiply_timer_name]->start();
        benchmarkMultiplyHost<MatType>(A); 
    }
    tm[multiply_timer_name]->stop();
}

void test_CSR ( RBFFD& der, Grid& grid, int platform) {
    typedef cusp::csr_matrix<int, double, cusp::host_memory> MatType; 
    typedef cusp::csr_matrix<int, double, cusp::device_memory> MatTypeGPU; 

    unsigned int N = grid.getNodeListSize(); 
    unsigned int n = grid.getMaxStencilSize(); 

    char *matString = "CSR"; 
    char platformString[4]; 
    if (platform) {
        sprintf(platformString, "GPU"); 
    } else {
        sprintf(platformString, "CPU"); 
    }

    char assemble_timer_name[256]; 
    char copy_timer_name[256]; 
    char multiply_timer_name[256]; 

    sprintf(assemble_timer_name, "%u %s %s Assemble", N, matString, platformString); 
    sprintf(copy_timer_name, "%u %s %s Send2Dev", N, matString, platformString); 
    sprintf(multiply_timer_name, "%u %s %s Multiply", N, matString, platformString); 

    if (!tm.contains(assemble_timer_name)) {
        tm[assemble_timer_name] = new EB::Timer(assemble_timer_name);  
        tm[copy_timer_name] = new EB::Timer(copy_timer_name);  
        tm[multiply_timer_name] = new EB::Timer(multiply_timer_name);
    }
    std::cout << "WORKING ON: " << assemble_timer_name << std::endl;
    tm[assemble_timer_name]->start();

    MatType A( N , N , N*n ); 

    unsigned int ind = 0; 
    for (int i = 0; i < A.num_rows; i++) {
        StencilType& sten = grid.getStencil(i); 
        // std::vector<unsigned int> sort_ind = grid.getSortedStencilInd(i); 
        double* lapl = der.getStencilWeights(RBFFD::LSFC, i); 

        A.row_offsets[i] = ind;

        // Off diagonals
        for (unsigned int j = 0; j < n; j++) {
            A.column_indices[ind] =  sten[j]; 
            A.values[ind] = -lapl[j]; 
            ind++; 
        }
    }
    A.row_offsets[A.num_rows] = ind; 
    tm[assemble_timer_name]->stop();

    std::cout << "\t\t\tMultiply\n"; 
#if 0
    std::cout << "N = " << N << "\t n = " << n << std::endl;
    cusp::array2d<double, cusp::host_memory> A_full(A); 
    cusp::print(A_full); 
    cusp::print(A); 
#endif 

    if (platform) {
        tm[copy_timer_name]->start();
        MatTypeGPU A_gpu(A); 
        tm[copy_timer_name]->stop();
        tm[multiply_timer_name]->start();
        benchmarkMultiplyDevice<MatTypeGPU>(A_gpu); 
    } else { 
        tm[multiply_timer_name]->start();
        benchmarkMultiplyHost<MatType>(A); 
    }
    tm[multiply_timer_name]->stop();
}


void test_ELL ( RBFFD& der, Grid& grid, int platform) {
    typedef cusp::ell_matrix<int, double, cusp::host_memory> MatType; 
    typedef cusp::ell_matrix<int, double, cusp::device_memory> MatTypeGPU; 

    unsigned int N = grid.getNodeListSize(); 
    unsigned int n = grid.getMaxStencilSize(); 

    char *matString = "ELL"; 
    char platformString[4]; 
    if (platform) {
        sprintf(platformString, "GPU"); 
    } else {
        sprintf(platformString, "CPU"); 
    }

    char assemble_timer_name[256]; 
    char copy_timer_name[256]; 
    char multiply_timer_name[256]; 

    sprintf(assemble_timer_name, "%u %s %s Assemble", N, matString, platformString); 
    sprintf(copy_timer_name, "%u %s %s Send2Dev", N, matString, platformString); 
    sprintf(multiply_timer_name, "%u %s %s Multiply", N, matString, platformString); 

    if (!tm.contains(assemble_timer_name)) {
        tm[assemble_timer_name] = new EB::Timer(assemble_timer_name);  
        tm[copy_timer_name] = new EB::Timer(copy_timer_name);  
        tm[multiply_timer_name] = new EB::Timer(multiply_timer_name);
    }
    std::cout << "WORKING ON: " << assemble_timer_name << std::endl;

    tm[assemble_timer_name]->start();

    // Allocate a (N,N) matrix with (N*n) total nonzeros and at most (n) nonzero per row
    MatType A( N , N , N*n , n ); 

    for (int i = 0; i < A.num_rows; i++) {
        StencilType& sten = grid.getStencil(i); 
        // std::vector<unsigned int> sort_ind = grid.getSortedStencilInd(i); 
        double* lapl = der.getStencilWeights(RBFFD::LSFC, i); 

        // Off diagonals
        for (unsigned int j = 0; j < n; j++) {
            A.column_indices(i, j) =  sten[j]; 
            A.values(i, j) = -lapl[j]; 
        }
    }
    tm[assemble_timer_name]->stop();
#if 0
    std::cout << "N = " << N << "\t n = " << n << std::endl;
    cusp::array2d<double, cusp::host_memory> A_full(A); 
    cusp::print(A_full); 
    cusp::print(A); 
#endif 
    std::cout << "\t\t\tMultiply\n";
    if (platform) {
        tm[copy_timer_name]->start();
        MatTypeGPU A_gpu(A); 
        tm[copy_timer_name]->stop();
        tm[multiply_timer_name]->start();
        benchmarkMultiplyDevice<MatTypeGPU>(A_gpu); 
    } else { 
        tm[multiply_timer_name]->start();
        benchmarkMultiplyHost<MatType>(A); 
    }
    tm[multiply_timer_name]->stop();
}

void test_HYB ( RBFFD& der, Grid& grid, int platform) {

    // The HYB format has both an ELL (where ALL rows have n nonzeros) and a
    // COO (surplus nonzeros per row. In our case we know we will ALWAYS have n
    // nonzeros for stencil weights per row, unless a weight computes to 0.
    // This means HYB is equivalent to ELL for us. If we convert from ELL to
    // HYB we *might* see a performance boost if their constructor is smart
    // enough to check for 0's, but I doubt it. We will still fill a HYB matrix
    // and test performance. perhaps there are other efficiency differences
    // between the two formats. 

    typedef cusp::hyb_matrix<int, double, cusp::host_memory> MatType; 
    typedef cusp::hyb_matrix<int, double, cusp::device_memory> MatTypeGPU; 

    unsigned int N = grid.getNodeListSize(); 
    unsigned int n = grid.getMaxStencilSize(); 

    char *matString = "HYB"; 
    char platformString[4]; 
    if (platform) {
        sprintf(platformString, "GPU"); 
    } else {
        sprintf(platformString, "CPU"); 
    }

    char assemble_timer_name[256]; 
    char copy_timer_name[256]; 
    char multiply_timer_name[256]; 

    sprintf(assemble_timer_name, "%u %s %s Assemble", N, matString, platformString); 
    sprintf(copy_timer_name, "%u %s %s Send2Dev", N, matString, platformString); 
    sprintf(multiply_timer_name, "%u %s %s Multiply", N, matString, platformString); 

    if (!tm.contains(assemble_timer_name)) {
        tm[assemble_timer_name] = new EB::Timer(assemble_timer_name);  
        tm[copy_timer_name] = new EB::Timer(copy_timer_name);  
        tm[multiply_timer_name] = new EB::Timer(multiply_timer_name);
    }
    std::cout << "WORKING ON: " << assemble_timer_name << std::endl;

    tm[assemble_timer_name]->start();
    // Allocate a (N,N) matrix with (N*n) total nonzeros and at most (n) nonzero per row
    // and 0 extra non-zeros per row
    MatType A( N , N , N*n , 0 , n ); 

    for (int i = 0; i < A.num_rows; i++) {
        StencilType& sten = grid.getStencil(i); 
        // std::vector<unsigned int> sort_ind = grid.getSortedStencilInd(i); 
        double* lapl = der.getStencilWeights(RBFFD::LSFC, i); 

        // Off diagonals
        for (unsigned int j = 0; j < n; j++) {
            A.ell.column_indices(i, j) =  sten[j]; 
            A.ell.values(i, j) = -lapl[j]; 
            // A.coo.row_indices[ind] = 0; ...
        }
    }
    tm[assemble_timer_name]->stop();
#if 0
    std::cout << "N = " << N << "\t n = " << n << std::endl;
    cusp::array2d<double, cusp::host_memory> A_full(A); 
    cusp::print(A_full); 
    cusp::print(A); 
#endif 
    std::cout << "\t\t\tMultiply\n";

    if (platform) {
        tm[copy_timer_name]->start();
        MatTypeGPU A_gpu(A); 
        tm[copy_timer_name]->stop();
        tm[multiply_timer_name]->start();
        benchmarkMultiplyDevice<MatTypeGPU>(A_gpu); 
    } else { 
        tm[multiply_timer_name]->start();
        benchmarkMultiplyHost<MatType>(A); 
    } 
    tm[multiply_timer_name]->stop();
}



void testSPMV(int MAT_TYPE, int PLATFORM, RBFFD& der, Grid& grid) { 

    switch (MAT_TYPE) {
        case 0:  
            test_COO(der, grid, PLATFORM); 
            break; 
        case 1: 
            test_CSR(der, grid, PLATFORM); 
            break; 
        case 2: 
            test_ELL(der, grid, PLATFORM); 
            break; 
        case 3: 
            test_HYB(der, grid, PLATFORM); 
            break; 
        default: 
            std::cout << "INVALID SPMV TYPE\n"; 
            break;  
    }
}


int main(void)
{
    bool writeIntermediate = true; 

    std::vector<std::string> grids; 
#if 1
    grids.push_back("~/GRIDS/md/md005.00036"); 
#if 1
    grids.push_back("~/GRIDS/md/md031.01024"); 
    grids.push_back("~/GRIDS/md/md050.02601"); 
    grids.push_back("~/GRIDS/md/md063.04096"); 
    grids.push_back("~/GRIDS/md/md089.08100"); 
    grids.push_back("~/GRIDS/md/md127.16384"); 
    grids.push_back("~/GRIDS/md/md165.27556"); 
#endif 
#else

//    grids.push_back("~/GRIDS/geoff/scvtmesh_100k_nodes.ascii"); 
    grids.push_back("~/GRIDS/geoff/scvtmesh_500k_nodes.ascii"); 
    grids.push_back("~/GRIDS/geoff/scvtmesh_1m_nodes.ascii"); 
#endif 

    for (size_t i = 0; i < grids.size(); i++) {
        std::string& grid_name = grids[i]; 

        std::string weight_timer_name = grid_name + " Calc Weights";  

        tm[weight_timer_name] = new EB::Timer(weight_timer_name.c_str()); 


        // Get contours from rbfzone.blogspot.com to choose eps_c1 and eps_c2 based on stencil_size (n)
        #if 0
        unsigned int stencil_size = 40;
        double eps_c1 = 0.027;
        double eps_c2 = 0.274;
#else 
        unsigned int stencil_size = 50;
        double eps_c1 = 0.027;
        double eps_c2 = 0.274;
#endif 

        GridReader* grid = new GridReader(grid_name, 4); 
        grid->setMaxStencilSize(stencil_size); 
        // We do not read until generate is called: 

        Grid::GridLoadErrType err = grid->loadFromFile(); 
        if (err == Grid::NO_GRID_FILES) 
        {
            grid->generate();
            if (writeIntermediate) {
                grid->writeToFile(); 
            }
        } 
        std::cout << "Generate Stencils\n";
        Grid::GridLoadErrType st_err = grid->loadStencilsFromFile(); 
        if (st_err == Grid::NO_STENCIL_FILES) {
            //            grid->generateStencils(Grid::ST_BRUTE_FORCE);   
            //grid->generateStencils(Grid::ST_KDTREE);   
            grid->setNSHashDims(50, 50,50);  
            grid->generateStencils(Grid::ST_HASH);   
            if (writeIntermediate) {
                grid->writeToFile(); 
            }
        }


        std::cout << "Generate RBFFD Weights\n"; 
        tm[weight_timer_name]->start(); 
        RBFFD der(RBFFD::LSFC | RBFFD::XSFC | RBFFD::YSFC | RBFFD::ZSFC, grid, 3, 0); 
        der.setEpsilonByParameters(eps_c1, eps_c2);
        int der_err = der.loadAllWeightsFromFile(); 
        if (der_err) {
            der.computeAllWeightsForAllStencils(); 

            tm[weight_timer_name]->start(); 
            if (writeIntermediate) {
                der.writeAllWeightsToFile(); 
            }
        }

        cout << "Allocating device memory\n" << std::endl;
#if 0
        for (int k = 0; k < 5; k++) 
#endif 
        {
            for (int j = 0; j < 4; j++) 
            {
                // CPU: 
                testSPMV(j, 0, der, *grid); 
                // GPU: 
                testSPMV(j, 1, der, *grid); 
            }
        }

        delete(grid); 
    }
#if 0
    exit(-1);
    // Convert the 2D memory block to a sparse representation
    cusp::coo_matrix<int, double, cusp::device_memory> A_dev(A);

    // allocate storage for solution (x) and right hand side (b)
    cusp::array1d<double, cusp::device_memory> x_dev(A.num_rows, 0);
    cusp::array1d<double, cusp::device_memory> b_dev(A.num_rows, 1);

    // set stopping criteria:
    //  iteration_limit    = 100
    //  relative_tolerance = 1e-8
    cusp::verbose_monitor<double> monitor(b, 100, 1e-8);

    cout << "Starting GMRES\n" << std::endl;

    // solve the linear system A x = b
    cusp::krylov::gmres(A_dev, x_dev, b_dev, 30, monitor);

    cout << "GMRES complete\n" << std::endl;

    // monitor will report solver progress and results
#endif 
    tm.printAll();
    tm.writeToFile();
    return EXIT_SUCCESS;
}

