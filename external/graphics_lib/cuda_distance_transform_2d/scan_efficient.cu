#include "hip/hip_runtime.h"

#ifndef _SCAN_WORKEFFICIENT_KERNEL_H_
#define _SCAN_WORKEFFICIENT_KERNEL_H_
#
#include "local_macros.h"

#define TIDX (__mul24(blockIdx.x,blockDim.x) + threadIdx.x)
#define TIDY (__mul24(blockIdx.y,blockDim.y) + threadIdx.y)
#define TWIDTH  (__mul24(gridDim.x,blockDim.x))
#define THEIGHT (__mul24(gridDim.y,blockDim.y))
#define ArrayID (TIDY*TWIDTH+TIDX)
#define MAKE_FLOAT4(arg) make_float4((arg), (arg), (arg), (arg))
#define MAKE_INT4(arg) make_int4((arg).x, (arg).y, (arg).z, (arg).w);

// Written by NVidia
// Modified by Gordon Erlebacher, Feb. 21, 2008

//----------------------------------------------------------------------
__global__ void scan_workefficient_2(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width)
{
    // Dynamically allocated shared memory for scan kernels
#if 1
    extern  __shared__  float4 temp[];
	float4 zero = make_float4(0.,0.,0.,0.);

	//if (blockIdx.x != 2) return;

	int numThreads = blockDim.x * blockDim.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int thid = threadIdx.x + blockDim.x * threadIdx.y;
	temp[2*thid] = zero;
	temp[2*thid+1] = zero;


	int blockId = blockIdx.x;
	int4& seed = *(seeds+blockId);

	// compute 2D flat texture coordinate from 3D seed coordinate
	int x = seed.x; 
	int y = seed.y;

	// edge should be part of the seed datastructure (per block)
	int edge2 = edge >> 1;
	int xorig = x - edge2;
	int yorig = y - edge2;
	int xid  = xorig + threadIdx.x; // 2 elements per thread
	int yid1 = yorig + threadIdx.y;
	int yid2 = yorig + edge - 1 - threadIdx.y;
#endif

	int flag  = 1;
	int flag1 = 1;
	int flag2 = 1;


	int WW = width;  // array width (argument)
	if (xid < 0 || xid >= WW) flag = 0;
	int HH = WW; // height of flat texture // MUST READ AS ARGUMENT
	if (yid1 < 0 || yid1 >= HH) flag1 = 0;
	if (yid2 < 0 || yid2 >= HH) flag2 = 0;

	int arrayid1 = xid + yid1 * WW;
	int arrayid2 = xid + yid2 * WW;

    __syncthreads();


	// the data can be in arbitrary order in the shared array

	float4 f;
	//flag  = 1;
	//flag1 = 1;
	//flag2 = 1;

	if (flag == 1 && flag1 == 1) {
		f = g_idata[arrayid1];
	
		if (int(f.w) == seed.w)
		{
			f.x = xid;
			f.y = yid1;
			f.w = 1.;
			temp[2*thid] = f;
		}
	}

	if (flag == 1 && flag2 == 1) {
		f = g_idata[arrayid2];
		if (int(f.w) == seed.w)
		{
			f.x = xid;
			f.y = yid2;
			f.w = 1.;
			temp[2*thid+1] = f;
		}
	}

#if 0
	__syncthreads();
	g_idata[2*thid] = temp[2*thid];
	g_idata[2*thid+1] = temp[2*thid+1];
	//g_idata[2*thid] = make_float4(arrayid1, arrayid2, 1,1);
	//g_idata[2*thid+1] = make_float4(xid,yid1,arrayid1,WW); // ok
	//g_idata[2*thid+1] = make_float4(f.w,seed.w,0,0);
	//g_idata[2*thid] = (g_idata[arrayid1]);
	//g_idata[2*thid+1] = (g_idata[arrayid2]);
	//g_idata[2*thid] = make_float4(seed.x,seed.y,seed.z,seed.w);
	//g_idata[2*thid+1] = make_float4(seed.x,seed.y,seed.z,seed.w);
	return;
#endif

    int offset = 1;

#if 1

// xorig - edge/2, xorig + edge/2 - 1

#if 1
	#if 1
    // build the sum in place up the tree
    for (int d = n>>1; d > 0; d >>= 1)
    {
        __syncthreads();

        if (thid < d)      
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;

            temp[bi].x += temp[ai].x;
            temp[bi].y += temp[ai].y;
            temp[bi].z += temp[ai].z;
            temp[bi].w += temp[ai].w;
        }

        offset <<= 1;
    }
	#endif
	#
	// Something wrong with the results

    // write results to global memory
    __syncthreads();
	if (thid == (numThreads-1)) {
		float nbs = temp[n-1].w;
		float nbs1 = 1./(nbs*width);
		if (nbs == 0) nbs = 1.;
		sum[blockId] = make_float4(temp[n-1].x*nbs1, temp[n-1].y*nbs1, 0., nbs); //, nbs);
	}
#endif
#endif
}
//----------------------------------------------------------------------
// More efficient version of scan_workefficient_2 (more threads + remove non-coalesced reads)
__global__ void scan_workefficient_3(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width)
{
    // Dynamically allocated shared memory for scan kernels
    extern  __shared__  float4 temp[];
	float* tempf = (float*) temp;

	//if (blockIdx.x != 2) return;

//	float* g_idata_f = (float*) g_idata;
	//float f1 = g_idata_f[0];

	int numThreads = blockDim.x * blockDim.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int thid = threadIdx.x + blockDim.x * threadIdx.y;

	// get data from global memory (should be coalesced)
	int thid2 = thid<<1;

	int blockId = blockIdx.x;

	int4 seed;

	if (thid == 0) {
		seed = *(seeds+blockId);
		temp[numThreads*2+1] = make_float4(seed.x,seed.y,seed.z,seed.w);
	}
	__syncthreads();
	seed = MAKE_INT4(temp[numThreads*2+1]);
	//int4 seed = make_int4(100,40,0,5);

	// compute 2D flat texture coordinate from 3D seed coordinate
	int x = seed.x; 
	int y = seed.y;

	// edge should be part of the seed datastructure (per block)
	int edge2 = edge >> 1;
	int xorig = x - edge2;
	int yorig = y - edge2;

	int flag1;

	int xid  = xorig + threadIdx.x; // 2 elements per thread
	int WW = width;  // array width (argument)
	int HH = WW; // height of flat texture // MUST READ AS ARGUMENT

	//--------------------
	for (int j=0; j < 1; j++) { // the loop added 2 registers (could be unrolled)
		__syncthreads();

		flag1 = 1;

		// need for each of the strings separately
		if (xid < 0 || xid >= WW) {
			flag1 = 0; 
		}

		temp[thid+j*numThreads] = g_idata[j*numThreads+thid];
		temp[thid+j*numThreads].w = 1.;

		int yid1 = yorig + threadIdx.y + j*numThreads;
		if (yid1 < 0 || yid1 >= HH) flag1 = 0;

 		int arrayid1 = xid + yid1 * WW;
    	__syncthreads();

	// the data can be in arbitrary order in the shared array

	//    CREATES uncoalesced  loads (HOW POSSIBLE?)
	// 1.1 ms if if statement is commented out
	// 1.8 ms if if statement is not commented out

	//return;

		if (flag1 == 0) {
			// creates incoherent loads
			temp[thid] = make_float4(0.,0.,0.,0.); 
		}

	} // end of for loop

	//return;
	//--------------------

    int offset = 1;

// xorig - edge/2, xorig + edge/2 - 1

    // build the sum in place up the tree
    for (int d = n>>1; d > 0; d >>= 1)
    {
        __syncthreads();

        if (thid < d)      
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;


			#if 1
            tempf[bi] += tempf[ai];
            tempf[bi+numThreads] += tempf[ai+numThreads];
            tempf[bi+numThreads << 1] += tempf[ai+numThreads << 1];
            tempf[bi+numThreads << 1 + numThreads] += tempf[ai+numThreads << 1 + numThreads];
			#endif
			#
			#if 0
            temp[bi].x += temp[ai].x;
            temp[bi].y += temp[ai].y;
            temp[bi].z += temp[ai].z;
            temp[bi].w += temp[ai].w;
			#endif
        }

        offset <<= 1;
    }

	// Something wrong with the results

    // write results to global memory
    __syncthreads();
	if (thid == (numThreads-1)) {
		float nbs = temp[n-1].w;
		float nbs1 = 1./(nbs*width);
		if (nbs == 0) nbs = 1.;
		sum[blockId] = make_float4(temp[n-1].x*nbs1, temp[n-1].y*nbs1, 0., nbs); //, nbs);
	}
}
//----------------------------------------------------------------------
// More efficient version of scan_workefficient_2 (more threads + remove non-coalesced reads)
__global__ void scan_test_incoherent(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width)
{
    // Dynamically allocated shared memory for scan kernels
    extern  __shared__  float4 temp[];

	//if (blockIdx.x != 2) return;

	//float* g_idata_f = (float*) g_idata;

	int numThreads = blockDim.x * blockDim.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int thid = threadIdx.x + blockDim.x * threadIdx.y;


	temp[thid] = make_float4(0.,0.,0.,0.); 

	return;
}
//----------------------------------------------------------------------
// More efficient version of scan_workefficient_2 (more threads + remove non-coalesced reads)
// Use more threads by reading floats instead of float4
__global__ void scan_workefficient_4(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width)
{
    // Dynamically allocated shared memory for scan kernels
    extern  __shared__  float4 temp[];
	float* tempf = (float*) temp;

	//if (blockIdx.x != 2) return;

	float* g_idata_f = (float*) g_idata;
	//float f1 = g_idata_f[0];

	// blockDim.x == edge (will generalize later)
	int numThreads = blockDim.x * blockDim.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int thid = threadIdx.x + blockDim.x * threadIdx.y;

	// get data from global memory (should be coalesced)
	int thid2 = thid<<1;

	int blockId = blockIdx.x;
	int4& seed = *(seeds+blockId);

	// compute 2D flat texture coordinate from 3D seed coordinate
	int x = seed.x; 
	int y = seed.y;

	// edge should be part of the seed datastructure (per block)
	int edge2 = edge >> 1;
	int xorig = x - edge2;
	int yorig = y - edge2;


	int flag1;

	int xid  = xorig + threadIdx.x; // 2 elements per thread
	int WW = width;  // array width (argument)
	int HH = WW; // height of flat texture // MUST READ AS ARGUMENT

	//--------------------
	int j = 0;
	//for (int j=0; j < 1; j++) { //} the loop added 2 registers (could be unrolled)
		__syncthreads();

		flag1 = 1;

		// need for each of the strings separately
		if (xid < 0 || xid >= WW) {
			flag1 = 0; 
		}

		int yid1 = yorig + threadIdx.y + j*numThreads;
 		int arrayid1 = xid + yid1 * WW;

		// 16 x 16 threads, tile: 16x16 float4 ==> 64 x 16 floats
		// break threads: 32 x 8


		int tid = threadIdx.x + blockDim.x * threadIdx.y;
		// tid = 0 ==. array[0,0] // column-major (Fortran)
		// tid = 1 ==. array[1,0]
		// tid = 15 ==. array[15,0]
		// tid = 16 ==. array[16,0]
		// tid = 17 ==. array[17,0]
		int warp_base = tid >> 5; // divide by 32 // array row  [0,...,7]
		int thread_in_warp = tid - (warp_base << 5);  // [0,...,31]

		// warp 0:   array[0,0]  --> array[31,0]
		// warp 1:   array[32,0] --> array[63,0]
		// warp 2:   array[0,1]  --> array[31,1]
		// warp 3:   array[32,1] --> array[63,1]
		// warp 4:   array[0,2]  --> array[31,2]
		// warp 5:   array[32,2] --> array[63,2]
		// warp 6:   array[0,3]  --> array[31,3]
		// warp 7:   array[32,3] --> array[63,3] // 4th row

		// There are 16 rows in the array. Create a loop: 
		// for (int i=0; i < 4; i++) {
		//   warp 0:   array[0,i*4] --> array[31,i*4]
		//   warp 7:   array[32,i*4+3] --> array[63,i*4]

		// Eventually generalize to more arrays

		// two warps per row

		// arrayid = thread_in_warp + warp_base * WW;
		// tempf[warp_base] = g_idata[array_id]     // floats (64 per row)
		// temp[thid+numThreads] = g_idata[array_id]


	// I could be exceeding memory bounds. So how to read coalesced without
		temp[thid] = g_idata[arrayid1];
		temp[thid].w = 1.;

		if (yid1 < 0 || yid1 >= HH) flag1 = 0;

    	__syncthreads();

	// the data can be in arbitrary order in the shared array

	//    CREATES uncoalesced  loads (HOW POSSIBLE?)
	// 1.1 ms if if statement is commented out
	// 1.8 ms if if statement is not commented out

	//return;

		#if 1 
		if (flag1 == 0) {
			// creates incoherent loads
			temp[thid] = make_float4(0.,0.,0.,0.); 
		}
		#endif

	//} // end of for loop
	//--------------------

	//return;

    int offset = 1;

// xorig - edge/2, xorig + edge/2 - 1

    // build the sum in place up the tree
    for (int d = n>>1; d > 0; d >>= 1)
    {
        __syncthreads();

        if (thid < d)      
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;

            temp[bi].x += temp[ai].x;
            temp[bi].y += temp[ai].y;
            temp[bi].z += temp[ai].z;
            temp[bi].w += temp[ai].w;
        }

        offset <<= 1;
    }

	// Something wrong with the results

    // write results to global memory
    __syncthreads();
	if (thid == (numThreads-1)) {
		float nbs = temp[n-1].w;
		float nbs1 = 1./(nbs*width);
		if (nbs == 0) nbs = 1.;
		sum[blockId] = make_float4(temp[n-1].x*nbs1, temp[n-1].y*nbs1, 0., nbs); //, nbs);
	}
}
//----------------------------------------------------------------------
__global__ void scan_workefficient_5(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width)
// More efficient version of scan_workefficient_2 (more threads + remove non-coalesced reads)
// Use more threads by reading floats instead of float4
{
// edge=16, 64 threads: scan each row, one float per thread

    // Dynamically allocated shared memory for scan kernels
    extern  __shared__  float4 temp[];
	float* tempf = (float*) temp;
	float* sumf = (float*) sum;

	// blockDim.x == edge (will generalize later)
	int numThreads = blockDim.x * blockDim.y;

//	if (blockIdx.x > 1000) return;

	#if 1
	//  SOMETHING NOT WORKING
	int last_share = edge*edge + ((edge*edge) >> LOG_NUM_BANKS);
	//int last_share = 0;
	if (threadIdx.x == 0) {
		int4 ss = seeds[blockIdx.x];
		temp[last_share] = make_float4(ss.x+0.1,ss.y+0.1,ss.z+0.1,ss.w+0.1);
	}
	__syncthreads();
	int4 seed = MAKE_INT4(temp[last_share]); // is int correct? Or must add 0.5?
	__syncthreads();
	#endif
	

	#if 0
	//int4 seed = make_int4(8,8,0,311); // TEST SEED
	int4 seed = seeds[blockIdx.x];
	#endif

	float* g_idata_f = (float*) g_idata;

	// get data from global memory (should be coalesced)

	int x = seed.x; 
	int y = seed.y;


	// edge should be part of the seed datastructure (per block)
	int edge2 = edge >> 2;
	int xorig = x - edge2; // one thread per float (edge float4's)
	int yorig = y - edge2; // edge rows

	// align xorig such that xorig is a multiple of 2 (speedup is evident)
	//xorig = (xorig >> 1) << 1; // xorig is divisble by 2^1
	int shift = xorig - ((xorig >> 2) << 2);
	if (shift == 1) xorig -= 1;
	else if (shift == 2) xorig += 2;
	else if (shift == 3) xorig += 1;
	//else return;

	int flag1;

	int WW = width;  // array width (argument) (in float4)
	int HH = WW; // height of flat texture // MUST READ AS ARGUMENT
	int xid  = 4*xorig + threadIdx.x; // measured in floats

	//--------------------
	// one iteration per row in the square tile
	for (int j=0; j < edge; j++) { // the loop added 2 registers (could be unrolled)
		__syncthreads();

		flag1 = 1;

		// need for each of the strings separately
		if (xid < 0 || xid >= (WW*4)) flag1 = 0; 

		int yid1 = yorig + j;
		if (yid1 < 0 || yid1 >= HH) flag1 = 0;

 		int arrayid1 = xid + yid1*WW*4; // WW*4 floats

		// I MUST ALSO CHECK THE SEED VALUE


		tempf[j*4*edge+threadIdx.x] = 0.;

		// crashes without this test
		if (flag1 != 0) {
			tempf[j*4*edge+threadIdx.x] = g_idata_f[arrayid1];
			//tempf[j*4*edge+threadIdx.x] = 0.; // very low overhead
		}


		//sumf[j*4*edge+threadIdx.x] = g_idata_f[arrayid1];
		//sumf[j*4*edge+threadIdx.x] = tempf[j*4*edge+threadIdx.x];
		//sumf[j*4*edge+threadIdx.x] = tempf[j*4*edge+threadIdx.x];
		//sumf[j*4*edge+threadIdx.x] = xorig;

		//if (j == 0) {
			//sum[threadIdx.x] = make_float4(seed.x,seed.y,seed.z,seed.w);
			//return;
		//}

    	__syncthreads();

	} // end of for loop

	//return;

	__syncthreads();

	float widthi = 1./width;

	for (int j=0; j < edge; j++) {
		__syncthreads();
		if (threadIdx.x < edge) {
			float f = temp[j*edge+threadIdx.x].w + 0.1;  // so that int() works
			if (int(f) != seed.w) {
				temp[j*edge+threadIdx.x] = make_float4(0.,0.,0.,0.);
			} else {
				temp[j*edge+threadIdx.x].x = (xorig+threadIdx.x) * widthi;
				temp[j*edge+threadIdx.x].y = (yorig+j) * widthi;
				temp[j*edge+threadIdx.x].w = 1.;
			}
		}
	}
	__syncthreads();
	#if 0
	for (int j=0; j < edge; j++) {
		sumf[j*4*edge+threadIdx.x] = tempf[j*4*edge+threadIdx.x];
	}
	return;
	#endif

	#if 0
	for (int j=0; j < edge; j++) { // the loop added 2 registers (could be unrolled)
		//sumf[j*4*edge+threadIdx.x] = g_idata_f[arrayid1];
		sumf[j*4*edge+threadIdx.x] = tempf[j*4*edge+threadIdx.x];
		//sumf[j*4*edge+threadIdx.x] = xorig;
		//sum[threadIdx.x] = make_float4(seed.x,seed.y,seed.z,seed.w);
	}
	return;
	#endif
	//--------------------

// xorig - edge/2, xorig + edge/2 - 1

// For the 16x16 case (hardcoded), the first pass with 64 threads can 
// only handle 1/2 the domain (1024 floats = 16x16x4). The for loop that
// follows had a thread handle two floats at a time, so can only handl
// 1/2 the domain on each pass

// manually treat each half of the domain

    int offset = 1;
	//====

	int tid, j, ai, bi;
	int mx = 8;

			tid = threadIdx.x >> 2;
			j = threadIdx.x - (tid << 2);


	for (int outer=0; outer < 3; outer++) { // HARDCODED
		for (int k=0; k < mx; k++) {
			__syncthreads();
			int off = k * 128 * (1 << outer); // HARDCODED

			ai = offset*(2*tid+1)-1;
			bi = offset*(2*tid+2)-1;

			ai = (ai << 2) + j;
			bi = (bi << 2) + j;

			tempf[bi+off] += tempf[ai+off];
		}
		mx >> 1;
		offset <<= 1;
	}
	//====
	#if 0
	for (int j=0; j < edge; j++) { // the loop added 2 registers (could be unrolled)
		//sumf[j*4*edge+threadIdx.x] = g_idata_f[arrayid1];
		sumf[j*4*edge+threadIdx.x] = tempf[j*4*edge+threadIdx.x];
		//sumf[j*4*edge+threadIdx.x] = xorig;
		//sum[threadIdx.x] = make_float4(seed.x,seed.y,seed.z,seed.w);
	}
	return;
	#endif
	#


	int sz = edge*edge / 2; //  (128 for 64 threads)

	#if 1
    // build the sum in place up the tree
    for (int d = sz>>1; d > 0; d >>= 1) {
        __syncthreads();

        if (threadIdx.x < d)      
        {
			//int tid = threadIdx.x >> 2; // thread id divided by 4
			//int j = threadIdx.x - (tid << 2); // 0,1,2,3

            int ai = offset*(2*tid+1)-1;
            int bi = offset*(2*tid+2)-1;

			ai = (ai << 2) + j;
			bi = (bi << 2) + j;

            tempf[bi] += tempf[ai];
        }

        offset <<= 1;
    }
	#endif
	#
	#if 0
	for (int j=0; j < edge; j++) { // the loop added 2 registers (could be unrolled)
		//sumf[j*4*edge+threadIdx.x] = g_idata_f[arrayid1];
		sumf[j*4*edge+threadIdx.x] = tempf[j*4*edge+threadIdx.x];
		//sumf[j*4*edge+threadIdx.x] = xorig;
		//sum[threadIdx.x] = make_float4(seed.x,seed.y,seed.z,seed.w);
	}
	//return;
	#endif

	// Something wrong with the results

    // write results to global memory
    __syncthreads();
	if (threadIdx.x == (numThreads-1)) {
		int el = edge*edge-1;
		//int el = 0;
		float nbs = temp[el].w;
		//float nbs1 = 1./(nbs*width);
		float nbs1 = 1./(nbs);
		if (nbs <= 0.5) nbs1 = 1.;
		//sum[blockIdx.x] = make_float4(temp[el].x*nbs1, temp[el].y*nbs1, width, nbs);
		sum[blockIdx.x] = make_float4(temp[el].x*nbs1, temp[el].y*nbs1, 0., nbs); // orig
	}
}
//----------------------------------------------------------------------

__global__ void scan_workefficient_6(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width)
// More efficient version of scan_workefficient_2 (more threads + remove non-coalesced reads)
// Use more threads by reading floats instead of float4
// Remove bank conflicts (decrease serialized_warps)
{
// edge=16, 64 threads: scan each row, one float per thread

    // Dynamically allocated shared memory for scan kernels
    extern  __shared__  float4 temp[];
	float* tempf = (float*) temp;
	float* sumf = (float*) sum;

	// blockDim.x == edge (will generalize later)
	int numThreads = blockDim.x * blockDim.y;

//	if (blockIdx.x > 1000) return;

	#if 1
	//  SOMETHING NOT WORKING
	int last_share = edge*edge + ((edge*edge) >> LOG_NUM_BANKS);
	//int last_share = 0;
	if (threadIdx.x == 0) {
		int4 ss = seeds[blockIdx.x];
		temp[last_share] = make_float4(ss.x+0.1,ss.y+0.1,ss.z+0.1,ss.w+0.1);
	}
	__syncthreads();
	int4 seed = MAKE_INT4(temp[last_share]); // is int correct? Or must add 0.5?
	__syncthreads();
	#endif
	

	#if 0
	//int4 seed = make_int4(8,8,0,311); // TEST SEED
	int4 seed = seeds[blockIdx.x];
	#endif

	float* g_idata_f = (float*) g_idata;

	// get data from global memory (should be coalesced)

	int x = seed.x; 
	int y = seed.y;


	// edge should be part of the seed datastructure (per block)
	int edge2 = edge >> 2;
	int xorig = x - edge2; // one thread per float (edge float4's)
	int yorig = y - edge2; // edge rows

	// align xorig such that xorig is a multiple of 2 (speedup is evident)
	//xorig = (xorig >> 1) << 1; // xorig is divisble by 2^1
	int shift = xorig - ((xorig >> 2) << 2);
	if (shift == 1) xorig -= 1;
	else if (shift == 2) xorig += 2;
	else if (shift == 3) xorig += 1;
	//else return;

	int flag1;

	int WW = width;  // array width (argument) (in float4)
	int HH = WW; // height of flat texture // MUST READ AS ARGUMENT
	int xid  = 4*xorig + threadIdx.x; // measured in floats

	//--------------------
	// one iteration per row in the square tile
	for (int j=0; j < edge; j++) { // the loop added 2 registers (could be unrolled)
		__syncthreads();

		flag1 = 1;

		// need for each of the strings separately
		if (xid < 0 || xid >= (WW*4)) flag1 = 0; 

		int yid1 = yorig + j;
		if (yid1 < 0 || yid1 >= HH) flag1 = 0;

 		int arrayid1 = xid + yid1*WW*4; // WW*4 floats

		// I MUST ALSO CHECK THE SEED VALUE


		TMPF(j*4*edge+threadIdx.x) = 0.;

		// crashes without this test
		if (flag1 != 0) {
			TMPF(j*4*edge+threadIdx.x) = g_idata_f[arrayid1];
			//tempf[j*4*edge+threadIdx.x] = 0.; // very low overhead
		}


		//sumf[j*4*edge+threadIdx.x] = g_idata_f[arrayid1];
		//sumf[j*4*edge+threadIdx.x] = tempf[j*4*edge+threadIdx.x];
		//sumf[j*4*edge+threadIdx.x] = tempf[j*4*edge+threadIdx.x];
		//sumf[j*4*edge+threadIdx.x] = xorig;

		//if (j == 0) {
			//sum[threadIdx.x] = make_float4(seed.x,seed.y,seed.z,seed.w);
			//return;
		//}

    	__syncthreads();

	} // end of for loop

	//return;

	__syncthreads();

	float widthi = 1./width;

	for (int j=0; j < edge; j++) {
		__syncthreads();
		if (threadIdx.x < edge) {
			float f = temp[j*edge+threadIdx.x].w + 0.1;  // so that int() works
			if (int(f) != seed.w) {
				temp[j*edge+threadIdx.x] = make_float4(0.,0.,0.,0.);
			} else {
				temp[j*edge+threadIdx.x].x = (xorig+threadIdx.x) * widthi;
				temp[j*edge+threadIdx.x].y = (yorig+j) * widthi;
				temp[j*edge+threadIdx.x].w = 1.;
			}
		}
	}
	__syncthreads();
	#if 0
	for (int j=0; j < edge; j++) {
		sumf[j*4*edge+threadIdx.x] = TMPF(j*4*edge+threadIdx.x);
	}
	return;
	#endif

	#if 0
	for (int j=0; j < edge; j++) { // the loop added 2 registers (could be unrolled)
		//sumf[j*4*edge+threadIdx.x] = g_idata_f[arrayid1];
		sumf[j*4*edge+threadIdx.x] = TMPF(j*4*edge+threadIdx.x);
		//sumf[j*4*edge+threadIdx.x] = xorig;
		//sum[threadIdx.x] = make_float4(seed.x,seed.y,seed.z,seed.w);
	}
	return;
	#endif
	//--------------------

// xorig - edge/2, xorig + edge/2 - 1

// For the 16x16 case (hardcoded), the first pass with 64 threads can 
// only handle 1/2 the domain (1024 floats = 16x16x4). The for loop that
// follows had a thread handle two floats at a time, so can only handl
// 1/2 the domain on each pass

// manually treat each half of the domain

    int offset = 1;
	//====

	int mx = 8;
	for (int outer=0; outer < 3; outer++) { // HARDCODED
		for (int k=0; k < mx; k++) {
			__syncthreads();
			int off = k * 128 * (1 << outer); // HARDCODED
			int tid = threadIdx.x >> 2;
			int j = threadIdx.x - (tid << 2);
		
			int ai = offset*(2*tid+1)-1;
			int bi = offset*(2*tid+2)-1;

			ai = (ai << 2) + j;
			bi = (bi << 2) + j;

			TMPF(bi+off) += TMPF(ai+off);
		}
		mx >> 1;
		offset <<= 1;
	}
	//====
	#if 0
	for (int j=0; j < edge; j++) { // the loop added 2 registers (could be unrolled)
		//sumf[j*4*edge+threadIdx.x] = g_idata_f[arrayid1];
		sumf[j*4*edge+threadIdx.x] = TMPF(j*4*edge+threadIdx.x);
		//sumf[j*4*edge+threadIdx.x] = xorig;
		//sum[threadIdx.x] = make_float4(seed.x,seed.y,seed.z,seed.w);
	}
	return;
	#endif


	int sz = edge*edge / 2; //  (128 for 64 threads)

	#if 1
    // build the sum in place up the tree
    for (int d = sz>>1; d > 0; d >>= 1) {
        __syncthreads();

        if (threadIdx.x < d)      
        {
			int tid = threadIdx.x >> 2; // thread id divided by 4
			int j = threadIdx.x - (tid << 2); // 0,1,2,3

            int ai = offset*(2*tid+1)-1;
            int bi = offset*(2*tid+2)-1;

			ai = (ai << 2) + j;
			bi = (bi << 2) + j;

            TMPF(bi) += TMPF(ai);
        }

        offset <<= 1;
    }
	#endif
	#
	#if 0
	for (int j=0; j < edge; j++) { // the loop added 2 registers (could be unrolled)
		//sumf[j*4*edge+threadIdx.x] = g_idata_f[arrayid1];
		sumf[j*4*edge+threadIdx.x] = TMPF(j*4*edge+threadIdx.x);
		//sumf[j*4*edge+threadIdx.x] = xorig;
		//sum[threadIdx.x] = make_float4(seed.x,seed.y,seed.z,seed.w);
	}
	return;
	#endif

	// Something wrong with the results

    // write results to global memory
    __syncthreads();
	if (threadIdx.x == (numThreads-1)) {
		int el = edge*edge-1;
		//int el = 0;
		float nbs = temp[el].w;
		//float nbs1 = 1./(nbs*width);
		float nbs1 = 1./(nbs);
		if (nbs <= 0.5) nbs1 = 1.;
		//sum[blockIdx.x] = make_float4(temp[el].x*nbs1, temp[el].y*nbs1, width, nbs);
		sum[blockIdx.x] = make_float4(temp[el].x*nbs1, temp[el].y*nbs1, 0., nbs); // orig
	}
}
//----------------------------------------------------------------------
__global__ void scan_workefficient_7(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width)
// More efficient version of scan_workefficient_2 (more threads + remove non-coalesced reads)
// Use more threads by reading floats instead of float4
{
// edge=16, 64 threads: scan each row, one float per thread

    // Dynamically allocated shared memory for scan kernels
    extern  __shared__  float4 temp[];
	float* tempf = (float*) temp;
	float* sumf = (float*) sum;

	// blockDim.x == edge (will generalize later)
	int numThreads = blockDim.x * blockDim.y;

	//if (blockIdx.x != 11) return; // block 13 has serial errors

	#if 1
	//  SOMETHING NOT WORKING
	int last_share = edge*edge + ((edge*edge) >> LOG_NUM_BANKS);
	//int last_share = 0;
	if (threadIdx.x == 0) {
		int4 ss = seeds[blockIdx.x];
		temp[last_share] = make_float4(ss.x+0.1,ss.y+0.1,ss.z+0.1,ss.w+0.1);
	}
	__syncthreads();
	int4 seed = MAKE_INT4(temp[last_share]); // is int correct? Or must add 0.5?
	__syncthreads();
	#endif

	#if 0
	//int4 seed = make_int4(8,8,0,311); // TEST SEED
	int4 seed = seeds[blockIdx.x];
	#endif

	float* g_idata_f = (float*) g_idata;

	// get data from global memory (should be coalesced)

	int x = seed.x; 
	int y = seed.y;

	// edge should be part of the seed datastructure (per block)
	int edge2 = edge >> 2;
	int xorig = x - edge2; // one thread per float (edge float4's)
	int yorig = y - edge2; // edge rows

	// align xorig such that xorig is a multiple of 2 (speedup is evident)
	//xorig = (xorig >> 1) << 1; // xorig is divisble by 2^1
	int shift = xorig - ((xorig >> 2) << 2);
	if (shift == 1) xorig -= 1;
	else if (shift == 2) xorig += 2;
	else if (shift == 3) xorig += 1;
	//else return;

	int flag1;

	int WW = width;  // array width (argument) (in float4)
	int HH = WW; // height of flat texture // MUST READ AS ARGUMENT

	//--------------------
	// one iteration per row in the square tile
	for (int j=0; j < edge; j+=4) { // the loop added 2 registers (could be unrolled)
		__syncthreads();
//
		int subtid = j >> 4; // 0, 1, ..., numThreads/4
		int subrow = j - subtid;
//
		flag1 = 1;
//
//		// need for each of the strings separately
		int xid  = 4*xorig + threadIdx.x; // measured in floats
		if (xid < 0 || xid >= (WW*4)) flag1 = 0; 
//
		int yid1 = yorig + j + threadIdx.y;
		if (yid1 < 0 || yid1 >= HH) flag1 = 0;
//
 		int arrayid1 = xid + yid1*WW*4; // WW*4 floats
//
//		// I MUST ALSO CHECK THE SEED VALUE
//
//
		int jj = j+threadIdx.y;
		tempf[jj*4*edge+threadIdx.x] = 0.;
//
//		// crashes without this test
		if (flag1 != 0) {
			tempf[jj*4*edge+threadIdx.x] = g_idata_f[arrayid1];
			//tempf[jj*4*edge+threadIdx.x] = 0.; // very low overhead
		}
//
//
//		//sumf[jj*4*edge+threadIdx.x] = g_idata_f[arrayid1];
//		//sumf[jj*4*edge+threadIdx.x] = tempf[jj*4*edge+threadIdx.x];
//		//sumf[jj*4*edge+threadIdx.x] = tempf[jj*4*edge+threadIdx.x];
//		//sumf[jj*4*edge+threadIdx.x] = xorig;
//
//		//if (j == 0) {
//			//sum[threadIdx.x] = make_float4(seed.x,seed.y,seed.z,seed.w);
//			//return;
//		//}
//
    	__syncthreads();

	} // end of for loop
//
//
	__syncthreads();
//
	float widthi = 1./width;
	int thread_id = threadIdx.x + blockDim.x * threadIdx.y;

	// use float4
	// not the problem
	#if 0
	// 256 threads
	int tid = thread_id >> 2;
	int j4 = thread_id - (tid << 2);

	int (int j=0; j < 4; j++) {
		__syncthreads();
			if (j4 == 3) {
				float f = tempf[j*256 + tid + j4] + 0.1;
				if (int(f) != seed.w) {
					//tempf[j*256+thread_id] = 0.;
					temp[j*64+threadIdx.x] = make_float4(0.,0.,0.,0.);
				}
			}
	}
	#endif

	// use float4
	// not the problem
	#if 1
	for (int j=0; j < edge; j++) {   // takes 1.7 ms
		__syncthreads();
		if (threadIdx.x < edge && threadIdx.y == 0) {
			int tid = threadIdx.x;
			float f = temp[j*16+tid].w + 0.1;  // so that int() works
			if (int(f) != seed.w) {
				temp[j*16+tid] = make_float4(0.,0.,0.,0.); // cause of serialization
			} // else {
				//float4 g; 
				// Will do this later
				//g.x = (xorig+threadIdx.x) * widthi;
				//g.y = (yorig+j) * widthi;
				//g.z = 0.;
				//g.w = 1.;
				//temp[j*edge+threadIdx.x] = g;
			// }
		}
	}
	#endif

	// use float
	#if 0
	for (int j=0; j < edge; j++) {   // takes 1.7 ms
		__syncthreads();
		if (threadIdx.x < edge && threadIdx.y == 0) {
			int tid = thread_id >> 2;
			int j = thread_id - (tid << 2);
			float f = tempf[4*j*16+tid+j] + 0.1;  // so that int() works
			if (int(f) != seed.w) {
				tempf[4*j*16+tid] = 0.; 
				tempf[4*j*16+tid+1] = 0.; 
				tempf[4*j*16+tid+2] = 0.; 
				tempf[4*j*16+tid+3] = 0.; 
			} else {
				float4 g; 
				// Will do this later
				//g.x = (xorig+threadIdx.x) * widthi;
				//g.y = (yorig+j) * widthi;
				//g.z = 0.;
				//g.w = 1.;
				//temp[jj*edge+threadIdx.x] = g;
			}
		}
	}
	#endif
	__syncthreads();

	//return;
	#if 0
	for (int j=0; j < 4; j++) { // the loop added 2 registers (could be unrolled)
		sumf[j*256+thread_id] = tempf[j*256+thread_id];
	}
	return;
	#endif

//	//--------------------
//
//// xorig - edge/2, xorig + edge/2 - 1
//
//// For the 16x16 case (hardcoded), the first pass with 64 threads can 
//// only handle 1/2 the domain (1024 floats = 16x16x4). The for loop that
//// follows had a thread handle two floats at a time, so can only handl
//// 1/2 the domain on each pass
//
//// manually treat each half of the domain
//
    int offset = 1;
//	//====
//
	int tid, j4, ai, bi;

//	for (int j=0; j < 4; j++) { // the loop added 2 registers (could be unrolled)
		//sumf[j*256+thread_id] = tempf[j*256+thread_id];
	//}
	//return;

	tid = thread_id >> 2;
	j4 = thread_id - (tid << 2);

		for (int k=0; k < 2; k++) {
			__syncthreads();
			int off = k * 512;

			ai = offset*(2*tid+1)-1;
			bi = offset*(2*tid+2)-1;

			ai = (ai << 2) + j4;
			bi = (bi << 2) + j4;

			tempf[bi+off] += tempf[ai+off];
		}
		offset <<= 1;
	//====

	__syncthreads();
	#if 0
	for (int j=0; j < 4; j++) { // the loop added 2 registers (could be unrolled)
		sumf[j*256+thread_id] = tempf[j*256+thread_id];
	}
	return;
	#endif
//
//
	int sz = 512; // * edge*edge; //  (512 for 256 threads)

	//return;

//
	#if 1
    // build the sum in place up the tree
    for (int d = sz>>3; d > 0; d >>= 1) {
        __syncthreads();

        if (thread_id < (d*4))       // 4 subthreads per thread: 64*4 = 256
        {
            int ai = offset*(2*tid+1)-1;
            int bi = offset*(2*tid+2)-1;

			ai = (ai << 2) + j4;
			bi = (bi << 2) + j4;

            tempf[bi] += tempf[ai];
        }

        offset <<= 1;
		//if (d == 0) break;
    }
	#endif
	#if 0
	for (int j=0; j < 4; j++) { // the loop added 2 registers (could be unrolled)
		sumf[j*256+thread_id] = tempf[j*256+thread_id];
	}
	return;
	#endif

	// Something wrong with the results

    // write results to global memory
    __syncthreads();
	if (threadIdx.x == (blockDim.x-1)) {
		int el = edge*edge-1;
		//int el = 0;
		float nbs = temp[el].w;
		//float nbs1 = 1./(nbs*width);
		float nbs1 = 1./(nbs);
		if (nbs <= 0.5) nbs1 = 1.;
		//sum[blockIdx.x] = make_float4(temp[el].x*nbs1, temp[el].y*nbs1, width, nbs);
		sum[blockIdx.x] = make_float4(temp[el].x*nbs1, temp[el].y*nbs1, 0., nbs); // orig
	}
}
//----------------------------------------------------------------------

#endif // #ifndef _SCAN_WORKEFFICIENT_KERNEL_H_

