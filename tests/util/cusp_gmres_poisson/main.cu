#include "hip/hip_runtime.h"
// TODO : test this: 
//#define CUSP_USE_TEXTURE_MEMORY

// THIS IS adapted from verbose_monitor.cu
// PROVIDED BY THE CUSP v0.1 EXAMPLES

#include "grids/grid_reader.h"
#include "rbffd/rbffd.h"
#include "timer_eb.h" 

#include <cusp/hyb_matrix.h>
#include <cusp/ell_matrix.h>
#include <cusp/csr_matrix.h>
#include <cusp/coo_matrix.h>
#include <cusp/monitor.h>
#include <cusp/krylov/cg.h>
#include <cusp/krylov/gmres.h>
#include <cusp/gallery/poisson.h>
#include <cusp/print.h>
#include <cusp/array2d.h>
#include <cusp/multiply.h>
#include <cusp/blas.h>
#include <cusp/io/matrix_market.h>


#include <thrust/host_vector.h> 
#include <thrust/device_vector.h>
#include <thrust/generate.h>


#include "utils/spherical_harmonics.h"

#include <iomanip>
#include <iostream>
#include <sstream> 
#include <map>
#include <fstream> 
#include <typeinfo> 
using namespace std;


typedef std::vector< std::map< unsigned int, double> > STL_MAT_t; 
typedef std::vector<double> STL_VEC_t; 


typedef cusp::array1d<double, cusp::host_memory> HOST_VEC_t; 
typedef cusp::array1d<double, cusp::device_memory> DEVICE_VEC_t; 
typedef cusp::csr_matrix<unsigned int, double, cusp::host_memory> HOST_MAT_t; 
typedef cusp::csr_matrix<unsigned int, double, cusp::device_memory> DEVICE_MAT_t; 

EB::TimerList tm;

//---------------------------------

// Perform GMRES on GPU
void GMRES_Device(DEVICE_MAT_t& A, DEVICE_VEC_t& F, DEVICE_VEC_t& U_exact, DEVICE_VEC_t& U_approx_gpu) {
#if 1
    size_t restart = 300; 
    int max_iters = 10000; 
    double rel_tol = 1e-8; 
#else 
    size_t restart = 50; 
    int max_iters = 100; 
    double rel_tol = 1e-8; 
#endif 

    try {

        //    cusp::convergence_monitor<double> monitor( F, max_iters, 0, 1e-3); 
        cusp::default_monitor<double> monitor( F, max_iters, rel_tol);// , 1e-3); 

        hipDeviceSynchronize();
        std::cout << "Generated monitor\n";
        // 1e-8, 10000, 300); 

        // Solve Au = F
        cusp::krylov::gmres(A, U_approx_gpu, F, restart, monitor); 
        hipDeviceSynchronize(); 

        //    monitor.print();

        if (monitor.converged())
        {
            std::cout << "\n[+++] Solver converged to " << monitor.relative_tolerance() << " relative tolerance";       
            std::cout << " after " << monitor.iteration_count() << " iterations" << std::endl << std::endl;
        }
        else
        {
            std::cout << "\n[XXX] Solver reached iteration limit " << monitor.iteration_limit() << " before converging";
            std::cout << " to " << monitor.relative_tolerance() << " relative tolerance " << std::endl << std::endl;
        }

        std::cout << "GMRES Iterations: " << monitor.iteration_count() << std::endl;
        std::cout << "GMRES Iteration Limit: " << monitor.iteration_limit() << std::endl;
        std::cout << "GMRES Residual Norm: " << monitor.residual_norm() << std::endl;
        std::cout << "GMRES Relative Tol: " << monitor.relative_tolerance() << std::endl;
        std::cout << "GMRES Absolute Tol: " << monitor.absolute_tolerance() << std::endl;
        std::cout << "GMRES Target Residual (Abs + Rel*norm(F)): " << monitor.tolerance() << std::endl;
    }
    catch(std::bad_alloc &e)
    {
        std::cerr << "Ran out of memory trying to compute GMRES: " << e.what() << std::endl;
        exit(-1);
    }
    catch(thrust::system_error &e)
    {
        std::cerr << "Some other error happened during GMRES: " << e.what() << std::endl;
        exit(-1);
    }


    try {

        typedef cusp::array1d<double, DEVICE_VEC_t>::view DEVICE_VEC_VIEW_t; 

        DEVICE_VEC_VIEW_t U_approx_view(U_exact.begin()+(U_exact.size() - F.size()), U_exact.end()); 

        DEVICE_VEC_t diff(U_approx_gpu); 

        //cusp::blas::axpy(U_exact.begin()+(U_exact.size() - F.size()), U_exact.end(), diff.begin(),  -1); 
        cusp::blas::axpy(U_approx_view, diff, -1); 

        std::cout << "Rel l1   Norm: " << cusp::blas::nrm1(diff) / cusp::blas::nrm1(U_exact) << std::endl;  
        std::cout << "Rel l2   Norm: " << cusp::blas::nrm2(diff) / cusp::blas::nrm2(U_exact) << std::endl;  
        std::cout << "Rel linf Norm: " << cusp::blas::nrmmax(diff) / cusp::blas::nrmmax(U_exact) << std::endl;  
    }
    catch(std::bad_alloc &e)
    {
        std::cerr << "Ran out of memory trying to compute Error Norms: " << e.what() << std::endl;
        exit(-1);
    }
    catch(thrust::system_error &e)
    {
        std::cerr << "Some other error happened during Error Norms: " << e.what() << std::endl;
        exit(-1);
    }
}

//---------------------------------

// Assemble the LHS matrix with the Identity for boundary nodes. Assume solver
// is intelligent enough to use information and converge
// 
void assemble_System_Compressed( RBFFD& der, Grid& grid, HOST_MAT_t& A, HOST_VEC_t& F, HOST_VEC_t& U_exact){
    unsigned int N = grid.getNodeListSize(); 
    unsigned int n = grid.getMaxStencilSize(); 

    unsigned int nb_bnd = grid.getBoundaryIndicesSize();

    std::cout << "Boundary nodes: " << nb_bnd << std::endl;


    //------ RHS ----------

    SphericalHarmonic::Sph105 UU; 

    std::vector<NodeType>& nodes = grid.getNodeList(); 

    // We want U_exact to have the FULL solution. 
    // F should only have the compressed problem. 
    for (unsigned int i = 0; i < nb_bnd; i++) {
        NodeType& node = nodes[i]; 
        double Xx = node.x(); 
        double Yy = node.y(); 
        double Zz = node.z(); 

        U_exact[i] = UU.eval(Xx, Yy, Zz) + 2*M_PI; 
    }

    for (unsigned int i = nb_bnd; i < N; i++) {
        NodeType& node = nodes[i]; 
        double Xx = node.x(); 
        double Yy = node.y(); 
        double Zz = node.z(); 

        U_exact[i] = UU.eval(Xx, Yy, Zz) + 2*M_PI; 
        // Solving -lapl(u + const) = f = -lapl(u) + 0
        // of course the lapl(const) is 0, so we will have a test to verify
        // that our null space is closed. 
        F[i-nb_bnd] = -UU.lapl(Xx, Yy, Zz); 
    }

    //------ LHS ----------

    unsigned int ind = 0; 
    // NOTE: assumes the boundary is sorted to the top of the node indices
    for (unsigned int i = nb_bnd; i < N; i++) {
        StencilType& sten = grid.getStencil(i); 
        double* lapl = der.getStencilWeights(RBFFD::LSFC, i); 

        A.row_offsets[i-nb_bnd] = ind;

        for (unsigned int j = 0; j < n; j++) {
            if (sten[j] < (int)nb_bnd) { 
                // Subtract the solution*weight from the element of the RHS. 
                F[i-nb_bnd] -= (U_exact[sten[j]] * ( -lapl[j] )); 
                // std::cout << "Node " << i << " depends on boundary\n"; 
            } else {
                // Offset by nb_bnd so we crop off anything related to the boundary
                A.column_indices[ind] = sten[j]-nb_bnd; 
                A.values[ind] = -lapl[j]; 
                ind++; 
            }
        }
    }    

    // VERY IMPORTANT. UNSPECIFIED LAUNCH FAILURES ARE CAUSED BY FORGETTING THIS!
    A.row_offsets[N-nb_bnd] = ind; 
}



// Assemble the LHS matrix with the Identity for boundary nodes. Assume solver
// is intelligent enough to use information and converge
// 
void assemble_System_Bnd_Eye( RBFFD& der, Grid& grid, HOST_MAT_t& A, HOST_VEC_t& F, HOST_VEC_t& U_exact){
    unsigned int N = grid.getNodeListSize(); 
    unsigned int n = grid.getMaxStencilSize(); 

    unsigned int nb_bnd = grid.getBoundaryIndicesSize();

    std::cout << "Boundary nodes: " << nb_bnd << std::endl;

    //------ RHS ----------

    SphericalHarmonic::Sph105 UU; 

    std::vector<NodeType>& nodes = grid.getNodeList(); 

    for (unsigned int i = 0; i < nb_bnd; i++) {
        NodeType& node = nodes[i]; 
        double Xx = node.x(); 
        double Yy = node.y(); 
        double Zz = node.z(); 

        U_exact[i] = UU.eval(Xx, Yy, Zz) + 2*M_PI; 
        F[i] = U_exact[i]; 
    }

    for (unsigned int i = nb_bnd; i < N; i++) {
        NodeType& node = nodes[i]; 
        double Xx = node.x(); 
        double Yy = node.y(); 
        double Zz = node.z(); 

        U_exact[i] = UU.eval(Xx, Yy, Zz) + 2*M_PI; 
        // Solving -lapl(u + const) = f = -lapl(u) + 0
        // of course the lapl(const) is 0, so we will have a test to verify
        // that our null space is closed. 
        F[i] = -UU.lapl(Xx, Yy, Zz); 
    }


    //------ LHS ----------
    unsigned ind = 0; 
    for (unsigned int i = 0; i < nb_bnd; i++) {
        A.row_offsets[i] = ind; 
        A.column_indices[ind] = i; 
        A.values[ind] = 1; 
        ind++; 
    }

    for (unsigned int i = nb_bnd; i < N; i++) {
        StencilType& sten = grid.getStencil(i); 
        double* lapl = der.getStencilWeights(RBFFD::LSFC, i); 

        A.row_offsets[i] = ind; 

        for (unsigned int j = 0; j < n; j++) {
            A.column_indices[ind] = sten[j]; 
            A.values[ind] = -lapl[j]; 
            ind++; 
        }
    }

    // VERY IMPORTANT. UNSPECIFIED LAUNCH FAILURES ARE CAUSED BY FORGETTING THIS!
    A.row_offsets[N] = ind; 
}

    template <typename VecT>
void write_to_file(VecT vec, std::string filename)
{
    std::ofstream fout;
    fout.open(filename.c_str());
    for (size_t i = 0; i < vec.size(); i++) {
        fout << std::setprecision(10) << vec[i] << std::endl;
    }
    fout.close();
    std::cout << "Wrote " << filename << std::endl;
}


void write_System ( HOST_MAT_t& A, HOST_VEC_t& F, HOST_VEC_t& U_exact )
{
    write_to_file(F, "output/F.mtx"); 
    write_to_file(U_exact, "output/U_exact.mtx"); 
    cusp::io::write_matrix_market_file(A,"output/LHS.mtx"); 
}

void write_Solution( Grid& grid, HOST_VEC_t& U_exact, DEVICE_VEC_t& U_approx_gpu ) 
{
    unsigned int nb_bnd = grid.getBoundaryIndicesSize();

    // IF we want to write details we need to copy back to host. 
    HOST_VEC_t U_approx(U_exact.size());

    if (U_approx_gpu.size() == U_exact.size()) {
        thrust::copy(U_approx_gpu.begin(), U_approx_gpu.end(), U_approx.begin());
    } else {
        thrust::copy(U_exact.begin(), U_exact.begin()+nb_bnd, U_approx.begin());
        thrust::copy(U_approx_gpu.begin(), U_approx_gpu.end(), U_approx.begin()+nb_bnd);
    }

    write_to_file(U_approx, "output/U_gpu.mtx"); 
}


//---------------------------------

void gpuTest(RBFFD& der, Grid& grid, int primeGPU=0) {
    unsigned int N = grid.getNodeListSize(); 
    unsigned int n = grid.getMaxStencilSize(); 

    char test_name[256]; 
    char assemble_timer_name[256]; 
    char copy_timer_name[512]; 
    char test_timer_name[256]; 

    if (primeGPU) {
        sprintf(test_name, "%u PRIMING THE GPU", N);  
        sprintf(assemble_timer_name, "%u Primer Assemble", N);
        sprintf(copy_timer_name,     "%u Primer Copy To CUSP_DEVICE_CSR", N); 
        sprintf(test_timer_name, "%u Primer GMRES test", N); 
    } else { 
        sprintf(test_name, "%u GMRES GPU (CUSP_DEVICE_CSR)", N);  
        sprintf(assemble_timer_name, "%u CUSP_HOST_CSR Assemble", N);
        sprintf(copy_timer_name,     "%u CUSP_HOST_CSR Copy To CUSP_DEVICE_CSR", N); 
        sprintf(test_timer_name, "%u GPU GMRES test", N); 
    }

    if (!tm.contains(assemble_timer_name)) { tm[assemble_timer_name] = new EB::Timer(assemble_timer_name); } 
    if (!tm.contains(copy_timer_name)) { tm[copy_timer_name] = new EB::Timer(copy_timer_name); } 
    if (!tm.contains(test_timer_name)) { tm[test_timer_name] = new EB::Timer(test_timer_name); } 


    std::cout << test_name << std::endl;


    // ----- ASSEMBLE -----
    tm[assemble_timer_name]->start(); 
#if 0
    // Keep rows in system for boundary
    HOST_MAT_t* A = new HOST_MAT_t(N, N, n*N); 
    HOST_VEC_t* F = new HOST_VEC_t(N, 1);
    HOST_VEC_t* U_exact = new HOST_VEC_t(N, 1);
    assemble_System_Bnd_Eye(der, grid, *A, *F, *U_exact); 
#else 
    // Compress system to remove boundary rows
    unsigned int nb_bnd = grid.getBoundaryIndicesSize();
    HOST_MAT_t* A = new HOST_MAT_t(N-nb_bnd, N-nb_bnd, n*(N-nb_bnd)); 
    HOST_VEC_t* F = new HOST_VEC_t(N-nb_bnd, 1);
    HOST_VEC_t* U_exact = new HOST_VEC_t(N, 1);
    assemble_System_Compressed(der, grid, *A, *F, *U_exact); 
#endif 
    tm[assemble_timer_name]->stop(); 

    if (!primeGPU) {
        //write_System(*A, *F, *U_exact); 
    }
    // ----- SOLVE -----

    tm[copy_timer_name]->start();

    DEVICE_MAT_t* A_gpu = new DEVICE_MAT_t(*A); 
    DEVICE_VEC_t* F_gpu = new DEVICE_VEC_t(*F); 
    DEVICE_VEC_t* U_exact_gpu = new DEVICE_VEC_t(*U_exact); 
    DEVICE_VEC_t* U_approx_gpu = new DEVICE_VEC_t(F->size(), 0);

    tm[copy_timer_name]->stop();

    tm[test_timer_name]->start();
    // Use GMRES to solve A*u = F
    GMRES_Device(*A_gpu, *F_gpu, *U_exact_gpu, *U_approx_gpu);
    tm[test_timer_name]->stop();

    if (!primeGPU) {
        write_Solution(grid, *U_exact, *U_approx_gpu); 
    }
    // Cleanup
    delete(A);
    delete(A_gpu);
    delete(F);
    delete(U_exact);
    delete(F_gpu);
    delete(U_exact_gpu);
    delete(U_approx_gpu);
}


int main(void)
{
    bool writeIntermediate = true; 
    bool primed = false; 

    std::vector<std::string> grids; 

    //grids.push_back("~/GRIDS/md/md005.00036"); 

//    grids.push_back("~/GRIDS/md/md165.27556"); 
#if 1 
    grids.push_back("~/GRIDS/md/md031.01024"); 
    grids.push_back("~/GRIDS/md/md050.02601"); 
    grids.push_back("~/GRIDS/md/md063.04096"); 
    grids.push_back("~/GRIDS/md/md089.08100"); 
    grids.push_back("~/GRIDS/md/md127.16384"); 
    grids.push_back("~/GRIDS/md/md165.27556"); 
#endif 
#if 0
    grids.push_back("~/GRIDS/geoff/scvtmesh_100k_nodes.ascii"); 
    grids.push_back("~/GRIDS/geoff/scvtmesh_500k_nodes.ascii"); 
    grids.push_back("~/GRIDS/geoff/scvtmesh_100k_nodes.ascii"); 
    grids.push_back("~/GRIDS/geoff/scvtmesh_500k_nodes.ascii"); 
    grids.push_back("~/GRIDS/geoff/scvtmesh_1m_nodes.ascii"); 
#endif 
    //grids.push_back("~/GRIDS/geoff/scvtmesh_1m_nodes.ascii"); 

    for (size_t i = 0; i < grids.size(); i++) {
        std::string& grid_name = grids[i]; 

        std::string weight_timer_name = grid_name + " Calc Weights";  

        tm[weight_timer_name] = new EB::Timer(weight_timer_name.c_str()); 

        // Get contours from rbfzone.blogspot.com to choose eps_c1 and eps_c2 based on stencil_size (n)
        unsigned int stencil_size = 40;
        double eps_c1 = 0.027;
        double eps_c2 = 0.274;


        GridReader* grid = new GridReader(grid_name, 4); 
        grid->setMaxStencilSize(stencil_size); 
        // We do not read until generate is called: 

        Grid::GridLoadErrType err = grid->loadFromFile(); 
        if (err == Grid::NO_GRID_FILES) 
        {
            grid->generate();
            // NOTE: We force at least one node in the domain to be a boundary. 
            //-----------------------------
            // We will set the first node as a boundary/ground point. We know
            // the normal because we're on teh sphere centered at (0,0,0)
            unsigned int nodeIndex = 0; 
            NodeType& node = grid->getNode(nodeIndex); 
            Vec3 nodeNormal = node - Vec3(0,0,0); 
            grid->appendBoundaryIndex(nodeIndex, nodeNormal); 
            //-----------------------------
            if (writeIntermediate) {
                grid->writeToFile(); 
            }
        } 
        std::cout << "Generate Stencils\n";
        Grid::GridLoadErrType st_err = grid->loadStencilsFromFile(); 
        if (st_err == Grid::NO_STENCIL_FILES) {
            //            grid->generateStencils(Grid::ST_BRUTE_FORCE);   
#if 1
            grid->generateStencils(Grid::ST_KDTREE);   
#else 
            grid->setNSHashDims(50, 50,50);  
            grid->generateStencils(Grid::ST_HASH);   
#endif 
            if (writeIntermediate) {
                grid->writeToFile(); 
            }
        }


        std::cout << "Generate RBFFD Weights\n"; 
        tm[weight_timer_name]->start(); 
        RBFFD der(RBFFD::LSFC | RBFFD::XSFC | RBFFD::YSFC | RBFFD::ZSFC, grid, 3, 0); 
        der.setEpsilonByParameters(eps_c1, eps_c2);
        int der_err = der.loadAllWeightsFromFile(); 
        if (der_err) {
            der.computeAllWeightsForAllStencils(); 
            tm[weight_timer_name]->stop(); 

            #if 0
            // Im finding that its more efficient to compute the weights than write and load from disk. 
            if (writeIntermediate) {
                der.writeAllWeightsToFile(); 
            }
            #endif 
        }

        if (!primed)  {
            std::cout << "\n\n"; 
            cout << "Priming GPU with dummy operations (removes compile from benchmarks)\n";
            gpuTest(der,*grid, 1);
            gpuTest(der,*grid, 1);
            primed = true; 
            std::cout << "\n\n"; 
        } 

        // No support for GMRES on the CPU yet. 
        //cpuTest(der,*grid);  
        gpuTest(der,*grid);  

        delete(grid); 
    }

    tm.printAll();
    tm.writeToFile();
    return EXIT_SUCCESS;
}

