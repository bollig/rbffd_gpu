#include "hip/hip_runtime.h"
#include <math.h>

#include "grid.h"
#include "ncar_poisson1_cusp.h"
#include "exact_solution.h"

#include <cusp/hyb_matrix.h>
#include <cusp/print.h>
using namespace std;


NCARPoisson1_CUSP::NCARPoisson1_CUSP(ExactSolution* _solution, GPU* subdomain_, Derivative* der_, int rank, int dim_num_) :
        NCARPoisson1(_solution, subdomain_, der_, rank, dim_num_)
{}

//----------------------------------------------------------------------

NCARPoisson1_CUSP::~NCARPoisson1_CUSP() {
}
//----------------------------------------------------------------------
// Solve the poisson system.
// NOTE: this routine is old and uses a possibly incorrect method for solving with the
// neumann boundary conditions. I am starting an alternate routine to solve the system
// in the same fashion that Joe solved the system.
void NCARPoisson1_CUSP::solve(Communicator* comm_unit) {

    if (subdomain == NULL) {
        cerr
                << "In " << __FILE__
                << " No GPU class passed to Constructor. Cannot perform intermediate communication/updates in solver."
                << endl;
        exit(EXIT_FAILURE);
    } else {

        int nb = subdomain->global_boundary_nodes.size();
        // All interior and boundary nodes are included in the stencils.
        // The first nb Q_stencils should be the global boundary nodes
        int ni = subdomain->Q_stencils.size() - nb;

        int nn = (nb + ni) ;
        double err_norm = 100;
        double prev_err_norm = 100;

        double left_eps = 0.1;
        double right_eps = 30.;
        double new_eps = left_eps;

        double prev_eps = left_eps;

        bool goodDirection, wentRight;
cout << "Allocating GPU arrays " <<endl;

        cusp::array1d<float, cusp::device_memory> exact(nn, 0);
        cusp::array1d<float, cusp::device_memory> approx_sol(nn, 0);
        cusp::array1d<float, cusp::device_memory> error(nn, 0);
        cusp::array1d<float, cusp::device_memory> expected(nn, 0);
        cusp::array1d<float, cusp::device_memory> diff_lapl(nn, 0);

        int iter = 0;
cout << "ENTERING EPSILON SEARCH LOOP " <<endl;
        while (err_norm > 1e-4 && iter < 10)
        {
            // TODO: solve this in parallel
            //     comm_unit->broadcastObjectUpdates(subdomain);

            // Do NOT use GPU as buffer for computation
            // Only go up to the number of stencils since we solve for a subset of the values in U_G
            // Since U_G in R is at end of U_G vector we can ignore those.
            //for (int i = 0; i < s.size(); i++) {
            //    s[i] = subdomain->U_G[i];
            //}

            //for (int i = 0; i < lapl_deriv.size(); i++) {
            //  Vec3& v = (*rbf_centers)[i];
            //    printf("(local: %d), lapl(%f,%f,%f)= %f\t%f\n", i, v.x(), v.y(), v.z(),
            //            lapl_deriv[i], s[i]);
            //}

            // Evan TODO:
            //
            // 1) Build a sparse matrix representation for all the interior derivative weights LA
            // 2) Build a full vector F = laplacian(u)
            // 3)

            // We are forming:
            //
            //  [ w_ddr -1/r*I  ]  [ A_boundary ]     [ 0  ]
            //  [ w_lapl    ]  [ A_interior ]  =  [ f_interior ]
            //
            // where w_lapl are the laplacian RBFFD weights for interior nodes (ni x nb + ni)
            // w_dr are the d/dr RBFFD weights (that is, the operator xd/dx +
            //

            // The w_lapl, weights for the laplacian, require (d^2 Phi / dx^2 + d^2 Phi / dy^2 + d^2 Phi / dz^2).
            // For w_ddr, weights for dA/dr, require (x*dPhi/dx + y*dPhi/dy + z*dPhi/dz). That means we will need to get
            // the stencil centers (Vec3) into the

            new_eps = left_eps + abs(right_eps - left_eps)/2.;

            der->setEpsilon(new_eps);
            cout << "USING EPSILON: " << new_eps << endl;

            for (int i = 0; i < nb + ni; i++) {
                //subdomain->printStencil(subdomain->Q_stencils[i], "Q[i]");
                // Compute all derivatives for our centers
                der->computeWeights(subdomain->G_centers, subdomain->Q_stencils[i], i, dim_num);
            }
#if 0
            // The derivative weights go into a matrix that is TotNumNodes x TotNumNodes
            // This is a sparse matrix though, so we're wasting memory and computation
            // TODO: replace this with a sparse solver
            arma::mat L(nn+1,nn+1);
            L.zeros();

            arma::colvec F(nn+1);
            F.zeros();

#if 1
            // This loop should add 1s to the far right column and bottom row; thereby removing the constant from the
            // possible solution and making the system nonsingular.
            for (int i = 0; i < nn; i++) {
                L(nn,i) = 1;
                L(i,nn) = 1;
            }
#else
            L(nn, nn) = 1;
#endif

            //    cout << "WARNING! using hardcoded constants for the boundaries!" << endl;
            //    cout << "WARNING! using x,y,z weights separately to compute d/dr!" << endl;
            // Block 1 (top left corner): d/dr weights for nb boundary points using nb+ni possible weights
            for (int i = 0; i < nb; i++) {
                //arma::mat& r_weights = der->getRWeights(subdomain->Q_stencils[i][0]);
                arma::mat& x_weights = der->getXWeights(subdomain->Q_stencils[i][0]);
                arma::mat& y_weights = der->getYWeights(subdomain->Q_stencils[i][0]);
                arma::mat& z_weights = der->getZWeights(subdomain->Q_stencils[i][0]);

                // DONT FORGET TO ADD IN THE -1/r on the stencil center weights
                Vec3& center = subdomain->G_centers[subdomain->Q_stencils[i][0]];
                double r = center.magnitude();
                //  r = 1.;
                for (int j = 0; j < subdomain->Q_stencils[i].size(); j++) {
                    //L(subdomain->Q_stencils[i][0],subdomain->Q_stencils[i][j]) = r_weights(j);        // Block 1 (weights for d/dr)
                    L(subdomain->Q_stencils[i][0],subdomain->Q_stencils[i][j]) = (center.x() / r) * x_weights(j) + (center.y()/r) * y_weights(j) + (center.z()/r) * z_weights(j);        // Block 1 (weights for d/dr)
                }

                if (r < 1e-8) {
                    cerr << "WARNING! VANISHING SPHERE RADIUS! CANNOT FILL -1/r in " << __FILE__ << endl;
                    exit(EXIT_FAILURE);
                }
                // Again, make sure we use Q_stencils[i][0] so we are forming the diagonals
                // correctly using the stencil center index (WARNING! this is not consistent for
                // domain decomposition... how to address this? TODO in the future..)
                L(subdomain->Q_stencils[i][0],subdomain->Q_stencils[i][0]) -= 1./r;
            }

            // Block 2 (bottom left corner): laplacian weights for ni interior points using nb+ni possible weights
            for (int i = 0; i < ni; i++) {
                int indx = i + nb; // offset into Q_stencils to get the interior stencils only
                arma::mat& l_weights = der->getLaplWeights(subdomain->Q_stencils[indx][0]);
                for (int j = 0; j < subdomain->Q_stencils[indx].size(); j++) {
                    L(subdomain->Q_stencils[indx][0],subdomain->Q_stencils[indx][j]) = l_weights(j);        // Block 1 (weights for laplacian)
                }
            }

            //L.print("L = ");

            for (int i = 0; i < ni; i ++) {
                int indx = i + nb;
                Vec3& v = subdomain->G_centers[subdomain->Q_stencils[indx][0]];
                F(subdomain->Q_stencils[indx][0]) = exactSolution->laplacian(v.x(), v.y(), v.z(), 0.);
            }
            F(nn) = 0.;

            //F.print("F = ");

            arma::mat sol = arma::solve(L,F);

            cout << "Measure sol(nn+1) = " << sol(nn) << endl;

            approx_sol = sol.rows(0,nn-1);

            // Get the subset of our full solution that corresponds to the solution we need
            // arma::mat A_sol = A.rows(0,nb+ni-1);

            // Fill our exact solution
            exact.zeros();
            for (int i = 0; i < nb + ni; i++) {
                exact(subdomain->Q_stencils[i][0]) = exactSolution->at(subdomain->G_centers[subdomain->Q_stencils[i][0]], 0.);
            }

            // Compute our errors
            error = (approx_sol - exact);

            //        expected = L*exact;
            //        diff_lapl = expected - F;

            prev_err_norm = err_norm;
            err_norm = this->maxNorm(error);
            cout << "INF NORM (ERROR) : " << err_norm << endl;

#endif
            if (prev_err_norm > err_norm) {
                goodDirection = true;
            } else {
                goodDirection = false;
            }

            if (goodDirection) {
                if (wentRight) {
                    prev_eps = left_eps;
                    left_eps = new_eps;
                    wentRight = true;
                } else {
                    prev_eps = right_eps;
                    right_eps = new_eps;
                    wentRight = false;
                }
            } else {
                if (wentRight) {
                    left_eps = prev_eps;
                    right_eps = new_eps;
                    wentRight = false;
                } else {
                    right_eps = prev_eps;
                    left_eps = new_eps;
                    wentRight = true;
                }
            }

            iter ++;
        }

#if 0
        cusp::array1d<float, cusp::host_memory> results[5](nn);

        results[0] = approx_sol;
        results[1] = exact;
        results[2] = error;
         results.print("\n\nRESULTS\n (APPROX SOLUTION; \tEXACT SOLUTION; \tABS ERROR \tExpected Laplacian(Using L*ExactSolution)\t Diff EXPECTED & EXACT Laplacian\n");
#endif
        // results.col(3) = expected;
        // results.col(4) = diff_lapl;
        cusp::print_matrix(approx_sol);

        //A.print("Full Solution (A) = ");
        //exact.print("Exact = ");
        //error.print("Error = ");


        cout.flush();
    }
    return;
}

//----------------------------------------------------------------------
