#include "hip/hip_runtime.h"
// TODO : test this: 
//#define CUSP_USE_TEXTURE_MEMORY

// THIS IS adapted from verbose_monitor.cu
// PROVIDED BY THE CUSP v0.1 EXAMPLES

#include "grids/grid_reader.h"
#include "rbffd/rbffd.h"
#include "timer_eb.h" 

#include <cusp/hyb_matrix.h>
#include <cusp/ell_matrix.h>
#include <cusp/csr_matrix.h>
#include <cusp/coo_matrix.h>
#include <cusp/monitor.h>
#include <cusp/krylov/cg.h>
#include <cusp/krylov/gmres.h>
#include <cusp/gallery/poisson.h>
#include <cusp/print.h>
#include <cusp/array2d.h>
#include <cusp/multiply.h>
#include <cusp/blas.h>
#include <cusp/io/matrix_market.h>
#include <cusp/precond/diagonal.h>
#include <cusp/precond/ainv.h>
#include <cusp/precond/smoothed_aggregation.h>
#include <cusp/precond/aggregate.h>
#include <cusp/precond/smooth.h>
#include <cusp/precond/strength.h>

#include <thrust/host_vector.h> 
#include <thrust/device_vector.h>
#include <thrust/generate.h>


#include "utils/spherical_harmonics.h"

#include <iomanip>
#include <iostream>
#include <sstream> 
#include <map>
#include <fstream> 
#include <typeinfo> 
using namespace std;


typedef std::vector< std::map< unsigned int, double> > STL_MAT_t; 
typedef std::vector<double> STL_VEC_t; 


typedef cusp::array1d<double, cusp::host_memory> HOST_VEC_t; 
typedef cusp::array1d<double, cusp::device_memory> DEVICE_VEC_t; 
typedef cusp::csr_matrix<unsigned int, double, cusp::host_memory> HOST_MAT_t; 
typedef cusp::csr_matrix<unsigned int, double, cusp::device_memory> DEVICE_MAT_t; 

EB::TimerList timers;

//---------------------------------

// Perform GMRES on GPU
void GMRES_Device(DEVICE_MAT_t& A, DEVICE_VEC_t& F, DEVICE_VEC_t& U_exact, DEVICE_VEC_t& U_approx_gpu) {
#if 1
    size_t restart = 300; 
    int max_iters = 1000; 
    double rel_tol = 1e-6; 
#else 
    // Maximum number of iterations (total) 
    size_t max_iters = 500; 
    // restart the process every "restart" iterations
    size_t restart = 200; 
    double rel_tol = 1e-8; 
#endif 

    try {

        //    cusp::convergence_monitor<double> monitor( F, max_iters, 0, 1e-3); 
        cusp::default_monitor<double> monitor( F, max_iters, rel_tol ); //, max_iters, rel_tol);// , 1e-3); 
        //cusp::default_monitor<double> monitor( F, -1, rel_tol ); //, max_iters, rel_tol);// , 1e-3); 

        std::cout << "GMRES Starting Residual Norm: " << monitor.residual_norm() << std::endl;

        // 1e-8, 10000, 300); 
        int precondType = -1; 
        switch (precondType) {
            case 0: 
                {
                    // Jacobi Preconditioning (DIAGONAL)
                    // Probably wont work well for RBF-FD since we're not diagonally dominant
                    cusp::precond::diagonal<double, cusp::device_memory> M(A);
                    cusp::krylov::gmres(A, U_approx_gpu, F, restart, monitor, M); 
                }
                break;
            case 1:
                {
                    // Smoothed Aggregation (Algebraic MultiGrid. Works for Nonsym?)
                    cusp::precond::smoothed_aggregation<int, double, cusp::device_memory> M(A);
                    cusp::krylov::gmres(A, U_approx_gpu, F, restart, monitor, M); 
                }
                break; 
#if 0 
                // ONLY SPD MATRICES
            case 0: 
                // AINV using static dropping
                cusp::precond::scaled_bridson_ainv<double, cusp::device_memory> M(A, 0, 10);
                cusp::krylov::gmres(A, U_approx_gpu, F, restart, monitor, M); 
                break; 
#endif 
#if 0 
                // ONLY SPD MATRICES
            case 1: 
                // AINV using standard drop tolerance
                cusp::precond::scaled_bridson_ainv<double, cusp::device_memory> M(A, .1);
                cusp::krylov::gmres(A, U_approx_gpu, F, restart, monitor, M); 
                break; 
#endif 
#if 0
                // ONLY FOR SPD MATRICES
            case 2: 
                // AINV using novel cusp dropping strategy (TODO: lookup) 
                cusp::precond::bridson_ainv<double, cusp::device_memory> M(A, 0, -1, true, 2);
                cusp::krylov::gmres(A, U_approx_gpu, F, restart, monitor, M); 
#endif 
            case 2: 
                {
                    // AINV using novel cusp dropping strategy 
                    // assumes that sparsity pattern of precond is same as A, plus
                    // 2 extra nonzeros per row 
                    // VERY SLOW TO BUILD; DOES NOT CONVERGE
                    cusp::precond::nonsym_bridson_ainv<double, cusp::device_memory> M(A, 0, -1, true, 2);
                    cusp::krylov::gmres(A, U_approx_gpu, F, restart, monitor, M); 
                }
            case 3: 
                {
                    // AINV using novel cusp dropping strategy 
                    // Assume 40 nonzeros per row, drop everthing else. 
                    // VERY SLOW TO BUILD; DOES NOT CONVERGE
                    cusp::precond::nonsym_bridson_ainv<double, cusp::device_memory> M(A, 0.1, 10, false, 0);
                    cusp::krylov::gmres(A, U_approx_gpu, F, restart, monitor, M); 
                }
            default: 
                // Solve unpreconditioned Au = F
                cusp::krylov::gmres(A, U_approx_gpu, F, restart, monitor); 
        }
        hipDeviceSynchronize(); 

        //    monitor.print();

        if (monitor.converged())
        {
            std::cout << "\n[+++] Solver converged to " << monitor.relative_tolerance() << " relative tolerance";       
            std::cout << " after " << monitor.iteration_count() << " iterations" << std::endl << std::endl;
        }
        else
        {
            std::cout << "\n[XXX] Solver reached iteration limit " << monitor.iteration_limit() << " before converging";
            std::cout << " to " << monitor.relative_tolerance() << " relative tolerance " << std::endl << std::endl;
        }

        std::cout << "GMRES Iterations: " << monitor.iteration_count() << std::endl;
        std::cout << "GMRES Iteration Limit: " << monitor.iteration_limit() << std::endl;
        std::cout << "GMRES Residual Norm: " << monitor.residual_norm() << std::endl;
        std::cout << "GMRES Relative Tol: " << monitor.relative_tolerance() << std::endl;
        std::cout << "GMRES Absolute Tol: " << monitor.absolute_tolerance() << std::endl;
        std::cout << "GMRES Target Residual (Abs + Rel*norm(F)): " << monitor.tolerance() << std::endl;
    }
    catch(std::bad_alloc &e)
    {
        std::cerr << "Ran out of memory trying to compute GMRES: " << e.what() << std::endl;
        exit(-1);
    }
    catch(thrust::system_error &e)
    {
        std::cerr << "Some other error happened during GMRES: " << e.what() << std::endl;
        exit(-1);
    }


    try {

        typedef cusp::array1d<double, DEVICE_VEC_t>::view DEVICE_VEC_VIEW_t; 

        DEVICE_VEC_VIEW_t U_approx_view(U_exact.begin()+(U_exact.size() - F.size()), U_exact.end()); 

        DEVICE_VEC_t diff(U_approx_gpu); 

        //cusp::blas::axpy(U_exact.begin()+(U_exact.size() - F.size()), U_exact.end(), diff.begin(),  -1); 
        cusp::blas::axpy(U_approx_view, diff, -1); 

        std::cout << "Rel l1   Norm: " << cusp::blas::nrm1(diff) / cusp::blas::nrm1(U_exact) << std::endl;  
        std::cout << "Rel l2   Norm: " << cusp::blas::nrm2(diff) / cusp::blas::nrm2(U_exact) << std::endl;  
        std::cout << "Rel linf Norm: " << cusp::blas::nrmmax(diff) / cusp::blas::nrmmax(U_exact) << std::endl;  
    }
    catch(std::bad_alloc &e)
    {
        std::cerr << "Ran out of memory trying to compute Error Norms: " << e.what() << std::endl;
        exit(-1);
    }
    catch(thrust::system_error &e)
    {
        std::cerr << "Some other error happened during Error Norms: " << e.what() << std::endl;
        exit(-1);
    }
}

//---------------------------------

void assemble_System_Stokes( RBFFD& der, Grid& grid, HOST_MAT_t& A, HOST_VEC_t& F, HOST_VEC_t& U_exact){
    double eta = 1.;
    //double Ra = 1.e6;

    // We have different nb_stencils and nb_nodes when we parallelize. The subblocks might not be full
    unsigned int nb_stencils = grid.getStencilsSize();
    unsigned int nb_nodes = grid.getNodeListSize(); 
    unsigned int max_stencil_size = grid.getMaxStencilSize();
    unsigned int N = nb_nodes;
    // ---------------------------------------------------

    //------------- Fill the RHS of the System -------------
    // This is our manufactured solution:
    SphericalHarmonic::Sph32 UU; 
    SphericalHarmonic::Sph32105 VV; 
    SphericalHarmonic::Sph32 WW; 
    SphericalHarmonic::Sph32 PP; 

    std::vector<NodeType>& nodes = grid.getNodeList(); 

    //------------- Fill F -------------

    // U
    for (unsigned int j = 0; j < N; j++) {
        unsigned int row_ind = j + 0*N;
        NodeType& node = nodes[j]; 
        double Xx = node.x(); 
        double Yy = node.y(); 
        double Zz = node.z(); 

        U_exact[row_ind] = UU.eval(Xx,Yy,Zz); 
        F[row_ind] = -UU.lapl(Xx,Yy,Zz) + PP.d_dx(Xx,Yy,Zz);  
    }
#if 1

    // V
    for (unsigned int j = 0; j < N; j++) {
        unsigned int row_ind = j + 1*N;
        NodeType& node = nodes[j]; 
        double Xx = node.x(); 
        double Yy = node.y(); 
        double Zz = node.z(); 
        //double rr = sqrt(node.x()*node.x() + node.y()*node.y() + node.z()*node.z());
        //double dir = node.y();

        // F[row_ind] = (Ra * Temperature(j) * dir) / rr;  
        U_exact[row_ind] = VV.eval(Xx,Yy,Zz); 
        F[row_ind] = -VV.lapl(Xx,Yy,Zz) + PP.d_dy(Xx,Yy,Zz);  
    }

    // W
    for (unsigned int j = 0; j < N; j++) {
        unsigned int row_ind = j + 2*N;
        NodeType& node = nodes[j];
        double Xx = node.x(); 
        double Yy = node.y(); 
        double Zz = node.z(); 

        U_exact[row_ind] = WW.eval(Xx,Yy,Zz); 
        F[row_ind] = -WW.lapl(Xx,Yy,Zz) + PP.d_dz(Xx,Yy,Zz);  
    }

    // P
    for (unsigned int j = 0; j < N; j++) {
        unsigned int row_ind = j + 3*N;
        NodeType& node = nodes[j]; 
        double Xx = node.x(); 
        double Yy = node.y(); 
        double Zz = node.z(); 

        U_exact[row_ind] = PP.eval(Xx,Yy,Zz); 
        F[row_ind] = UU.d_dx(Xx,Yy,Zz) + VV.d_dy(Xx,Yy,Zz) + WW.d_dz(Xx,Yy,Zz);  
    }
#endif
    // Sum of U
    F[4*N+0] = 0.;

    // Sum of V
    F[4*N+1] = 0.;

    // Sum of W
    F[4*N+2] = 0.;

    // Sum of P
    F[4*N+3] = 0.;
 



    unsigned int ind = 0; 

    // -----------------  Fill LHS --------------------
    //
    // U (block)  row
    for (unsigned int i = 0; i < nb_stencils; i++) {
        StencilType& st = grid.getStencil(i);

        // TODO: change these to *SFC weights (when computed)
        double* ddx = der.getStencilWeights(RBFFD::XSFC, i);
        double* lapl = der.getStencilWeights(RBFFD::LSFC, i); 

        unsigned int diag_row_ind = i + 0*N;
        
        A.row_offsets[diag_row_ind] = ind; 


        for (unsigned int j = 0; j < st.size(); j++) {
            unsigned int diag_col_ind = st[j] + 0*N;
            
            A.column_indices[ind] = diag_col_ind; 
            A.values[ind]  = -eta * lapl[j];  
            ind++; 
        }
        for (unsigned int j = 0; j < st.size(); j++) {
            unsigned int diag_col_ind = st[j] + 3*N;

            A.column_indices[ind] = diag_col_ind; 
            A.values[ind]  = ddx[j];  
            ind++; 
        }

        // Added constraint to square mat and close nullspace
        A.column_indices[ind] = 4*N+0; 
        A.values[ind]  = 1;  
        ind++; 
    }

    // V (block)  row
    for (unsigned int i = 0; i < nb_stencils; i++) {
        StencilType& st = grid.getStencil(i);

        // TODO: change these to *SFC weights (when computed)
        double* ddy = der.getStencilWeights(RBFFD::YSFC, i);
        double* lapl = der.getStencilWeights(RBFFD::LSFC, i); 

        unsigned int diag_row_ind = i + 1*N;
        A.row_offsets[diag_row_ind] = ind; 

        for (unsigned int j = 0; j < st.size(); j++) {
            unsigned int diag_col_ind = st[j] + 1*N;

            A.column_indices[ind] = diag_col_ind; 
            A.values[ind]  = -eta * lapl[j];  
            ind++; 
        }
        for (unsigned int j = 0; j < st.size(); j++) {
            unsigned int diag_col_ind = st[j] + 3*N;

            A.column_indices[ind] = diag_col_ind; 
            A.values[ind]  = ddy[j];  
            ind++; 
        }

        // Added constraint to square mat and close nullspace
        A.column_indices[ind] = 4*N+1; 
        A.values[ind]  = 1;  
        ind++; 
    }

    // W (block)  row
    for (unsigned int i = 0; i < nb_stencils; i++) {
        StencilType& st = grid.getStencil(i);

        // TODO: change these to *SFC weights (when computed)
        double* ddz = der.getStencilWeights(RBFFD::ZSFC, i);
        double* lapl = der.getStencilWeights(RBFFD::LSFC, i); 

        unsigned int diag_row_ind = i + 2*N;
        A.row_offsets[diag_row_ind] = ind; 

        for (unsigned int j = 0; j < st.size(); j++) {
            unsigned int diag_col_ind = st[j] + 2*N;

            A.column_indices[ind] = diag_col_ind; 
            A.values[ind]  = -eta * lapl[j];  
            ind++; 
        }
        for (unsigned int j = 0; j < st.size(); j++) {
            unsigned int diag_col_ind = st[j] + 3*N;

            A.column_indices[ind] = diag_col_ind; 
            A.values[ind]  = ddz[j];
            ind++; 
        }

        // Added constraint to square mat and close nullspace
        A.column_indices[ind] = 4*N+2; 
        A.values[ind]  = 1;  
        ind++; 
    }


    // P (block)  row
    for (unsigned int i = 0; i < nb_stencils; i++) {
        StencilType& st = grid.getStencil(i);

        // TODO: change these to *SFC weights (when computed)
        double* ddx = der.getStencilWeights(RBFFD::XSFC, i);
        double* ddy = der.getStencilWeights(RBFFD::YSFC, i);
        double* ddz = der.getStencilWeights(RBFFD::ZSFC, i);

        unsigned int diag_row_ind = i + 3*N;
        A.row_offsets[diag_row_ind] = ind; 

        for (unsigned int j = 0; j < st.size(); j++) {
            unsigned int diag_col_ind = st[j] + 0*N;

            A.column_indices[ind] = diag_col_ind; 
            A.values[ind]  = ddx[j]; 
            ind++; 
        }
        for (unsigned int j = 0; j < st.size(); j++) {
            unsigned int diag_col_ind = st[j] + 1*N;

            A.column_indices[ind] = diag_col_ind; 
            A.values[ind]  = ddy[j]; 
            ind++; 
        }
        for (unsigned int j = 0; j < st.size(); j++) {
            unsigned int diag_col_ind = st[j] + 2*N;

            A.column_indices[ind] = diag_col_ind; 
            A.values[ind]  = ddz[j]; 
            ind++; 
        }

        // Added constraint to square mat and close nullspace
        A.column_indices[ind] = 4*N+3; 
        A.values[ind]  = 1;  
        ind++; 
    }

    // ------ EXTRA CONSTRAINT ROWS -----
    unsigned int diag_row_ind = 4*N;
    A.row_offsets[diag_row_ind] = ind;
    // U
    for (unsigned int j = 0; j < N; j++) {
        unsigned int diag_col_ind = j + 0*N;

        A.column_indices[ind] = diag_col_ind; 
        A.values[ind]  = 1;  
        ind++; 
    }

    diag_row_ind++; 
    A.row_offsets[diag_row_ind] = ind; 
    // V
    for (unsigned int j = 0; j < N; j++) {
        unsigned int diag_col_ind = j + 1*N;

        A.column_indices[ind] = diag_col_ind; 
        A.values[ind]  = 1;  
        ind++; 
    }

    diag_row_ind++; 
    A.row_offsets[diag_row_ind] = ind; 
    // W
    for (unsigned int j = 0; j < N; j++) {
        unsigned int diag_col_ind = j + 2*N;

        A.column_indices[ind] = diag_col_ind; 
        A.values[ind]  = 1;  
        ind++; 
    }

    diag_row_ind++; 
    A.row_offsets[diag_row_ind] = ind; 
    // P
    for (unsigned int j = 0; j < N; j++) {
        unsigned int diag_col_ind = j + 3*N;

        A.column_indices[ind] = diag_col_ind; 
        A.values[ind]  = 1;  
        ind++; 
    }

    // VERY IMPORTANT. UNSPECIFIED LAUNCH FAILURES ARE CAUSED BY FORGETTING THIS!
    A.row_offsets[4*N+4] = ind; 
}




    template <typename VecT>
void write_to_file(VecT vec, std::string filename)
{
    std::ofstream fout;
    fout.open(filename.c_str());
    for (size_t i = 0; i < vec.size(); i++) {
        fout << std::setprecision(10) << vec[i] << std::endl;
    }
    fout.close();
    std::cout << "Wrote " << filename << std::endl;
}


void write_System ( HOST_MAT_t& A, HOST_VEC_t& F, HOST_VEC_t& U_exact )
{
    write_to_file(F, "output/F.mtx"); 
    write_to_file(U_exact, "output/U_exact.mtx"); 
    cusp::io::write_matrix_market_file(A,"output/LHS.mtx"); 
}

void write_Solution( Grid& grid, HOST_VEC_t& U_exact, DEVICE_VEC_t& U_approx_gpu ) 
{
    unsigned int nb_bnd = grid.getBoundaryIndicesSize();

    // IF we want to write details we need to copy back to host. 
    HOST_VEC_t U_approx(U_exact.size());

    if (U_approx_gpu.size() == U_exact.size()) {
        thrust::copy(U_approx_gpu.begin(), U_approx_gpu.end(), U_approx.begin());
    } else {
        thrust::copy(U_exact.begin(), U_exact.begin()+nb_bnd, U_approx.begin());
        thrust::copy(U_approx_gpu.begin(), U_approx_gpu.end(), U_approx.begin()+nb_bnd);
    }

    write_to_file(U_approx, "output/U_gpu.mtx"); 
}


//---------------------------------

void gpuTest(RBFFD& der, Grid& grid, int primeGPU=0) {
    unsigned int N = grid.getNodeListSize(); 
    unsigned int n = grid.getMaxStencilSize(); 
    unsigned int nrows = 4 * N + 4; 
    unsigned int ncols = 4 * N + 4; 
    unsigned int NNZ = 9*n*N+2*(4*N)+2*(3*N);  
 
    char test_name[256]; 
    char assemble_timer_name[256]; 
    char copy_timer_name[512]; 
    char test_timer_name[256]; 

    if (primeGPU) {
        sprintf(test_name, "%u PRIMING THE GPU", N);  
        sprintf(assemble_timer_name, "%u Primer Assemble", N);
        sprintf(copy_timer_name,     "%u Primer Copy To CUSP_DEVICE_CSR", N); 
        sprintf(test_timer_name, "%u Primer GMRES test", N); 
    } else { 
        sprintf(test_name, "%u GMRES GPU (CUSP_DEVICE_CSR)", N);  
        sprintf(assemble_timer_name, "%u CUSP_HOST_CSR Assemble", N);
        sprintf(copy_timer_name,     "%u CUSP_HOST_CSR Copy To CUSP_DEVICE_CSR", N); 
        sprintf(test_timer_name, "%u GPU GMRES test", N); 
    }

    if (!timers.contains(assemble_timer_name)) { timers[assemble_timer_name] = new EB::Timer(assemble_timer_name); } 
    if (!timers.contains(copy_timer_name)) { timers[copy_timer_name] = new EB::Timer(copy_timer_name); } 
    if (!timers.contains(test_timer_name)) { timers[test_timer_name] = new EB::Timer(test_timer_name); } 


    std::cout << test_name << std::endl;


    // ----- ASSEMBLE -----
    timers[assemble_timer_name]->start(); 
    HOST_MAT_t* A = new HOST_MAT_t(nrows, ncols, NNZ); 
    HOST_VEC_t* F = new HOST_VEC_t(nrows, 0);
    HOST_VEC_t* U_exact = new HOST_VEC_t(nrows, 0);
    assemble_System_Stokes(der, grid, *A, *F, *U_exact); 
    
    timers[assemble_timer_name]->stop(); 

    if (!primeGPU) {
        //write_System(*A, *F, *U_exact); 
    }
    // ----- SOLVE -----

    timers[copy_timer_name]->start();

    DEVICE_MAT_t* A_gpu = new DEVICE_MAT_t(*A); 
    DEVICE_VEC_t* F_gpu = new DEVICE_VEC_t(*F); 
    DEVICE_VEC_t* U_exact_gpu = new DEVICE_VEC_t(*U_exact); 
    DEVICE_VEC_t* U_approx_gpu = new DEVICE_VEC_t(F->size(), 0);

    timers[copy_timer_name]->stop();

    timers[test_timer_name]->start();
    // Use GMRES to solve A*u = F
    GMRES_Device(*A_gpu, *F_gpu, *U_exact_gpu, *U_approx_gpu);
    timers[test_timer_name]->stop();

    if (!primeGPU) {
        write_Solution(grid, *U_exact, *U_approx_gpu); 
    }
    // Cleanup
    delete(A);
    delete(A_gpu);
    delete(F);
    delete(U_exact);
    delete(F_gpu);
    delete(U_exact_gpu);
    delete(U_approx_gpu);
}


int main(void)
{
    bool writeIntermediate = true; 
    bool primed = false; 

    std::vector<std::string> grids; 

    //grids.push_back("~/GRIDS/md/md005.00036"); 

    //    grids.push_back("~/GRIDS/md/md165.27556"); 
    //grids.push_back("~/GRIDS/md/md063.04096"); 
#if 1 
    grids.push_back("~/GRIDS/md/md031.01024"); 
    grids.push_back("~/GRIDS/md/md050.02601"); 
    grids.push_back("~/GRIDS/md/md063.04096"); 
    grids.push_back("~/GRIDS/md/md089.08100"); 
    grids.push_back("~/GRIDS/md/md127.16384"); 
    grids.push_back("~/GRIDS/md/md165.27556"); 
#endif 
#if 0
    grids.push_back("~/GRIDS/geoff/scvtimersesh_100k_nodes.ascii"); 
    grids.push_back("~/GRIDS/geoff/scvtimersesh_500k_nodes.ascii"); 
    grids.push_back("~/GRIDS/geoff/scvtimersesh_100k_nodes.ascii"); 
    grids.push_back("~/GRIDS/geoff/scvtimersesh_500k_nodes.ascii"); 
    grids.push_back("~/GRIDS/geoff/scvtimersesh_1m_nodes.ascii"); 
#endif 
    //grids.push_back("~/GRIDS/geoff/scvtimersesh_1m_nodes.ascii"); 

    for (size_t i = 0; i < grids.size(); i++) {
        std::string& grid_name = grids[i]; 

        std::string weight_timer_name = grid_name + " Calc Weights";  

        timers[weight_timer_name] = new EB::Timer(weight_timer_name.c_str()); 

        // Get contours from rbfzone.blogspot.com to choose eps_c1 and eps_c2 based on stencil_size (n)
        unsigned int stencil_size = 40;
        double eps_c1 = 0.027;
        double eps_c2 = 0.274;


        GridReader* grid = new GridReader(grid_name, 4); 
        grid->setMaxStencilSize(stencil_size); 
        // We do not read until generate is called: 

        Grid::GridLoadErrType err = grid->loadFromFile(); 
        if (err == Grid::NO_GRID_FILES) 
        {
            grid->generate();
            // NOTE: We force at least one node in the domain to be a boundary. 
            //-----------------------------
            // We will set the first node as a boundary/ground point. We know
            // the normal because we're on teh sphere centered at (0,0,0)
            unsigned int nodeIndex = 0; 
            NodeType& node = grid->getNode(nodeIndex); 
            Vec3 nodeNormal = node - Vec3(0,0,0); 
            grid->appendBoundaryIndex(nodeIndex, nodeNormal); 
            //-----------------------------
            if (writeIntermediate) {
                grid->writeToFile(); 
            }
        } 
        std::cout << "Generate Stencils\n";
        Grid::GridLoadErrType st_err = grid->loadStencilsFromFile(); 
        if (st_err == Grid::NO_STENCIL_FILES) {
            //            grid->generateStencils(Grid::ST_BRUTE_FORCE);   
#if 1
            grid->generateStencils(Grid::ST_KDTREE);   
#else 
            grid->setNSHashDims(50, 50,50);  
            grid->generateStencils(Grid::ST_HASH);   
#endif 
            if (writeIntermediate) {
                grid->writeToFile(); 
            }
        }


        std::cout << "Generate RBFFD Weights\n"; 
        timers[weight_timer_name]->start(); 
        RBFFD der(RBFFD::LSFC | RBFFD::XSFC | RBFFD::YSFC | RBFFD::ZSFC, grid, 3, 0); 
        der.setEpsilonByParameters(eps_c1, eps_c2);
        int der_err = der.loadAllWeightsFromFile(); 
        if (der_err) {
            der.computeAllWeightsForAllStencils(); 
            timers[weight_timer_name]->stop(); 

#if 0
            // Im finding that its more efficient to compute the weights than write and load from disk. 
            if (writeIntermediate) {
                der.writeAllWeightsToFile(); 
            }
#endif 
        }

        if (!primed)  {
            std::cout << "\n\n"; 
            cout << "Priming GPU with dummy operations (removes compile from benchmarks)\n";
            gpuTest(der,*grid, 1);
            primed = true; 
            std::cout << "\n\n"; 
        } 

        // No support for GMRES on the CPU yet. 
        //cpuTest(der,*grid);  
        gpuTest(der,*grid);  

        delete(grid); 
    }

    timers.printAll();
    timers.writeToFile();
    return EXIT_SUCCESS;
}

