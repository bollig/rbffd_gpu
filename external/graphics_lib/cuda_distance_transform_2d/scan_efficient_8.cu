#include "hip/hip_runtime.h"
#ifndef _SCAN_WORKEFFICIENT_KERNEL_8_H_
#define _SCAN_WORKEFFICIENT_KERNEL_8_H_
#
#include "local_macros.h"

#define TIDX (__mul24(blockIdx.x,blockDim.x) + threadIdx.x)
#define TIDY (__mul24(blockIdx.y,blockDim.y) + threadIdx.y)
#define TWIDTH  (__mul24(gridDim.x,blockDim.x))
#define THEIGHT (__mul24(gridDim.y,blockDim.y))
#define ArrayID (TIDY*TWIDTH+TIDX)
#define MAKE_FLOAT4(arg) make_float4((arg), (arg), (arg), (arg))
#define MAKE_INT4(arg) make_int4((arg).x, (arg).y, (arg).z, (arg).w);

// Written by NVidia
// Modified by Gordon Erlebacher, Feb. 21, 2008

//----------------------------------------------------------------------
__global__ void scan_workefficient_8(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width)
// Version working with float4's again. Using floats is rather difficult
// hardcoded for 16x16 tiles (minimum size), with 256 threads per block
{
// edge=16, 64 threads: scan each row, one float per thread

    // Dynamically allocated shared memory for scan kernels
    extern  __shared__  float4 temp[];

	int numThreads = blockDim.x * blockDim.y;

	//if (blockIdx.x != 11) return; // block 13 has serial errors

	#if 1
	int last_share = edge*edge + ((edge*edge) >> LOG_NUM_BANKS);
	if (threadIdx.x == 0) {
		int4 ss = seeds[blockIdx.x];
		temp[last_share] = make_float4(ss.x+0.1,ss.y+0.1,ss.z+0.1,ss.w+0.1);
		//TMP(last_share) = make_float4(ss.x+0.1,ss.y+0.1,ss.z+0.1,ss.w+0.1);
	}
	__syncthreads();
	int4 seed = MAKE_INT4(temp[last_share]); // is int correct? Or must add 0.5?
	__syncthreads();
	#endif


	// get data from global memory (should be coalesced)

	int x = seed.x; 
	int y = seed.y;

	// edge should be part of the seed datastructure (per block)
	int edge2 = edge >> 2;
	int xorig = x - edge2; // one thread per float (edge float4's)
	int yorig = y - edge2; // edge rows

	// align xorig such that xorig is a multiple of 2 (speedup is evident)
	int shift = xorig - ((xorig >> 2) << 2);
	if (shift == 1) xorig -= 1;
	else if (shift == 2) xorig += 2;
	else if (shift == 3) xorig += 1;


	int flag1;
	float widthi = 1./width;

	int WW = width;  // array width (argument) (in float4)
	int HH = WW; // height of flat texture // MUST READ AS ARGUMENT

	int thread_id = threadIdx.x + blockDim.x * threadIdx.y;

	//--------------------
	// one iteration per row in the square tile
	// process 4 rows at a time

	__syncthreads();

		flag1 = 1;

		int xid = xorig + threadIdx.x; // xorig + 0...15
		if (xid < 0 || xid >= WW) flag1 = 0; 

		int yid1 = yorig + threadIdx.y;
		if (yid1 < 0 || yid1 >= HH) flag1 = 0;

 		int arrayid = xid + yid1*WW; 
		//temp[thread_id].x = float(xid*widthi);
		TMP(thread_id).x = float(xid*widthi);
		//temp[thread_id].y = float(yid1*widthi);
		TMP(thread_id).y = float(yid1*widthi);

		#if 1
		if (flag1 == 0) {
			//temp[thread_id] = make_float4(0.,0.,0.,0.);
			TMP(thread_id) = make_float4(0.,0.,0.,0.);
		} else {
			float f = g_idata[arrayid].w; // ERROR
			if (int(f+.1) != seed.w) {
				//temp[thread_id] = make_float4(0.,0.,0.,0.);
				TMP(thread_id) = make_float4(0.,0.,0.,0.);
			} else {
				//temp[thread_id].w = 1.0;
				TMP(thread_id).w = 1.0;
			}
		}
		#endif

	__syncthreads();


	#if 0
		sum[thread_id] = temp[thread_id];
		return;
	#endif

//	//--------------------
//
//// xorig - edge/2, xorig + edge/2 - 1
//
//// For the 16x16 case (hardcoded), the first pass with 64 threads can 
//// only handle 1/2 the domain (1024 floats = 16x16x4). The for loop that
//// follows had a thread handle two floats at a time, so can only handl
//// 1/2 the domain on each pass
//// manually treat each half of the domain

    int offset = 1;
	int ai, bi;
	int sz;

	#if 1
	sz = 128;  // hardcoded for 16x16=256 tile
    // build the sum in place up the tree
    for (int d = sz; d > 0; d >>= 1) {
        __syncthreads();

        if (thread_id < d) {
            ai = offset*(2*thread_id+1)-1;
            bi = offset*(2*thread_id+2)-1;

			//temp[bi].x += temp[ai].x;
			//temp[bi].y += temp[ai].y;
			//temp[bi].w += temp[ai].w;

			TMP(bi).x += TMP(ai).x;
			TMP(bi).y += TMP(ai).y;
			TMP(bi).w += TMP(ai).w;
        }

        offset <<= 1;
    }
	#endif

	__syncthreads();
	#if 0
		sum[thread_id] = temp[thread_id];
	return;
	#endif

	// Something wrong with the results

    // write results to global memory
    __syncthreads();
	if (threadIdx.x == (blockDim.x-1)) {
		int el = edge*edge-1;
		//float nbs = temp[el].w;
		float nbs = TMP(el).w;
		float nbs1 = 1./(nbs);
		if (nbs <= 0.5) nbs1 = 1.;
		//sum[blockIdx.x] = make_float4(temp[el].x*nbs1, temp[el].y*nbs1, 0., nbs); // orig
		sum[blockIdx.x] = make_float4(TMP(el).x*nbs1, TMP(el).y*nbs1, 0., nbs); // orig
	}
}
//----------------------------------------------------------------------
__global__ void scan_workefficient_8_larger(float4 *g_idata, float4 *sum, int4* seeds, int n, int edge, int width)
// Version working with float4's again. Using floats is rather difficult
// hardcoded for 16x16 tiles (minimum size), with 256 threads per block
{
// edge=16, 64 threads: scan each row, one float per thread

    // Dynamically allocated shared memory for scan kernels
    extern  __shared__  float4 temp[];

	int numThreads = blockDim.x * blockDim.y;

	//if (blockIdx.x != 11) return; // block 13 has serial errors

	#if 1
	int last_share = edge*edge + ((edge*edge) >> LOG_NUM_BANKS);
	if (threadIdx.x == 0) {
		int4 ss = seeds[blockIdx.x];
		temp[last_share] = make_float4(ss.x+0.1,ss.y+0.1,ss.z+0.1,ss.w+0.1);
		//TMP(last_share) = make_float4(ss.x+0.1,ss.y+0.1,ss.z+0.1,ss.w+0.1);
	}
	__syncthreads();
	int4 seed = MAKE_INT4(temp[last_share]); // is int correct? Or must add 0.5?
	__syncthreads();
	#endif


	// get data from global memory (should be coalesced)

	int x = seed.x; 
	int y = seed.y;

	// edge should be part of the seed datastructure (per block)
	int edge2 = edge >> 2;
	int xorig = x - edge; // one thread per float (edge float4's)
	int yorig = y - edge; // edge rows

	// align xorig such that xorig is a multiple of 2 (speedup is evident)
	int shift = xorig - ((xorig >> 2) << 2);
	if (shift == 1) xorig -= 1;
	else if (shift == 2) xorig += 2;
	else if (shift == 3) xorig += 1;


	int flag1;
	float widthi = 1./width;

	int WW = width;  // array width (argument) (in float4)
	int HH = WW; // height of flat texture // MUST READ AS ARGUMENT

	int thread_id = threadIdx.x + blockDim.x * threadIdx.y;

	//--------------------
	// one iteration per row in the square tile
	// process 4 rows at a time

	__syncthreads();

		flag1 = 1;

		int xid = xorig + (threadIdx.x << 2); // xorig + 0...15
		if (xid < 0 || xid >= WW) flag1 = 0; 

		int yid1 = yorig + (threadIdx.y << 2);
		if (yid1 < 0 || yid1 >= HH) flag1 = 0;

 		int arrayid = xid + yid1*WW; 
		TMP(thread_id).x = float(xid*widthi);
		TMP(thread_id).y = float(yid1*widthi);

		#if 1
		if (flag1 == 0) {
			TMP(thread_id) = make_float4(0.,0.,0.,0.);
		} else {
			float f = g_idata[arrayid].w; // ERROR
			if (int(f+.1) != seed.w) {
				TMP(thread_id) = make_float4(0.,0.,0.,0.);
			} else {
				TMP(thread_id).w = 1.0;
			}
		}
		#endif

	__syncthreads();


	#if 0
		sum[thread_id] = temp[thread_id];
		return;
	#endif

//	//--------------------
//
//// xorig - edge/2, xorig + edge/2 - 1
//
//// For the 16x16 case (hardcoded), the first pass with 64 threads can 
//// only handle 1/2 the domain (1024 floats = 16x16x4). The for loop that
//// follows had a thread handle two floats at a time, so can only handl
//// 1/2 the domain on each pass
//// manually treat each half of the domain

    int offset = 1;
	int ai, bi;
	int sz;

	#if 1
	sz = 128;  // hardcoded for 16x16=256 tile
    // build the sum in place up the tree
    for (int d = sz; d > 0; d >>= 1) {
        __syncthreads();

        if (thread_id < d) {
            ai = offset*(2*thread_id+1)-1;
            bi = offset*(2*thread_id+2)-1;

			//temp[bi].x += temp[ai].x;
			//temp[bi].y += temp[ai].y;
			//temp[bi].w += temp[ai].w;

			TMP(bi).x += TMP(ai).x;
			TMP(bi).y += TMP(ai).y;
			TMP(bi).w += TMP(ai).w;
        }

        offset <<= 1;
    }
	#endif

	__syncthreads();
	#if 0
		sum[thread_id] = temp[thread_id];
	return;
	#endif

	// Something wrong with the results

    // write results to global memory
    __syncthreads();
	if (threadIdx.x == (blockDim.x-1)) {
		int el = edge*edge-1;
		//float nbs = temp[el].w;
		float nbs = TMP(el).w;
		float nbs1 = 1./(nbs);
		if (nbs <= 0.5) nbs1 = 1.;
		//sum[blockIdx.x] = make_float4(temp[el].x*nbs1, temp[el].y*nbs1, 0., nbs); // orig
		sum[blockIdx.x] = make_float4(TMP(el).x*nbs1, TMP(el).y*nbs1, 0., nbs); // orig
	}
}
//----------------------------------------------------------------------

#endif
