#include <hip/driver_types.h>

extern "C" 
void copyToDeviceFromHost_1d(void* dst, const void* src, size_t count)
{
	hipMemcpy(dst, src, count, hipMemcpyHostToDevice);
}
//----------------------------------------------------------------------
extern "C"
void copyToHostFromDevice(void* dst, const void* src, size_t count)
{
	hipMemcpy(dst, src, count, hipMemcpyDeviceToHost);
}
//----------------------------------------------------------------------
extern "C" void cudaMalloc_1d(void** data , int nbBytes) 
{
	hipError_t hipError_t = hipMalloc(data, nbBytes);
	// ERROR PROCESSING
}
//----------------------------------------------------------------------
extern "C" void clear_1d(void* data, size_t count)
{
	hipError_t hipError_t = hipMemset(data, 0, count);
}
//----------------------------------------------------------------------
extern "C" void cudaConfigureCall_ge(int gx, int gy, int gz, int bx, int by, int bz, size_t shared, int tokens)
{
	dim3 grid(gx,gy,gz);
	dim3 block(bx,by,bz);
	hipConfigureCall(grid, block, shared, tokens);
}
//----------------------------------------------------------------------
extern "C" void cudaLaunch_ge(const char* entry)
{
	hipLaunchByPtr(entry);
}
//----------------------------------------------------------------------
extern "C" void cudaSetupArgument_ge(void* arg, size_t count, size_t offset)
{
	hipSetupArgument(arg, count, offset);
}
//----------------------------------------------------------------------
