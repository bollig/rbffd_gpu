#include "hip/hip_runtime.h"
#include <cusp/multiply.h>
#include <cusp/io/matrix_market.h>
#include <cusp/blas.h>

#include "heat_pde_cusp.h"

#include "rbffd/rbffd_cl.h"

//----------------------------------------------------------------------

void HeatPDE_CL::setupTimers()
{
    tm["advance_gpu"] = new EB::Timer("Advance the PDE one step on the GPU") ;
    tm["loadAttach"] = new EB::Timer("Load the GPU Kernels for HeatPDE");
}

void HeatPDE_CL::fillInitialConditions(ExactSolution* exact) {
    // Fill U_G with initial conditions
    this->HeatPDE::fillInitialConditions(exact);

    this->sendrecvUpdates(U_G, "U_G");

    unsigned int nb_nodes = grid_ref.G.size();
    unsigned int solution_mem_bytes = nb_nodes*this->getFloatSize(); 

    std::vector<double> diffusivity(nb_nodes, 0.);

    //FIXME: we're assuming float type on diffusivity. IF we need double, we'll
    //have to move this down.
    this->fillDiffusion(diffusivity, U_G, 0., nb_nodes);

    std::cout << "[HeatPDE_CL] Writing initial conditions to GPU\n"; 
    if (useDouble) {
        cusp::array1d<float, cusp::host_memory> U_cpu(this->U_G.begin(), this->U_G.end());
        // Fill GPU mem with initial solution 
        gpu_solution[INDX_IN] = U_cpu;
        gpu_solution[INDX_OUT] = U_cpu;

        cusp::array1d<float, cusp::host_memory> diffusivity_cpu(diffusivity.begin(), diffusivity.end()); 
        gpu_diffusivity = diffusivity_cpu;
#if 0
        //cusp::io::write_matrix_market_file(gpu_solution[INDX_IN], "Input.mtx"); 
        float* U_G_f = new float[nb_nodes];
        float* diffusivity_f = new float[nb_nodes];
        for (unsigned int i = 0; i < nb_nodes; i++) {
            U_G_f[i] = (float)U_G[i];
            diffusivity_f[i] = (float)diffusivity[i];
        }

        err = queue.enqueueWriteBuffer(gpu_diffusivity, CL_FALSE, 0, solution_mem_bytes, &diffusivity_f[0], NULL, &event);
        queue.finish();

        delete [] U_G_f; 
        delete [] diffusivity_f; 
    // FIXME: change all unsigned int to int. Or unsigned int. Size_t is not supported by GPU.
    std::vector<unsigned int>& bindices = grid_ref.getBoundaryIndices();
    unsigned int nb_bnd = bindices.size();
    //    err = queue.enqueueWriteBuffer(gpu_boundary_indices, CL_FALSE, 0, nb_bnd*sizeof(unsigned int), &bindices[0], NULL, &event);

#endif 
    }
    std::cout << "[HeatPDE_CL] Done\n"; 
}

// Handle the boundary conditions however we want to. 
// NOTE: we must update the solution on the GPU too. 
void HeatPDE_CL::enforceBoundaryConditions(std::vector<SolutionType>& u_t, cusp::array1d<float, cusp::device_memory>& sol, double t)
{
#if 0
    // FIXME: should we mirror the CPU first?
    this->HeatPDE::enforceBoundaryConditions(u_t, t);

    unsigned int nb_stencils = grid_ref.getStencilsSize(); 
    unsigned int stencil_size = grid_ref.getMaxStencilSize(); 
    unsigned int nb_bnd = grid_ref.getBoundaryIndicesSize();
    unsigned int nb_nodes = grid_ref.getNodeListSize(); 
    float cur_time_f = (float) cur_time;

    try {
        bc_kernel.setArg(0, sol);                 // COPY_IN  / COPY OUT
        bc_kernel.setArg(1, this->gpu_boundary_indices);                 // COPY_IN 
        bc_kernel.setArg(2, sizeof(unsigned int), &nb_bnd);               // const 
        bc_kernel.setArg(3, sizeof(float), &cur_time_f);
    } catch (cl::Error er) {
        printf("[setKernelArg] ERROR: %s(%s)\n", er.what(), oclErrorString(er.err()));
    }
    unsigned int safe_launch_size = (nb_bnd > 32) ? nb_bnd : 32;

    err = queue.enqueueNDRangeKernel(bc_kernel, /* offset */ cl::NullRange, 
            /* GLOBAL (work-groups in the grid)  */   cl::NDRange(safe_launch_size), 
            /* LOCAL (work-items per work-group) */    cl::NullRange, NULL, &event);

    if (err != CL_SUCCESS) {
        std::cerr << "CommandQueue::enqueueNDRangeKernel()" \
            " failed (" << err << ")\n";
        std::cout << "FAILED TO ENQUEUE KERNEL" << std::endl;
        exit(EXIT_FAILURE);
    }

    //queue.finish();
#endif 
}


void HeatPDE_CL::fillGPUMat(RBFFD::DerType which, cusp::csr_matrix<unsigned int, float, cusp::device_memory>& gpu_buffer) {
    unsigned int nb_nodes = grid_ref.getNodeListSize(); 
    unsigned int nb_stencils = grid_ref.getStencilsSize();
    unsigned int max_st_size = grid_ref.getMaxStencilSize();
    
    cusp::coo_matrix<unsigned int, float, cusp::host_memory> weights_cpu(nb_nodes, nb_nodes, nb_nodes*max_st_size); 
    std::vector<double*>& weights = der_ref.getWeights(which);

    unsigned int indx = 0;
    for (unsigned int i = 0; i < nb_stencils; i++) {
        StencilType& st = grid_ref.getStencil(i);
        unsigned int stencil_size = st.size();
        for (unsigned int j = 0; j < stencil_size; j++) {
            weights_cpu.row_indices[indx] = st[0]; 
            weights_cpu.column_indices[indx] = st[j];
            weights_cpu.values[indx] = weights[i][j];
        }
        // 0's automatically pad the end of our stencil
    }

    // Copies to GPU.
    gpu_buffer = weights_cpu; 
}

//----------------------------------------------------------------------

void HeatPDE_CL::assemble() 
{
    if (!weightsPrecomputed) {
        der_ref.computeAllWeightsForAllStencils();
    }

    // Put weights on GPU. 
    if (!assembled) {
        this->fillGPUMat(RBFFD::X, this->x_weights_gpu); 
    std::cout << "Done assembling\n";
        this->fillGPUMat(RBFFD::Y, this->y_weights_gpu); 
    std::cout << "Done assembling\n";
        this->fillGPUMat(RBFFD::Z, this->z_weights_gpu); 
    std::cout << "Done assembling\n";
        this->fillGPUMat(RBFFD::LAPL, this->l_weights_gpu); 
        assembled = true;
    }

}

//----------------------------------------------------------------------

void HeatPDE_CL::advance(TimeScheme which, double delta_t) {
    tm["advance_gpu"]->start(); 
    switch (which) 
    {
        case EULER: 
            advanceFirstOrderEuler(delta_t); 
            break; 

        case MIDPOINT: 
            advanceSecondOrderMidpoint(delta_t);
            break;  
#if 0
        case RK4: 
            advanceRungeKutta4(delta_t); 
            break;
#endif 
        default: 
            std::cout << "[HeatPDE_CL] Invalid TimeScheme specified. Bailing...\n";
            exit(EXIT_FAILURE); 
            break; 
    };
    cur_time += delta_t; 
    tm["advance_gpu"]->stop(); 
}

void HeatPDE_CL::syncSetRSingle(std::vector<SolutionType>& vec, cusp::array1d<float,cusp::device_memory>& gpu_vec) {
    unsigned int nb_nodes = grid_ref.getNodeListSize();
    unsigned int set_G_size = grid_ref.G.size();
    unsigned int set_Q_size = grid_ref.Q.size(); 
    unsigned int set_R_size = grid_ref.R.size();

    unsigned int float_size = this->getFloatSize();

    // OUR SOLUTION IS ARRANGED IN THIS FASHION: 
    //  { Q\B D O R } where B = union(O, D) and Q = union(Q\B D O)
    unsigned int offset_to_set_R = set_Q_size;

    unsigned int solution_mem_bytes = set_G_size*float_size; 
    unsigned int set_R_bytes = set_R_size * float_size;

    // backup the current solution so we can perform intermediate steps
    std::vector<float> r_update_f(set_R_size,-1.); 

    if (set_R_size > 0) {

        // Update CPU mem with R; 
        // NOTE: This is a single precision kernel call so we need to convert
        // the U_G to single precision
        for (int i = 0 ; i < set_R_size; i++) {
            r_update_f[i] = (float)vec[offset_to_set_R + i]; 
        }

        // Synchronize just the R part on GPU (CL_FALSE here indicates we dont block on write
        // NOTE: offset parameter to enqueueWriteBuffer is ONLY for the GPU side offset. The CPU offset needs to be managed directly on the CPU pointer: &U_G[offset_cpu]
        //       err = queue.enqueueWriteBuffer(gpu_vec, CL_FALSE, offset_to_set_R * float_size, set_R_bytes, &r_update_f[0], NULL, &event);

    }
}


// General routine to copy the set R indices vec up to gpu_vec
void HeatPDE_CL::syncSetRDouble(std::vector<SolutionType>& vec, cusp::array1d<float, cusp::device_memory>& gpu_vec) {
    unsigned int nb_nodes = grid_ref.getNodeListSize();
    unsigned int set_G_size = grid_ref.G.size();
    unsigned int set_Q_size = grid_ref.Q.size(); 
    unsigned int set_R_size = grid_ref.R.size();

    unsigned int float_size = this->getFloatSize();

    // OUR SOLUTION IS ARRANGED IN THIS FASHION: 
    //  { Q\B D O R } where B = union(O, D) and Q = union(Q\B D O)
    unsigned int offset_to_set_R = set_Q_size;

    unsigned int solution_mem_bytes = set_G_size*float_size; 
    unsigned int set_R_bytes = set_R_size * float_size;

    if (set_R_size > 0) {

        // Synchronize just the R part on GPU (CL_FALSE here indicates we dont
        // block on write NOTE: offset parameter to enqueueWriteBuffer is ONLY
        // for the GPU side offset. The CPU offset needs to be managed directly
        // on the CPU pointer: &U_G[offset_cpu]
#if 0
        err = queue.enqueueWriteBuffer(gpu_vec, CL_FALSE, offset_to_set_R * float_size, set_R_bytes, &vec[offset_to_set_R], NULL, &event);
#endif   
    }
}

// General routine to copy the set O indices from gpu_vec down to vec
void HeatPDE_CL::syncSetOSingle(std::vector<SolutionType>& vec, cusp::array1d<float, cusp::device_memory>& gpu_vec) {
    unsigned int nb_nodes = grid_ref.getNodeListSize();
    unsigned int set_G_size = grid_ref.G.size();
    unsigned int set_Q_size = grid_ref.Q.size(); 
    unsigned int set_O_size = grid_ref.O.size();

    unsigned int float_size = this->getFloatSize();

    // OUR SOLUTION IS ARRANGED IN THIS FASHION: 
    //  { Q\B D O R } where B = union(O, D) and Q = union(Q\B D O)
    unsigned int offset_to_set_O = (set_Q_size - set_O_size);

    unsigned int solution_mem_bytes = set_G_size*float_size; 
    unsigned int set_O_bytes = set_O_size * float_size;

    // backup the current solution so we can perform intermediate steps
    std::vector<float> o_update_f(set_O_size,1.);


    if (set_O_size > 0) {
        // Pull only information required for neighboring domains back to the CPU 
        //        err = queue.enqueueReadBuffer(gpu_vec, CL_FALSE, offset_to_set_O * float_size, set_O_bytes, &o_update_f[0], NULL, &event);

        // Probably dont need this if we want to overlap comm and comp. 
        //       queue.finish();

        // NOTE: this is only required because we're calling a single precision
        // kernel 
        for (unsigned int i = 0; i < set_O_size; i++) {
            //    std::cout << "output u[" << i << "(global: " << grid_ref.l2g(offset_to_set_O+i) << ")] = " << U_G[offset_to_set_O + i] << "\t" << o_update_f[i] << std::endl;
            vec[offset_to_set_O + i] = (double) o_update_f[i];
        }
    }
}


void HeatPDE_CL::syncSetODouble(std::vector<SolutionType>& vec, cusp::array1d<float, cusp::device_memory>& gpu_vec) {
    unsigned int nb_nodes = grid_ref.getNodeListSize();
    unsigned int set_G_size = grid_ref.G.size();
    unsigned int set_Q_size = grid_ref.Q.size(); 
    unsigned int set_O_size = grid_ref.O.size();

    unsigned int float_size = this->getFloatSize();

    // OUR SOLUTION IS ARRANGED IN THIS FASHION: 
    //  { Q\B D O R } where B = union(O, D) and Q = union(Q\B D O)
    unsigned int offset_to_set_O = (set_Q_size - set_O_size);

    unsigned int solution_mem_bytes = set_G_size*float_size; 
    unsigned int set_O_bytes = set_O_size * float_size;

    // backup the current solution so we can perform intermediate steps
    std::vector<float> o_update_f(set_O_size,1.);


    if (set_O_size > 0) {
        // Pull only information required for neighboring domains back to the CPU 
        //        err = queue.enqueueReadBuffer(gpu_vec, CL_FALSE, offset_to_set_O * float_size, set_O_bytes, &vec[offset_to_set_O], NULL, &event);

    }
}


//----------------------------------------------------------------------
// FIXME: this is a single precision version
void HeatPDE_CL::advanceFirstOrderEuler(double delta_t) {

    // Target (st5): 0.3991 ms
    //        (st33): 1.2 ms
    // GPU: 
    // Without diffusion, boundary or f(u) eval (st33): 0.3599
    // Without boundary or f(u) eval (st33): 0.3562
    // Without boundary (st33): 4.8389
    // no boundary, K*Laplacian only (no gradK . gradU) (st33): 1.1898

    // If we need to assemble a matrix L for solving implicitly, this is the routine to do that. 
    // For explicit schemes we can just solve for our weights and have them stored in memory.
    this->assemble(); 

    // 1) Launch kernel for set QmD (will take a while, so in the meantime...)
    this->launchEulerSetQmDKernel(delta_t, this->gpu_solution[INDX_IN], this->gpu_solution[INDX_OUT]); 

    // NOTE: when run in serial only one kernel launch is required. 
    if (comm_ref.getSize() > 1) {
        std::cout << "INSIDE EULER set D STUFF\n";
        // 2) OVERLAP: Transfer set O from the input to the CPU for synchronization acros CPUs
        if (useDouble) {
            this->syncSetODouble(this->U_G, gpu_solution[INDX_IN]);
        } else {
            this->syncSetOSingle(this->U_G, gpu_solution[INDX_IN]); 
        }

        // 3) OVERLAP: Transmit between CPUs
        // NOTE: Require an MPI barrier here
        this->sendrecvUpdates(U_G, "U_G");


        // 4) OVERLAP: Update the input with set R
        if (useDouble) {
            this->syncSetRDouble(this->U_G, gpu_solution[INDX_IN]);
        } else {
            this->syncSetRSingle(this->U_G, gpu_solution[INDX_IN]); 
        }

        // 6) Launch a SECOND kernel to complete set D for this step (NOTE: in
        // higher order timeschemes we need to perform ADDITIONAL communication
        // here. Also, this MIGHT modify the boundary value so we should enforce
        // conditions AFTER this kernel)
        this->launchEulerSetDKernel(delta_t, this->gpu_solution[INDX_IN], this->gpu_solution[INDX_OUT]);
        //queue.finish();
    }
    //    queue.finish();

    // 5) FINAL: reset boundary solution on INDX_OUT
    // COST: 0.3 ms
    this->enforceBoundaryConditions(U_G, this->gpu_solution[INDX_OUT], cur_time); 

    // Fire events to force the queue to execute.
    //queue.finish();

    // Flip our ping pong buffers. 
    swap(INDX_IN, INDX_OUT);
}



void HeatPDE_CL::syncCPUtoGPU() {
    std::cout << "SYNC CPU to GPU: " << INDX_IN << std::endl;
    unsigned int nb_nodes = grid_ref.getNodeListSize();
    unsigned int solution_mem_bytes = nb_nodes * this->getFloatSize();

    if (useDouble) {
        //        err = queue.enqueueReadBuffer(gpu_solution[INDX_IN], CL_FALSE, 0, solution_mem_bytes, &U_G[0], NULL, &event);
    } else {
        float* U_G_f = new float[nb_nodes]; 
        //       err = queue.enqueueReadBuffer(gpu_solution[INDX_IN], CL_FALSE, 0, solution_mem_bytes, &U_G_f[0], NULL, &event);

        //        queue.finish();
        for (unsigned int i = 0; i < nb_nodes; i++) {
#if 0
            double diff = fabs( U_G[i] - U_G_f[i] ); 
            if (diff > 1e-4) {
                std::cout << "GPUvsCPU diff[" << i << "]: " << diff << std::endl;
            }
#endif 
            U_G[i] = (double)U_G_f[i]; 
        }
        delete [] U_G_f; 
    }
}

//----------------------------------------------------------------------
// FIXME: this is a single precision version
void HeatPDE_CL::advanceSecondOrderMidpoint(double delta_t) {
#if 0 
    // If we need to assemble a matrix L for solving implicitly, this is the routine to do that. 
    // For explicit schemes we can just solve for our weights and have them stored in memory.
    this->assemble(); 

    //-------- Overlap beweeen these: ------------
    // NOTE: syncSet*** ONLY copies between CPU and GPU. It does not synchronize across CPUs.
    // Use sendrecvUpdates to perform an interproc comm.
    if (useDouble) {
        this->syncSetRDouble(this->U_G, gpu_solution[INDX_IN]);
    } else {
        this->syncSetRSingle(this->U_G, gpu_solution[INDX_IN]); 
    }

    // Launch kernel
    //  params: timestep, vec_for_deriv_calc, vec_for_sum_rhs, vec_for_sum_lhs
    //  In other words: s2 = s1 + dt * d(s0)/dt; 
    //
    //  Euler: 
    //      s1 = s0 + dt * d(s0)/dt
    //
    //  Midpoint: 
    //      s1 = s0 + 0.5 dt * d(s0)/dt
    //      s2 = s0 + dt * d(s1)/dt
    //
    //  RK4: 
    //      s1 = s0 + dt
    this->launchStepKernel( 0.5*delta_t, this->gpu_solution[INDX_IN], this->gpu_solution[INDX_IN], this->gpu_solution[INDX_INTERMEDIATE_1] ); 

    // Enforce boundary using GPU, but specify we want to use the intermediate buffer
    this->enforceBoundaryConditions(U_G, this->gpu_solution[INDX_INTERMEDIATE_1], cur_time+0.5*delta_t); 

    // Since our syncSet****(..) routines ONLY sync the sets at the tail end of
    // the solution (i.e., sets O and R), 
    // we'll just re-use U_G as scratch space. So long as we dont copy U_G to
    // the GPU calling syncSet*** on our INDX_OUT will overwrite any
    // intermediate values stored there temporarily
    // If we want to match the GPU we
    // should do: syncCPUtoGPU()
    if (useDouble) {
        this->syncSetODouble(this->U_G, gpu_solution[INDX_INTERMEDIATE_1]);
    } else {
        this->syncSetOSingle(this->U_G, gpu_solution[INDX_INTERMEDIATE_1]); 
    }

    // Should send intermediate steps by copying down from GPU, sending, then
    // copying back up to GPU
    this->sendrecvUpdates(this->U_G, "intermediate_U_G");

    if (useDouble) {
        this->syncSetRDouble(this->U_G, gpu_solution[INDX_INTERMEDIATE_1]);
    } else {
        this->syncSetRSingle(this->U_G, gpu_solution[INDX_INTERMEDIATE_1]); 
    }

    this->launchStepKernel( delta_t, this->gpu_solution[INDX_IN], this->gpu_solution[INDX_INTERMEDIATE_1], this->gpu_solution[INDX_OUT] ); 
    //-------- END OVERLAP -----------------------

    // reset boundary solution on INDX_OUT
    this->enforceBoundaryConditions(U_G, this->gpu_solution[INDX_OUT], cur_time); 

    if (useDouble) {
        this->syncSetODouble(this->U_G, gpu_solution[INDX_OUT]);
    } else {
        this->syncSetOSingle(this->U_G, gpu_solution[INDX_OUT]); 
    }

    //    queue.finish();

    //    this->syncCPUtoGPU(); 

#if 0
    for (int i = 0; i < nb_nodes; i++) {
        std::cout << "u[" << i << "] = " << U_G[i] << std::endl;
    }
#endif 

    // synchronize();
    this->sendrecvUpdates(U_G, "U_G");

    //exit(EXIT_FAILURE);

    swap(INDX_IN, INDX_OUT);
#endif 
}


//----------------------------------------------------------------------
// FIXME: this is a single precision version
void HeatPDE_CL::advanceRungeKutta4(double delta_t) {

#if 0
    // If we need to assemble a matrix L for solving implicitly, this is the routine to do that. 
    // For explicit schemes we can just solve for our weights and have them stored in memory.
    this->assemble(); 

    //-------- Overlap beweeen these: ------------
    // NOTE: syncSet*** ONLY copies between CPU and GPU. It does not synchronize across CPUs.
    // Use sendrecvUpdates to perform an interproc comm.
    if (useDouble) {
        this->syncSetRDouble(this->U_G, gpu_solution[INDX_IN]);
    } else {
        this->syncSetRSingle(this->U_G, gpu_solution[INDX_IN]); 
    }

    // Launch kernel
    //  params: timestep, vec_for_deriv_calc, vec_for_sum_rhs, vec_for_sum_lhs
    //  In other words: s2 = s1 + dt * d(s0)/dt; 
    //
    //  Euler: 
    //      s1 = s0 + dt * d(s0)/dt
    //
    //  Midpoint: 
    //      s1 = s0 + 0.5 dt * d(s0)/dt
    //      s2 = s0 + dt * d(s1)/dt
    //
    //  RK4: 
    //  
    // K1 t_n = cur_time + 0*dt
    // S1 = s0 + 0.5dt * (k1 + f)
    // params: dt on solve, dt on advance, input solve, output solve, input advance, output advance
    this->launchRK4_K_Kernel( 0.f, 0.5*delta_t, this->gpu_solution[INDX_IN], this->gpu_feval[0], this->gpu_solution[INDX_IN], this->gpu_solution[INDX_INTERMEDIATE_1] ); 

    // Enforce boundary using GPU, but specify we want to use the intermediate buffer
    this->enforceBoundaryConditions(U_G, this->gpu_solution[INDX_INTERMEDIATE_1], cur_time+0.5*delta_t); 

    // Since our syncSet****(..) routines ONLY sync the sets at the tail end of
    // the solution (i.e., sets O and R), 
    // we'll just re-use U_G as scratch space. So long as we dont copy U_G to
    // the GPU calling syncSet*** on our INDX_OUT will overwrite any
    // intermediate values stored there temporarily
    // If we want to match the GPU we
    // should do: syncCPUtoGPU()
    if (useDouble) {
        this->syncSetODouble(this->U_G, gpu_solution[INDX_INTERMEDIATE_1]);
    } else {
        this->syncSetOSingle(this->U_G, gpu_solution[INDX_INTERMEDIATE_1]); 
    }

    // Should send intermediate steps by copying down from GPU, sending, then
    // copying back up to GPU
    this->sendrecvUpdates(this->U_G, "intermediate_U_G");

    if (useDouble) {
        this->syncSetRDouble(this->U_G, gpu_solution[INDX_INTERMEDIATE_1]);
    } else {
        this->syncSetRSingle(this->U_G, gpu_solution[INDX_INTERMEDIATE_1]); 
    }

    // K2 t_n = cur_time + 0.5*dt
    // S2 = s0 + 0.5dt * (k2 + f)
    this->launchRK4_K_Kernel( 0.5f*delta_t, 0.5*delta_t, this->gpu_solution[INDX_INTERMEDIATE_1], this->gpu_feval[1], this->gpu_solution[INDX_IN], this->gpu_solution[INDX_INTERMEDIATE_1] ); 

    // Enforce boundary using GPU, but specify we want to use the intermediate buffer
    this->enforceBoundaryConditions(U_G, this->gpu_solution[INDX_INTERMEDIATE_1], cur_time+0.5*delta_t); 

    // Since our syncSet****(..) routines ONLY sync the sets at the tail end of
    // the solution (i.e., sets O and R), 
    // we'll just re-use U_G as scratch space. So long as we dont copy U_G to
    // the GPU, calling syncSet*** on our INDX_OUT will overwrite any
    // intermediate values stored there temporarily
    // If we want to match the GPU we
    // should do: syncCPUtoGPU()
    if (useDouble) {
        this->syncSetODouble(this->U_G, gpu_solution[INDX_INTERMEDIATE_1]);
    } else {
        this->syncSetOSingle(this->U_G, gpu_solution[INDX_INTERMEDIATE_1]); 
    }

    // Should send intermediate steps by copying down from GPU, sending, then
    // copying back up to GPU
    this->sendrecvUpdates(this->U_G, "intermediate_U_G");

    if (useDouble) {
        this->syncSetRDouble(this->U_G, gpu_solution[INDX_INTERMEDIATE_1]);
    } else {
        this->syncSetRSingle(this->U_G, gpu_solution[INDX_INTERMEDIATE_1]); 
    }


    // K3 t_n = cur_time + 0.5*dt
    // S3 = s0 + dt * (k3 + f)
    this->launchRK4_K_Kernel( 0.5f*delta_t, delta_t, this->gpu_solution[INDX_INTERMEDIATE_1], this->gpu_feval[2], this->gpu_solution[INDX_IN], this->gpu_solution[INDX_INTERMEDIATE_1] ); 

    // Enforce boundary using GPU, but specify we want to use the intermediate buffer
    this->enforceBoundaryConditions(U_G, this->gpu_solution[INDX_INTERMEDIATE_1], cur_time+delta_t); 
    // Since our syncSet****(..) routines ONLY sync the sets at the tail end of
    // the solution (i.e., sets O and R), 
    // we'll just re-use U_G as scratch space. So long as we dont copy U_G to
    // the GPU calling syncSet*** on our INDX_OUT will overwrite any
    // intermediate values stored there temporarily
    // If we want to match the GPU we
    // should do: syncCPUtoGPU()
    if (useDouble) {
        this->syncSetODouble(this->U_G, gpu_solution[INDX_INTERMEDIATE_1]);
    } else {
        this->syncSetOSingle(this->U_G, gpu_solution[INDX_INTERMEDIATE_1]); 
    }

    // Should send intermediate steps by copying down from GPU, sending, then
    // copying back up to GPU
    this->sendrecvUpdates(this->U_G, "intermediate_U_G");

    if (useDouble) {
        this->syncSetRDouble(this->U_G, gpu_solution[INDX_INTERMEDIATE_1]);
    } else {
        this->syncSetRSingle(this->U_G, gpu_solution[INDX_INTERMEDIATE_1]); 
    }

    // K3 t_n = cur_time + 0.5*dt
    // S3 = s0 + dt * (k3 + f)
    this->launchRK4_Final_Kernel( 0.5f*delta_t, delta_t, this->gpu_solution[INDX_IN], this->gpu_feval[0], this->gpu_feval[1], this->gpu_feval[2], this->gpu_solution[INDX_OUT] ); 

    // reset boundary solution on INDX_OUT
    this->enforceBoundaryConditions(U_G, this->gpu_solution[INDX_OUT], cur_time); 

    if (useDouble) {
        this->syncSetODouble(this->U_G, gpu_solution[INDX_OUT]);
    } else {
        this->syncSetOSingle(this->U_G, gpu_solution[INDX_OUT]); 
    }

    //queue.finish();

    //    this->syncCPUtoGPU(); 

#if 0
    for (int i = 0; i < nb_nodes; i++) {
        std::cout << "u[" << i << "] = " << U_G[i] << std::endl;
    }
#endif 

    // synchronize();
    this->sendrecvUpdates(U_G, "U_G");

    //exit(EXIT_FAILURE);

    swap(INDX_IN, INDX_OUT);
#endif 
}


//----------------------------------------------------------------------
//
void HeatPDE_CL::allocateGPUMem() {

    unsigned int nb_nodes = grid_ref.getNodeListSize();
    unsigned int nb_stencils = grid_ref.getStencilsSize();
    unsigned int nb_bnd = grid_ref.getBoundaryIndicesSize();

    cout << "Allocating GPU memory for HeatPDE\n";

    unsigned int solution_mem_bytes = nb_nodes * this->getFloatSize();

    unsigned int bytesAllocated = 0;
#if 0
    gpu_solution[INDX_IN] = cl::Buffer(context, CL_MEM_READ_WRITE, solution_mem_bytes, NULL, &err);
    bytesAllocated += solution_mem_bytes; 
    gpu_solution[INDX_OUT] = cl::Buffer(context, CL_MEM_READ_WRITE, solution_mem_bytes, NULL, &err);
    bytesAllocated += solution_mem_bytes; 
    gpu_solution[INDX_INTERMEDIATE_1] = cl::Buffer(context, CL_MEM_READ_WRITE, solution_mem_bytes, NULL, &err);
    bytesAllocated += solution_mem_bytes; 

    gpu_diffusivity = cl::Buffer(context, CL_MEM_READ_WRITE, solution_mem_bytes, NULL, &err);

    gpu_boundary_indices = cl::Buffer(context, CL_MEM_READ_ONLY, nb_bnd * sizeof(unsigned int), NULL, &err);

    std::cout << "Allocated: " << bytesAllocated << " bytes (" << ((bytesAllocated / 1024.)/1024.) << "MB)" << std::endl;
#endif 
}

//----------------------------------------------------------------------
//

void HeatPDE_CL::launchEulerSetQmDKernel( double dt, cusp::array1d<float, cusp::device_memory>& sol_in, cusp::array1d<float, cusp::device_memory>& sol_out)
{
    // 1) Assume no parallelism to start. Then all we need is:  y = y + dt*f
    //      f = A * y
    cusp::multiply(l_weights_gpu, sol_in, sol_out);  
    cusp::blas::axpy(sol_in, sol_out, (float)dt);  
}
void HeatPDE_CL::launchEulerSetDKernel( double dt, cusp::array1d<float, cusp::device_memory>& sol_in, cusp::array1d<float, cusp::device_memory>& sol_out) 
{
    ;
}
